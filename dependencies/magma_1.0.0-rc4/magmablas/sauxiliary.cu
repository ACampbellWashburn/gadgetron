#include "hip/hip_runtime.h"
/*
    -- MAGMA (version 1.0) --
       Univ. of Tennessee, Knoxville
       Univ. of California, Berkeley
       Univ. of Colorado, Denver
       November 2010

       @generated s

*/
#include "common_magma.h"

/* ////////////////////////////////////////////////////////////////////////////
   -- This is an auxiliary routine called from sgehrd.  The routine is called
      in 16 blocks, 32 thread per block and initializes to zero the 1st 
      32x32 block of A.
*/

__global__ void sset_to_zero(float *A, int lda){
    int ind = blockIdx.x*lda + threadIdx.x;
    
    A += ind;
    A[0] = MAGMA_S_ZERO;
//   A[16*lda] = 0.;
}

__global__ void sset_nbxnb_to_zero(int nb, float *A, int lda){
   int ind = blockIdx.x*lda + threadIdx.x, i, j;

   A += ind;
   for(i=0; i<nb; i+=32){
     for(j=0; j<nb; j+=32)
         A[j] = MAGMA_S_ZERO;
     A += 32*lda;
   }
}

void szero_32x32_block(float *A, int lda)
{
  // sset_to_zero<<<16, 32>>>(A, lda);
  sset_to_zero<<<32, 32>>>(A, lda);
}

void szero_nbxnb_block(int nb, float *A, int lda)
{
  sset_nbxnb_to_zero<<<32, 32>>>(nb, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- GPU kernel for initializing a matrix by 0
*/
#define slaset_threads 64

__global__ void slaset(int m, int n, float *A, int lda){
   int ibx = blockIdx.x * slaset_threads;
   int iby = blockIdx.y * 32;

   int ind = ibx + threadIdx.x;

   A += ind + __mul24(iby, lda);

   #pragma unroll
   for(int i=0; i<32; i++)
     if (iby+i < n && ind < m)
        A[i*lda] = MAGMA_S_ZERO;
}

/* ////////////////////////////////////////////////////////////////////////////
   -- Set the m x n matrix pointed by A to 0 on the GPU.
*/
extern "C" void
magmablas_slaset(magma_int_t m, magma_int_t n, 
                 float *A, magma_int_t lda)
{
   dim3 threads(slaset_threads, 1, 1);
   dim3 grid(m/slaset_threads+(m % slaset_threads != 0), n/32+(n%32!=0));

   slaset<<< grid, threads >>> (m, n, A, lda);
}

/* ////////////////////////////////////////////////////////////////////////////
   -- Given two matrices, 'a' on the CPU and 'da' on the GPU, this function
      returns the Frobenious norm of the difference of the two matrices.
      The function is used for debugging.
*/
float cpu_gpu_sdiff(int M, int N, float * a, int lda, float *da, int ldda)
{
  int one = 1, j;
  float mone = MAGMA_S_NEG_ONE;
  float  work[1];
  float *ha = (float*)malloc( M * N * sizeof(float));
  float res;

  hipblasGetMatrix(M, N, sizeof(float), da, ldda, ha, M);
  for(j=0; j<N; j++)
    blasf77_saxpy(&M, &mone, a+j*lda, &one, ha+j*M, &one);
  res = lapackf77_slange("f", &M, &N, ha, &M, work);

  free(ha);
  return res;
}

