#include "hip/hip_runtime.h"
#include "cuNDArray.h"

template <class T> 
__global__ void cuNDArray_permute_kernel(T* in, T* out, 
					 unsigned int ndim,
					 unsigned int* dims,
					 unsigned int* strides_out,
					 unsigned long int elements,
					 int shift_mode)
{
  unsigned long idx_in = blockIdx.x*blockDim.x+threadIdx.x;
  unsigned long idx_out = 0;

  unsigned long idx_in_tmp = idx_in;
  if (idx_in < elements) {

    unsigned int cur_index;
    for (unsigned int i = 0; i < ndim; i++) {
      unsigned long idx_in_remainder = idx_in_tmp / dims[i];
      cur_index = idx_in_tmp-(idx_in_remainder*dims[i]); //cur_index = idx_in_tmp%dims[i];
      if (shift_mode < 0) { //IFFTSHIFT
	idx_out += ((cur_index+(dims[i]>>1))%dims[i])*strides_out[i];
      } else if (shift_mode > 0) { //FFTSHIFT
	idx_out += ((cur_index+((dims[i]+1)>>1))%dims[i])*strides_out[i];
      } else {
	idx_out += cur_index*strides_out[i];
      }
      idx_in_tmp = idx_in_remainder;
    }

    out[idx_in] = in[idx_out];

  }

}


template <class T> int cuNDArray_permute(cuNDArray<T>* in,
 					 cuNDArray<T>* out,
					 std::vector<unsigned int> order,
					 int shift_mode)
{
  hipError_t err;

  T* in_ptr = in->data_;
  T* out_ptr = 0;

  if (out) {
    out_ptr = out->data_;
  } else {
    if (hipMalloc((void**) &out_ptr, in->elements_*sizeof(T)) != hipSuccess) {
      std::cerr << "cuNDArray_permute : Error allocating CUDA memory" << std::endl;
      out_ptr = 0;
      return -1;
    }
  }

  unsigned int* dims        = new unsigned int[in->dimensions_.size()];
  unsigned int* strides_out = new unsigned int[in->dimensions_.size()];
  if (!dims || !strides_out) {
    std::cerr << "cuNDArray_permute: failed to allocate temporary storage for arrays" << std::endl;
    return -1;
  }

  for (unsigned int i = 0; i < in->dimensions_.size(); i++) {
    dims[i] = in->dimensions_[order[i]];
    strides_out[i] = 1;
    
    for (unsigned int j = 0; j < order[i]; j++) {
      strides_out[i] *= in->dimensions_[j];
    }
  }

  unsigned int* dims_dev        = 0;
  unsigned int* strides_out_dev = 0;
  
  if (hipMalloc((void**) &dims_dev, in->dimensions_.size()*sizeof(unsigned int)) != hipSuccess) {
    std::cerr << "cuNDArray_permute : Error allocating CUDA dims memory" << std::endl;
    return -1;
  }
  
  if (hipMalloc((void**) &strides_out_dev, in->dimensions_.size()*sizeof(unsigned int)) != hipSuccess) {
    std::cerr << "cuNDArray_permute : Error allocating CUDA strides_out memory" << std::endl;
    return -1;
  }
  
  if (hipMemcpy(dims_dev, dims, in->dimensions_.size()*sizeof(unsigned int), hipMemcpyHostToDevice) !=
      hipSuccess) {

    err = hipGetLastError();
    std::cerr << "cuNDArray_permute : Error uploading dimensions to device, " 
	      << hipGetErrorString(err) << std::endl;
    return -1;
  }

  if (hipMemcpy(strides_out_dev, strides_out, in->dimensions_.size()*sizeof(unsigned int), hipMemcpyHostToDevice) !=
      hipSuccess) {
    std::cerr << "cuNDArray_permute : Error uploading strides to device" << std::endl;
    return -1;
  }

  dim3 blockDim(512,1,1);
  dim3 gridDim((unsigned int) ceil((double)in->elements_/blockDim.x), 1, 1 );

  cuNDArray_permute_kernel<<< gridDim, blockDim >>>( in_ptr, out_ptr, in->dimensions_.size(), 
						     dims_dev, strides_out_dev, in->elements_, shift_mode);

  err = hipGetLastError();
  if( err != hipSuccess ){
    std::cerr << "cuNDArray_permute : Error during kernel call: " << hipGetErrorString(err) << std::endl;
    return -1;
  }

  if (hipFree(dims_dev) != hipSuccess) {
    err = hipGetLastError();
    std::cerr << "cuNDArray_permute: failed to delete device memory (dims_dev) " 
	      << hipGetErrorString(err) << std::endl;
    return -1;
  }

  if (hipFree(strides_out_dev) != hipSuccess) {
    err = hipGetLastError();
    std::cerr << "cuNDArray_permute: failed to delete device memory (strides_out_dev) " 
	      << hipGetErrorString(err) << std::endl;
    return -1;
  }
  
  delete [] dims;
  delete [] strides_out;

  if (!out) {
    std::vector<unsigned int> new_dims;
    for (unsigned int i = 0; i < in->dimensions_.size(); i++) {
      new_dims.push_back(in->dimensions_[order[i]]);
    }
    in->dimensions_ = new_dims;
    if (hipFree(in->data_) != hipSuccess) {
	std::cerr << "cuNDArray_permute: failed to delete device memory" << std::endl;
	return -1;
    }
    in->data_ = out_ptr;
  }

  return 0;
}

template int cuNDArray_permute<>(cuNDArray<float>* in,
				 cuNDArray<float>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template int cuNDArray_permute<>(cuNDArray<float2>* in,
				 cuNDArray<float2>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template int cuNDArray_permute<>(cuNDArray<float3>* in,
				 cuNDArray<float3>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template int cuNDArray_permute<>(cuNDArray<float4>* in,
				 cuNDArray<float4>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template int cuNDArray_permute<>(cuNDArray<unsigned int>* in,
				 cuNDArray<unsigned int>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template int cuNDArray_permute<>(cuNDArray<uint2>* in,
				 cuNDArray<uint2>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template int cuNDArray_permute<>(cuNDArray<uint3>* in,
				 cuNDArray<uint3>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template int cuNDArray_permute<>(cuNDArray<uint4>* in,
				 cuNDArray<uint4>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);



