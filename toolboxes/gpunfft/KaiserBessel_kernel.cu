#include "hip/hip_runtime.h"
//
// Kaiser-Bessel convolution kernels
//

// 'bessi0' is taken from numerical recipes in C

__inline__ __device__ double 
bessi0(double x)
{
  double ax,ans,y;
  if ((ax=fabs(x))<3.75) 
    {
      y=x/3.75;
      y*=y;
      ans=1.0+y*(3.5156229+y*(3.0899424+y*(1.2067492+y*(0.2659732+y*(0.0360768+y*0.0045813)))));
    } 
  else 
    {
      y=3.75/ax;
      ans=(-0.02057706+y*(0.02635537+y*(-0.01647633+(y*0.00392377))));
      ans=(std::exp(ax)/std::sqrt(ax))*(0.39894228+y*(0.01328592+y*(0.00225319+y*(-0.00157565+y*(0.00916281+y*ans)))));
    }
  //const double scale = 1.0/1000000000.0;
  return ans;//*scale;
}

__inline__ __device__ float 
bessi0(float x)
{
  float ax,ans,y;
  if ((ax=fabsf(x)) <3.75f) 
    {
      y=x/3.75f;
      y*=y;
      ans=1.0f+y*(3.5156229f+y*(3.0899424f+y*(1.2067492f+y*(0.2659732f+y*(0.0360768f+y*0.0045813f)))));
    } 
  else 
    {
      y=3.75f/ax;
      ans=(-0.02057706f+y*(0.02635537f+y*(-0.01647633f+(y*0.00392377f))));
      ans=(std::exp(ax)/std::sqrt(ax))*(0.39894228f+y*(0.01328592f+y*(0.00225319f+y*(-0.00157565f+y*(0.00916281f+y*ans)))));
    }
  //const float scale = 1.0f/1000000000.0f;
  return ans;//*scale;
}

// Kaiser Bessel according to Beatty et. al. IEEE TMI 2005;24(6):799-808.
// There is a slight difference wrt Jackson's formulation, IEEE TMI 1991;10(3):473-478.

__inline__ __device__ double
KaiserBessel( double u, double matrix_size_os, double one_over_W, double beta )
{
  double _tmp = 2.0*u*one_over_W;
  double tmp = _tmp*_tmp;
  double arg = beta*std::sqrt(1.0-tmp);
  double bessi = bessi0(arg);
  double ret = matrix_size_os*bessi*one_over_W;
  return ret;
}

__inline__ __device__ float
KaiserBessel( float u, float matrix_size_os, float one_over_W, float beta )
{
  float _tmp = 2.0f*u*one_over_W;
  float tmp = _tmp*_tmp;
  float arg = beta*std::sqrt(1.0f-tmp);
  float bessi = bessi0(arg);
  float ret = matrix_size_os*bessi*one_over_W;
  return ret;
}

//
// Below the intended interface
//

template<class REAL> __inline__ __device__ REAL
KaiserBessel( const typename reald<REAL,2>::Type &u, const typename reald<REAL,2>::Type &matrix_size_os, REAL one_over_W, REAL beta )
{

  REAL phi_x = KaiserBessel( u.vec[0], matrix_size_os.vec[0], one_over_W, beta );
  REAL phi_y = KaiserBessel( u.vec[1], matrix_size_os.vec[1], one_over_W, beta );

  return phi_x*phi_y;
}

template<class REAL> __inline__ __device__ REAL
KaiserBessel( const typename reald<REAL,3>::Type &u, const typename reald<REAL,3>::Type &matrix_size_os, REAL one_over_W, REAL beta )
{

  REAL phi_x = KaiserBessel( u.vec[0], matrix_size_os.vec[0], one_over_W, beta );
  REAL phi_y = KaiserBessel( u.vec[1], matrix_size_os.vec[1], one_over_W, beta );
  REAL phi_z = KaiserBessel( u.vec[2], matrix_size_os.vec[2], one_over_W, beta );

  return phi_x*phi_y*phi_z;
}

template<class REAL> __inline__ __device__ REAL
KaiserBessel( const typename reald<REAL,4>::Type &u, const typename reald<REAL,4>::Type &matrix_size_os, REAL one_over_W, REAL beta )
{

  REAL phi_x = KaiserBessel( u.vec[0], matrix_size_os.vec[0], one_over_W, beta );
  REAL phi_y = KaiserBessel( u.vec[1], matrix_size_os.vec[1], one_over_W, beta );
  REAL phi_z = KaiserBessel( u.vec[2], matrix_size_os.vec[2], one_over_W, beta );
  REAL phi_w = KaiserBessel( u.vec[3], matrix_size_os.vec[3], one_over_W, beta );

  return phi_x*phi_y*phi_z*phi_w;
}
