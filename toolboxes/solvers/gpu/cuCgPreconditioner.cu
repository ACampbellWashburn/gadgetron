#include "hip/hip_runtime.h"
#include "cuCgPreconditioner.h"
#include "vector_td_utilities.h"

#include <sstream>

namespace Gadgetron{

  template<class T> __global__ void 
  weight_multiplication( T* in, T* out, T* weight, unsigned long elements )
  {
    unsigned long idx = blockIdx.x*blockDim.x+threadIdx.x;
    if (idx < elements) {
      const unsigned int frame_offset = blockIdx.y*elements;
      out[idx+frame_offset] = in[idx+frame_offset]*weight[idx];
    }
  }

  template <class T> void
  cuCgPreconditioner<T>::set_weights( boost::shared_ptr< cuNDArray<T> > w ) 
  {
    weights_ = w;
  }
  
  template <class T> void
  cuCgPreconditioner<T>::apply(cuNDArray<T>* in, cuNDArray<T>* out)
  {
    if( !weights_.get() ){
      throw std::runtime_error( "cuCGPreconWeight::apply: weights not set");
    }
 
    if ( !in || !out || in->get_number_of_elements() != out->get_number_of_elements()) {
      throw std::runtime_error("cuCGPreconWeight::apply: input and output dimensions mismatch");
    }

    if (in->get_number_of_elements() % weights_->get_number_of_elements()) {
      throw std::runtime_error( "cuCGPreconWeight::apply: input dimensions don't match weights dimensions" );
    }
  
    cuNDArray<T> *in_int, *out_int;
    in_int = in;
    out_int = out;
  
    unsigned int num_frames = in->get_number_of_elements() / weights_->get_number_of_elements();

    dim3 blockDim(256);
    dim3 gridDim((weights_->get_number_of_elements()+blockDim.x-1)/blockDim.x, num_frames );
    weight_multiplication<<< gridDim, blockDim >>>( in_int->get_data_ptr(), out_int->get_data_ptr(),
						    weights_->get_data_ptr(), weights_->get_number_of_elements());

    hipError_t err = hipGetLastError();
    if( err != hipSuccess ){
      std::stringstream ss;
      ss << "cuCGPreconWeight::apply: Unable to apply weights: " <<
	hipGetErrorString(err);
      throw std::runtime_error(ss.str());
    }  
  }

  //
  // Instantiation
  //

  template class EXPORTGPUSOLVERS cuCgPreconditioner<float>;
  template class EXPORTGPUSOLVERS cuCgPreconditioner<float_complext>;

  template class EXPORTGPUSOLVERS cuCgPreconditioner<double>;
  template class EXPORTGPUSOLVERS cuCgPreconditioner<double_complext>;
}
