#include "hip/hip_runtime.h"
#include "cuGPBBSolver.h"
#include "complext.h"

#define MAX_THREADS_PER_BLOCK 512

using namespace Gadgetron;
template <class T> __global__ void filter_kernel(T* x, T* g, int elements){
  typedef typename realType<T>::Type REAL;
  const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < elements){
    if ( real(x[idx]) <= REAL(0) && real(g[idx]) > 0) g[idx]=T(0);
  }
}

template <class T> void Gadgetron::cuGPBBSolver<T>::solver_non_negativity_filter(Gadgetron::cuNDArray<T>* x , Gadgetron::cuNDArray<T>* g){
  int elements = g->get_number_of_elements();

  int threadsPerBlock = std::min(elements,MAX_THREADS_PER_BLOCK);
  dim3 dimBlock( threadsPerBlock);
  int totalBlocksPerGrid = std::max(1,elements/MAX_THREADS_PER_BLOCK);
  dim3 dimGrid(totalBlocksPerGrid);

  filter_kernel<T><<<dimGrid,dimBlock>>>(x->get_data_ptr(),g->get_data_ptr(),elements);
}


template class EXPORTGPUSOLVERS Gadgetron::cuGPBBSolver<float>;
template class EXPORTGPUSOLVERS Gadgetron::cuGPBBSolver<double>;
