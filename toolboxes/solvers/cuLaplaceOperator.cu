#include "hip/hip_runtime.h"
#include "cuLaplaceOperator.h"
#include "vector_td.h"
#include "vector_td_utilities.h"
#include "check_CUDA.h"

// Template Power function
template<unsigned int i, unsigned int j>
struct Pow
{
  enum { Value = i*Pow<i,j-1>::Value};
};

template <unsigned int i>
struct Pow<i,1>
{
  enum { Value = i};
};

template<class REAL, class T, unsigned int D> __global__ void
laplace_kernel( typename intd<D>::Type dims, T *in, T *out )
{  
  const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
  if( idx < prod(dims) ){
    
    T val = T(0);
    typename intd<D>::Type coN;

    typename intd<D>::Type co = idx_to_co<D>(idx, dims);

    typename intd<D>::Type stride;

    for (int i = 0; i < D; i++) stride.vec[i]=0;

    for (int d1 = -1; d1 < 2; d1++){
      stride.vec[0] = d1;

      if (D > 1){
	for (int d2 = -1; d2 < 2; d2++){
	  stride.vec[1] = d2;
	  if (D > 2){
	    for (int d3 = -1; d3 < 2; d3++){
	      stride.vec[2] = d3;
	      coN = (co+dims+stride)%dims;
	      val -=  in[co_to_idx<D>(coN, dims)];
	    }
	  } else { 
	      coN = (co+dims+stride)%dims;
	      val += T(0) - in[co_to_idx<D>(coN, dims)];
	  }
	}
      } else {
	coN = (co+dims+stride)%dims;
	val -= in[co_to_idx<D>(coN, dims)];
      }
    }
    out[idx] = val+in[co_to_idx<D>(co, dims)]*((REAL) Pow<3,D>::Value);
  }
}

template< class T, unsigned int D> void
cuLaplaceOperator<T,D>::compute_laplace( cuNDArray<T> *in, cuNDArray<T> *out, bool accumulate )
{
  
  if( !in || !out || in->get_number_of_elements() != out->get_number_of_elements() ){
  	throw std::runtime_error("laplaceOperator::compute_laplace : array dimensions mismatch.");

  }
  
  typename uintd<D>::Type _dims = vector_to_uintd<D>( *(in->get_dimensions().get()) );
  typename intd<D>::Type dims;
  for( unsigned int i=0; i<D; i++ ){
    dims.vec[i] = (int)_dims.vec[i];
  }  
  


  _set_device();
  
  dim3 dimBlock( dims.vec[0] );
  dim3 dimGrid( 1, dims.vec[D-1] );
  
  for( unsigned int d=1; d<D-1; d++ )
    dimGrid.x *= dims.vec[d];
  
  // Invoke kernel
  laplace_kernel<typename realType<T>::type ,T,D><<< dimGrid, dimBlock >>> (dims, in->get_data_ptr(), out->get_data_ptr() );
  
  CHECK_FOR_CUDA_ERROR();

  _restore_device();


}

// Instantiations

template class EXPORTSOLVERS cuLaplaceOperator<float, 1>;
template class EXPORTSOLVERS cuLaplaceOperator<float, 2>;
template class EXPORTSOLVERS cuLaplaceOperator<float, 3>;

template class EXPORTSOLVERS cuLaplaceOperator<float_complext, 1>;
template class EXPORTSOLVERS cuLaplaceOperator<float_complext, 2>;
template class EXPORTSOLVERS cuLaplaceOperator<float_complext, 3>;


template class EXPORTSOLVERS cuLaplaceOperator<double, 1>;
template class EXPORTSOLVERS cuLaplaceOperator<double, 2>;
template class EXPORTSOLVERS cuLaplaceOperator<double, 3>;

template class EXPORTSOLVERS cuLaplaceOperator<double_complext, 1>;
template class EXPORTSOLVERS cuLaplaceOperator<double_complext, 2>;
template class EXPORTSOLVERS cuLaplaceOperator<double_complext, 3>;

