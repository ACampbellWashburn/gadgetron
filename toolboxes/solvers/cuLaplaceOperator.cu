#include "hip/hip_runtime.h"
#include "cuLaplaceOperator.h"
#include "vector_td.h"
#include "vector_td_utilities.h"
#include "check_CUDA.h"

// Template Power function
template<unsigned int i, unsigned int j>
struct Pow
{
  enum { Value = i*Pow<i,j-1>::Value};
};

template <unsigned int i>
struct Pow<i,1>
{
  enum { Value = i};
};

template<class REAL, class T, unsigned int D> __global__ void
laplace_kernel( typename intd<D>::Type dims, T *in, T *out )
{  
  const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
  if( idx < prod(dims) ){
    
    T val = T(0);
    typename intd<D>::Type coN;

    typename intd<D>::Type co = idx_to_co<D>(idx, dims);

    typename intd<D>::Type stride;

    for (int i = 0; i < D; i++) stride.vec[i]=0;

    for (int d1 = -1; d1 < 2; d1++){
      stride.vec[0] = d1;

      if (D > 1){
	for (int d2 = -1; d2 < 2; d2++){
	  stride.vec[1] = d2;
	  if (D > 2){
	    for (int d3 = -1; d3 < 2; d3++){
	      stride.vec[2] = d3;
	      coN = (co+dims+stride)%dims;
	      val -=  in[co_to_idx<D>(coN, dims)];
	    }
	  } else { 
	      coN = (co+dims+stride)%dims;
	      val += T(0) - in[co_to_idx<D>(coN, dims)];
	  }
	}
      } else {
	coN = (co+dims+stride)%dims;
	val -= in[co_to_idx<D>(coN, dims)];
      }
    }
    out[idx] = val+in[co_to_idx<D>(co, dims)]*((REAL) Pow<3,D>::Value);
  }
}

template< class REAL, class T, unsigned int D> int 
cuLaplaceOperator<REAL,T,D>::compute_laplace( cuNDArray<T> *in, cuNDArray<T> *out, bool accumulate )
{
  
  if( !in || !out || in->get_number_of_elements() != out->get_number_of_elements() ){
    std::cerr << std::endl << "laplaceOperator::compute_laplace : array dimensions mismatch." << std::endl;
    return -1;
  }
  
  typename uintd<D>::Type _dims = vector_to_uintd<D>( *(in->get_dimensions().get()) );
  typename intd<D>::Type dims;
  for( unsigned int i=0; i<D; i++ ){
    dims.vec[i] = (int)_dims.vec[i];
  }  
  
  if( D>3 ){
    std::cerr << std::endl << "partialDerivativeOperator::compute_laplace : internal error (only D<4 supported for now)." << std::endl;
    return -1;
  }

  _set_device();
  
  dim3 dimBlock( dims.vec[0] );
  dim3 dimGrid( 1, dims.vec[D-1] );
  
  for( unsigned int d=1; d<D-1; d++ )
    dimGrid.x *= dims.vec[d];
  
  // Invoke kernel
  laplace_kernel<REAL,T,D><<< dimGrid, dimBlock >>> (dims, in->get_data_ptr(), out->get_data_ptr() );
  
  CHECK_FOR_CUDA_ERROR();

  _restore_device();

  return 0;
}

// Instantiations

template class EXPORTSOLVERS cuLaplaceOperator<float, float, 1>;
template class EXPORTSOLVERS cuLaplaceOperator<float, float, 2>;
template class EXPORTSOLVERS cuLaplaceOperator<float, float, 3>;

template class EXPORTSOLVERS cuLaplaceOperator<float, float_complext, 1>;
template class EXPORTSOLVERS cuLaplaceOperator<float, float_complext, 2>;
template class EXPORTSOLVERS cuLaplaceOperator<float, float_complext, 3>;


template class EXPORTSOLVERS cuLaplaceOperator<double, double, 1>;
template class EXPORTSOLVERS cuLaplaceOperator<double, double, 2>;
template class EXPORTSOLVERS cuLaplaceOperator<double, double, 3>;

template class EXPORTSOLVERS cuLaplaceOperator<double, double_complext, 1>;
template class EXPORTSOLVERS cuLaplaceOperator<double, double_complext, 2>;
template class EXPORTSOLVERS cuLaplaceOperator<double, double_complext, 3>;

