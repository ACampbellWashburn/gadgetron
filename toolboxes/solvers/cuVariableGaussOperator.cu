#include "hip/hip_runtime.h"
#include "cuVariableGaussOperator.h"
#include "hip/hip_math_constants.h"
#include "check_CUDA.h"
#include "vector_td_utilities.h"

#define BLOCK_SIZE 512
template<class REAL, class T, unsigned int D> __global__ void
mult_M_kernel( typename intd<D>::Type dims, T *in, T *out,REAL *sigma, REAL *norm )
{  
  const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
  if( idx < prod(dims) ){
    
    __shared__ REAL shared[BLOCK_SIZE];
    __shared__ REAL sSigma[BLOCK_SIZE];
    __shared__ REAL sNorm[BLOCK_SIZE];

    T val = T(0);
    //REAL s = 1.0/(2.0*5.0*5.0);
    REAL s;

    typename intd<D>::Type co2;
    typename intd<D>::Type co = idx_to_co<D>(idx, dims);
    
    for (int k = 0; k < gridDim.x; k++){
      shared[threadIdx.x] = in[k*blockDim.x + threadIdx.x];
      sSigma[threadIdx.x] = sigma[k*blockDim.x + threadIdx.x];
      sNorm[threadIdx.x] = norm[k*blockDim.x + threadIdx.x];
      __syncthreads();

      for (int i = 0; i < blockDim.x; i++){
	s = sSigma[i];

	co2 = idx_to_co<D>(k*blockDim.x+i, dims)-co;
	val += shared[i]*sNorm[i]*exp(- ((REAL)dot<int,D>(co2,co2))*s*s*0.5);		  
      }
    }
    out[idx] = val;    
  }   
}

template<class REAL, class T, unsigned int D> __global__ void
norm_kernel( typename intd<D>::Type dims, REAL *sigma, REAL *out )
{
  const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
  if( idx < prod(dims) ){
    
    __shared__ REAL shared[BLOCK_SIZE];

    T val = T(0);
    REAL s = sigma[idx];

    typename intd<D>::Type co2;
    typename intd<D>::Type co = idx_to_co<D>(idx, dims);
    
    for (int k = 0; k < gridDim.x; k++){
      for (int i = 0; i < blockDim.x; i++){
	co2 = idx_to_co<D>(k*blockDim.x+i, dims)-co;
	val += exp(- ((REAL)dot<int,D>(co2,co2))*s*s*0.5);		  
      }
    }
    out[idx] = 1.0/val;    
  }    
}

template<class REAL, class T, unsigned int D> __global__ void
mult_MH_kernel( typename intd<D>::Type dims, T *in, T *out,REAL *sigma,REAL *norm )
{
  const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
  if( idx < prod(dims) ){
    
    __shared__ REAL shared[BLOCK_SIZE];

    T val = T(0);
    REAL s = sigma[idx];

    typename intd<D>::Type co2;
    typename intd<D>::Type co = idx_to_co<D>(idx, dims);
    
    for (int k = 0; k < gridDim.x; k++){
      shared[threadIdx.x] = in[k*blockDim.x + threadIdx.x];
      __syncthreads();

      for (int i = 0; i < blockDim.x; i++){
	co2 = idx_to_co<D>(k*blockDim.x+i, dims)-co;
	val += shared[i]*exp(- ((REAL)dot<int,D>(co2,co2))*s*s*0.5);		  
      }
    }
    out[idx] = val*norm[idx];    
  }    
}

template< class REAL, class T, unsigned int D> void
cuVariableGaussOperator<REAL,T,D>::set_sigma( cuNDArray<REAL> *sigma )
{
  _sigma = sigma;
  
  typename uintd<D>::Type _dims = vector_to_uintd<D>( *(_sigma->get_dimensions().get()) );
  typename intd<D>::Type dims;
  for( unsigned int i=0; i<D; i++ ){
    dims.vec[i] = (int)_dims.vec[i];
  }  

  dim3 dimBlock( BLOCK_SIZE );
  dim3 dimGrid( prod(dims)/BLOCK_SIZE );
  
  _norm = boost::shared_ptr<cuNDArray<REAL> >(new cuNDArray<REAL>);
  _norm->create(_sigma->get_dimensions().get());
  
  // Invoke kernel
  norm_kernel<REAL,T,D><<< dimGrid, dimBlock >>> (dims, sigma->get_data_ptr(), _norm->get_data_ptr() );
  
  CHECK_FOR_CUDA_ERROR();
}

template< class REAL, class T, unsigned int D> int
cuVariableGaussOperator<REAL,T,D>::mult_M( cuNDArray<T> *in, cuNDArray<T> *out, bool accumulate )
{
  if( !in || !out || in->get_number_of_elements() != out->get_number_of_elements() ){
    std::cerr << std::endl << "laplaceOperator::compute_laplace : array dimensions mismatch." << std::endl;
    return -1;
  }
  
  typename uintd<D>::Type _dims = vector_to_uintd<D>( *(in->get_dimensions().get()) );
  typename intd<D>::Type dims;
  for( unsigned int i=0; i<D; i++ ){
    dims.vec[i] = (int)_dims.vec[i];
  }  
  
  if( D>3 ){
    std::cerr << std::endl << "partialDerivativeOperator::compute_laplace : internal error (only D<4 supported for now)." << std::endl;
    return -1;
  }

  _set_device();
  
  dim3 dimBlock( BLOCK_SIZE );
  dim3 dimGrid( prod(dims)/BLOCK_SIZE );
    
  // Invoke kernel
  mult_M_kernel<REAL,T,D><<< dimGrid, dimBlock >>> (dims, in->get_data_ptr(), out->get_data_ptr(), _sigma->get_data_ptr() ,_norm->get_data_ptr() );
  
  CHECK_FOR_CUDA_ERROR();

  _restore_device();

  return 0;
}

template< class REAL, class T, unsigned int D> int
cuVariableGaussOperator<REAL,T,D>::mult_MH( cuNDArray<T> *in, cuNDArray<T> *out, bool accumulate)
{
  if( !in || !out || in->get_number_of_elements() != out->get_number_of_elements() ){
    std::cerr << std::endl << "laplaceOperator::compute_laplace : array dimensions mismatch." << std::endl;
    return -1;
  }
  
  typename uintd<D>::Type _dims = vector_to_uintd<D>( *(in->get_dimensions().get()) );
  typename intd<D>::Type dims;
  for( unsigned int i=0; i<D; i++ ){
    dims.vec[i] = (int)_dims.vec[i];
  }  
  
  if( D>3 ){
    std::cerr << std::endl << "partialDerivativeOperator::compute_laplace : internal error (only D<4 supported for now)." << std::endl;
    return -1;
  }

  _set_device();

  dim3 dimBlock( BLOCK_SIZE );
  dim3 dimGrid( prod(dims)/BLOCK_SIZE );
  
  // Invoke kernel

  mult_MH_kernel<REAL,T,D><<< dimGrid, dimBlock >>> (dims, in->get_data_ptr(), out->get_data_ptr(), _sigma->get_data_ptr() ,_norm->get_data_ptr());
  
  CHECK_FOR_CUDA_ERROR();

  _restore_device();
  
  return 0;
}

template< class REAL, class T, unsigned int D> int
cuVariableGaussOperator<REAL,T,D>::mult_MH_M( cuNDArray<T> *in, cuNDArray<T> *out, bool accumulate)
{
  cuNDArray<T> tmp;
  tmp.create(out->get_dimensions().get());

  int res = mult_M(in,&tmp);

  if (res < 0){
    return -1;
  }

  return mult_MH(&tmp,out,accumulate);
}

// Instantiations


template class EXPORTSOLVERS cuVariableGaussOperator<float, float, 1>;
template class EXPORTSOLVERS cuVariableGaussOperator<float, float, 2>;
template class EXPORTSOLVERS cuVariableGaussOperator<float, float, 3>;

/*
template class EXPORTSOLVERS cuVariableGaussOperator<float, float_complext::Type, 1>;
template class EXPORTSOLVERS cuVariableGaussOperator<float, float_complext::Type, 2>;
template class EXPORTSOLVERS cuVariableGaussOperator<float, float_complext::Type, 3>;
*/

template class EXPORTSOLVERS cuVariableGaussOperator<double, double, 1>;
template class EXPORTSOLVERS cuVariableGaussOperator<double, double, 2>;
template class EXPORTSOLVERS cuVariableGaussOperator<double, double, 3>;

/*
template class EXPORTSOLVERS cuVariableGaussOperator<double, double_complext::Type, 1>;
template class EXPORTSOLVERS cuVariableGaussOperator<double, double_complext::Type, 2>;
template class EXPORTSOLVERS cuVariableGaussOperator<double, double_complext::Type, 3>;
*/
