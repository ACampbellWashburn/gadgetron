#include "hip/hip_runtime.h"
#include "cuResampleOperator.h"

#include <thrust/host_vector.h>
#include <thrust/generate.h>
#include <thrust/pair.h>
#include <thrust/sort.h> 
#include <thrust/binary_search.h>
#include <thrust/iterator/counting_iterator.h>

using namespace thrust;

template<class REAL, class T, unsigned int D> bool 
cuResampleOperator<REAL,T,D>::setup_grid( dim3 *blockDim, dim3* gridDim, 
					  unsigned int number_of_elements, unsigned int num_batches )
{
  int device;
  hipDeviceProp_t deviceProp; 
  
  if( hipGetDevice( &device ) != hipSuccess) {
    std::cout << std::endl << "Error: unable to determine current device." << std::endl ;
    return false;
  }

  if( hipGetDeviceProperties( &deviceProp, device ) != hipSuccess) {
    std::cout << std::endl << "Error: unable to query device properties." << std::endl ;
    return false;
  }
  
  int max_blockdim = deviceProp.maxThreadsDim[0];
  int max_griddim  = deviceProp.maxGridSize[0];
  
  // For small arrays we keep the block dimension fairly small
  *blockDim = dim3(256);
  *gridDim = dim3((number_of_elements+blockDim->x-1)/blockDim->x, num_batches);

  // Extend block/grid dimensions for large arrays
  if( gridDim->x > max_griddim ){
    blockDim->x = max_blockdim;
    gridDim->x = (number_of_elements+blockDim->x-1)/blockDim->x;
  }

  if( gridDim->x > max_griddim ){
    gridDim->x = ((unsigned int)sqrt((float)number_of_elements)+blockDim->x-1)/blockDim->x;
    gridDim->y *= ((number_of_elements+blockDim->x*gridDim->x-1)/(blockDim->x*gridDim->x));
  }
   
  if( gridDim->x > max_griddim || gridDim->y > max_griddim )
    return false;
  else 
    return true;
}

template<class REAL, class T, unsigned int D> bool 
cuResampleOperator<REAL,T,D>::mult_MH_preprocess()
{
  this->preprocessed_ = false;
  
  // Check if a displacement field has been provided
  //
  
  if( !this->offsets_.get() ){
    std::cout << std::endl << "Error: cuResampleOperator :: mult_MH_preprocess : displacement field not set." << std::endl; 
    return false;
  }

  // Make a device vector wrap of the displacement field
  //

  std::vector<unsigned int> _dims_disp = *this->offsets_->get_dimensions(); _dims_disp.pop_back(); 
  unsigned int num_elements_disp = D;
  while(!_dims_disp.empty()){
    num_elements_disp *= _dims_disp.back();
    _dims_disp.pop_back();
  }
  
  device_vector<REAL> displacements
    ( device_pointer_cast<REAL>(this->offsets_->get_data_ptr()), 
      device_pointer_cast<REAL>(this->offsets_->get_data_ptr()+num_elements_disp) );
  
  // Make sort keys/values array from the deformation field
  //

  unsigned int num_elements_sort = num_elements_disp/D;
  
  this->lower_bounds_ = device_vector<unsigned int>(num_elements_sort);
  this->upper_bounds_ = device_vector<unsigned int>(num_elements_sort);
  
  this->indices_ = device_vector<unsigned int>(get_num_neighbors()*num_elements_sort);
  this->weights_ = device_vector<REAL>(get_num_neighbors()*num_elements_sort);

  device_vector<unsigned int> sort_keys = device_vector<unsigned int>
    (get_num_neighbors()*num_elements_sort);
  
  // Fill arrays
  //

  if( !write_sort_arrays(sort_keys)){
    std::cout << std::endl << "Error: cuResampleOperator :: mult_MH_preprocess : failed to write tuples" << std::endl; 
    return false;
  }
  
  // Make copy of sort_keys before the sort modifies it
  //

  device_vector<unsigned int> sort_keys_copy(sort_keys);
  
  // Sort (twice since we have two value arrays)
  //

  sort_by_key(sort_keys.begin(), sort_keys.end(), this->indices_.begin() );
  sort_by_key(sort_keys_copy.begin(), sort_keys_copy.end(), this->weights_.begin() );
  
  // Find start/end indices (buckets) in the two values arrays
  //
  
  counting_iterator<unsigned int> search_begin(0);
    
  lower_bound( sort_keys.begin(), sort_keys.end(), 
	       search_begin, search_begin + num_elements_sort, this->lower_bounds_.begin() );
  
  upper_bound( sort_keys.begin(), sort_keys.end(), 
	       search_begin, search_begin + num_elements_sort, this->upper_bounds_.begin() );
    
  this->preprocessed_ = true;
  return true;
}

template class EXPORTGPUREG cuResampleOperator<float,float,1>;
template class EXPORTGPUREG cuResampleOperator<float,float_complext,1>;

template class EXPORTGPUREG cuResampleOperator<float,float,2>;
template class EXPORTGPUREG cuResampleOperator<float,float_complext,2>;

template class EXPORTGPUREG cuResampleOperator<float,float,3>;
template class EXPORTGPUREG cuResampleOperator<float,float_complext,3>;

template class EXPORTGPUREG cuResampleOperator<float,float,4>;
template class EXPORTGPUREG cuResampleOperator<float,float_complext,4>;

template class EXPORTGPUREG cuResampleOperator<double,double,1>;
template class EXPORTGPUREG cuResampleOperator<double,double_complext,1>;

template class EXPORTGPUREG cuResampleOperator<double,double,2>;
template class EXPORTGPUREG cuResampleOperator<double,double_complext,2>;

template class EXPORTGPUREG cuResampleOperator<double,double,3>;
template class EXPORTGPUREG cuResampleOperator<double,double_complext,3>;

template class EXPORTGPUREG cuResampleOperator<double,double,4>;
template class EXPORTGPUREG cuResampleOperator<double,double_complext,4>;
