#include "hip/hip_runtime.h"
#include "cuCGPreconditioner.h"

//#include <hip/hip_complex.h>
#include "vector_td_utilities.h"

template<class T> __global__ void 
weight_multiplication( T* in, T* out, T* weight, unsigned long elements )
{
  unsigned long idx_in = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx_in < elements) {
    out[idx_in] = mul<T,T>(in[idx_in], weight[idx_in]);
  }
}

template <class T> int cuCGPrecondWeight<T>::apply(cuNDArray<T>* in, cuNDArray<T>* out)
{
  if (in->get_number_of_elements() != out->get_number_of_elements()) {
    std::cerr << "cuCGPreconWeight::apply : input and output dimensions mismatch" << std::endl;
    return -1;
  }

  if (in->get_number_of_elements() != weights_.get_number_of_elements()) {
    std::cerr << "cuCGPreconWeight::apply : input dimensions don't match weights dimensions" << std::endl;
    return -1;
  }

  dim3 blockDim(256,1,1);
  dim3 gridDim((unsigned int) ceil((double)in->get_number_of_elements()/blockDim.x), 1, 1 );
  weight_multiplication<<< gridDim, blockDim >>>( in->get_data_ptr(), out->get_data_ptr(),
						  weights_.get_data_ptr(), in->get_number_of_elements());

  hipError_t err = hipGetLastError();
  if( err != hipSuccess ){
    std::cerr << "cuCGPreconWeight::apply : Unable to apply weights: " << 
      hipGetErrorString(err) << std::endl;
    return -2;
  }
  
  return 0;
}


//
// Instantiation
//

template class cuCGPrecondWeight<float>;
//template class cuCGPrecondWeight<float2>;
template class cuCGPrecondWeight<float_complext::Type>;
