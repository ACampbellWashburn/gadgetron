#include "hip/hip_runtime.h"
#include "b1_map.h"
#include "vector_td_utilities.h"
#include "ndarray_vector_td_utilities.h"
#include "real_utilities.h"
#include "real_utilities_device.h"
#include "check_CUDA.h"
#include "cuNDFFT.h"

#include <iostream>

using namespace std;

const int kernel_width = 7;

template<class REAL, unsigned int D> void
smooth_correlation_matrices( cuNDArray<typename complext<REAL>::Type> *corrm, cuNDArray<typename complext<REAL>::Type> *corrm_smooth );

template<class REAL> __host__ 
boost::shared_ptr< cuNDArray<typename complext<REAL>::Type> > extract_csm( cuNDArray<typename complext<REAL>::Type> *corrm_in, unsigned int number_of_batches, unsigned int number_of_elements );

template<class REAL> __host__ 
void set_phase_reference( cuNDArray<typename complext<REAL>::Type> *csm, unsigned int number_of_batches, unsigned int number_of_elements );

//
// Main method
//

template<class REAL, unsigned int D> boost::shared_ptr< cuNDArray<typename complext<REAL>::Type> >
estimate_b1_map( cuNDArray<typename complext<REAL>::Type> *data_in )
{
  if( data_in->get_number_of_dimensions() < 2 ){
    cout << endl << "estimate_b1_map:: dimensionality mismatch." << endl; 
    return boost::shared_ptr< cuNDArray<typename complext<REAL>::Type > >();
  }

  if( data_in->get_number_of_dimensions()-1 != D ){
    cout << endl << "estimate_b1_map:: dimensionality mismatch." << endl; 
    return boost::shared_ptr< cuNDArray<typename complext<REAL>::Type > >();
  }

  vector<unsigned int> image_dims, dims_to_xform;
  unsigned int pixels_per_coil = 1;
  
  for( unsigned int i=0; i<D; i++ ){
    image_dims.push_back(data_in->get_size(i));
    dims_to_xform.push_back(i);
    pixels_per_coil *= data_in->get_size(i);
  }
  
  unsigned int ncoils = data_in->get_size(D);

  // Make a copy of input data
  cuNDArray<typename complext<REAL>::Type > *_data_out = new cuNDArray<typename complext<REAL>::Type>(*data_in);
  boost::shared_ptr< cuNDArray<typename complext<REAL>::Type> > data_out(_data_out);
  
  // Normalize by the RSS of the coils
  if( !cuNDA_rss_normalize<REAL>( data_out.get(), D ) ){
    cout << endl << "estimate_b1_map:: error in rss_normalize" << endl;
    return boost::shared_ptr< cuNDArray<typename complext<REAL>::Type> >();
  }
  
  // Now calculate the correlation matrices
  boost::shared_ptr<cuNDArray<typename complext<REAL>::Type> > corrm = cuNDA_correlation<typename complext<REAL>::Type>( data_out.get() );
  data_out.reset();
  
  // Smooth (onto copy of corrm)
  cuNDArray<typename complext<REAL>::Type > *_corrm_smooth = new cuNDArray<typename complext<REAL>::Type>();
  _corrm_smooth->create(corrm->get_dimensions().get());
  boost::shared_ptr<cuNDArray<typename complext<REAL>::Type> > corrm_smooth(_corrm_smooth);

  smooth_correlation_matrices<REAL,D>( corrm.get(), corrm_smooth.get() );
  corrm.reset();

  // Get the dominant eigenvector for each correlation matrix.
  boost::shared_ptr<cuNDArray<typename complext<REAL>::Type> > csm = extract_csm<REAL>( corrm_smooth.get(), ncoils, pixels_per_coil );
  corrm_smooth.reset();
  
  // Set phase according to reference (coil 0)
  set_phase_reference<REAL>( csm.get(), ncoils, pixels_per_coil );
  
  return csm;
}

// Smooth correlation matrices by box filter (1D)
template<class REAL> __global__ void
smooth_correlation_matrices_kernel( typename complext<REAL>::Type *corrm, typename complext<REAL>::Type *corrm_smooth, intd<1>::Type image_dims )
{
  const int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const int batch = blockIdx.y;

  const int num_image_elements = prod(image_dims);

  if( idx < num_image_elements ){
    
    const int co = idx;    
    const int x = co;
    
    const int size_x = image_dims.vec[0];
    
    const REAL scale = get_one<REAL>()/((REAL)kernel_width);
    
    typename complext<REAL>::Type result = get_zero<typename complext<REAL>::Type>();
    
    for (int kx = 0; kx < kernel_width; kx++) {
      
      if ((x-(kernel_width>>1)+kx) >= 0 &&
	  (x-(kernel_width>>1)+kx) < size_x)
	{	    
	  int source_offset = 
	    batch*num_image_elements +
	    (x-(kernel_width>>1)+kx);
	  
	  result += corrm[source_offset];
	}
    }
    corrm_smooth[batch*num_image_elements+idx] = scale*result;
  }
}

// Smooth correlation matrices by box filter (2D)
template<class REAL> __global__ void
smooth_correlation_matrices_kernel( typename complext<REAL>::Type *corrm, typename complext<REAL>::Type *corrm_smooth, intd<2>::Type image_dims )
{
  const int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const int batch = blockIdx.y;

  const int num_image_elements = prod(image_dims);

  if( idx < num_image_elements ){
    
    const intd2::Type co = idx_to_co<2>(idx, image_dims);
    
    const int x = co.vec[0];
    const int y = co.vec[1];
    
    const int size_x = image_dims.vec[0];
    const int size_y = image_dims.vec[1];
    
    const int half_width = kernel_width>>1;

    const int yminus = y-half_width;
    const int xminus = x-half_width;
    const int yplus = y+half_width;
    const int xplus = x+half_width;

    const REAL scale = get_one<REAL>()/((REAL)(kernel_width*kernel_width));
    
    typename complext<REAL>::Type result = get_zero<typename complext<REAL>::Type>();
   
    if( (yminus >=0) ){
      if( yplus < size_y ){
	if( xminus >= 0 ){
	  if( xplus < size_x ){

#pragma unroll
	    for (int ky = 0; ky < kernel_width; ky++){
#pragma unroll
	      for (int kx = 0; kx < kernel_width; kx++) {
		
		int cy = yminus+ky;
		int cx = xminus+kx;
		
		int source_offset = batch*num_image_elements + cy*size_x + cx;
		result += corrm[source_offset];
	      }
	    }
	  }
	}
      }
    }
    corrm_smooth[batch*num_image_elements+idx] = scale*result;
  }
}

// Smooth correlation matrices by box filter (3D)
template<class REAL> __global__ void
smooth_correlation_matrices_kernel( typename complext<REAL>::Type *corrm, typename complext<REAL>::Type *corrm_smooth, intd<3>::Type image_dims )
{
  const int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const int batch = blockIdx.y;

  const int num_image_elements = prod(image_dims);

  if( idx < num_image_elements ){
    
    const intd3::Type co = idx_to_co<3>(idx, image_dims);
    
    const int x = co.vec[0];
    const int y = co.vec[1];
    const int z = co.vec[2];
    
    const int size_x = image_dims.vec[0];
    const int size_y = image_dims.vec[1];
    const int size_z = image_dims.vec[2];
    
    const REAL scale = get_one<REAL>()/((REAL)(kernel_width*kernel_width*kernel_width));
    
    typename complext<REAL>::Type result = get_zero<typename complext<REAL>::Type>();
    
    for (int kz = 0; kz < kernel_width; kz++) {
      for (int ky = 0; ky < kernel_width; ky++) {
	for (int kx = 0; kx < kernel_width; kx++) {
	
	  if ((z-(kernel_width>>1)+kz) >= 0 &&
	      (z-(kernel_width>>1)+kz) < size_z &&
	      (y-(kernel_width>>1)+ky) >= 0 &&
	      (y-(kernel_width>>1)+ky) < size_y &&
	      (x-(kernel_width>>1)+kx) >= 0 &&
	      (x-(kernel_width>>1)+kx) < size_x) 
	    {	    
	      int source_offset = 
		batch*num_image_elements +
		(z-(kernel_width>>1)+kz)*size_x*size_y +
		(y-(kernel_width>>1)+ky)*size_x +
		(x-(kernel_width>>1)+kx);
	    
	      result += corrm[source_offset];
	    }
	}
      }
    }
    corrm_smooth[batch*num_image_elements+idx] = scale*result;
  }
}

// Smooth correlation matrices by box filter (3D)
template<class REAL> __global__ void
smooth_correlation_matrices_kernel( typename complext<REAL>::Type *corrm, typename complext<REAL>::Type *corrm_smooth, intd<4>::Type image_dims )
{
  const int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const int batch = blockIdx.y;

  const int num_image_elements = prod(image_dims);

  if( idx < num_image_elements ){
    
    const intd4::Type co = idx_to_co<4>(idx, image_dims);
    
    const int x = co.vec[0];
    const int y = co.vec[1];
    const int z = co.vec[2];
    const int w = co.vec[3];
    
    const int size_x = image_dims.vec[0];
    const int size_y = image_dims.vec[1];
    const int size_z = image_dims.vec[2];    
    const int size_w = image_dims.vec[3];
    
    const REAL scale = get_one<REAL>()/((REAL)(kernel_width*kernel_width*kernel_width*kernel_width));
    
    typename complext<REAL>::Type result = get_zero<typename complext<REAL>::Type>();
    
    for (int kw = 0; kw < kernel_width; kw++) {
      for (int kz = 0; kz < kernel_width; kz++) {
	for (int ky = 0; ky < kernel_width; ky++) {
	  for (int kx = 0; kx < kernel_width; kx++) {
	
	    if ((w-(kernel_width>>1)+kw) >= 0 &&
		(w-(kernel_width>>1)+kw) < size_w &&
		(z-(kernel_width>>1)+kz) >= 0 &&
		(z-(kernel_width>>1)+kz) < size_z &&
		(y-(kernel_width>>1)+ky) >= 0 &&
		(y-(kernel_width>>1)+ky) < size_y &&
		(x-(kernel_width>>1)+kx) >= 0 &&
		(x-(kernel_width>>1)+kx) < size_x) 
	      {	    
		int source_offset = 
		  batch*num_image_elements +
		  (w-(kernel_width>>1)+kw)*size_x*size_y*size_z +
		  (z-(kernel_width>>1)+kz)*size_x*size_y +
		  (y-(kernel_width>>1)+ky)*size_x +
		  (x-(kernel_width>>1)+kx);
	    
		result += corrm[source_offset];
	      }
	  }
	}
      }
    }
    corrm_smooth[batch*num_image_elements+idx] = scale*result;
  }
}

// Smooth correlation matrices border by box filter (2D)
template<class REAL> __global__ void
smooth_correlation_matrices_border_kernel( typename complext<REAL>::Type *corrm, typename complext<REAL>::Type *corrm_smooth, intd<2>::Type image_dims, unsigned int number_of_border_threads )
{
  const int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const int batch = blockIdx.y;

  const int num_image_elements = prod(image_dims);

  if( idx < number_of_border_threads ){
    
    intd<2>::Type co;
    const int half_width = kernel_width>>1;

    co.vec[1] = idx/image_dims.vec[0];
    co.vec[1] = min(co.vec[1], half_width );
    
    if( co.vec[1] == half_width ){
      int new_idx = idx-half_width*image_dims.vec[0];
      int num_skips = new_idx/half_width;
      int rows_offset = min(num_skips>>1, image_dims.vec[1]-(half_width<<1) );
      co.vec[1] += rows_offset;

      if( co.vec[1] == (half_width + image_dims.vec[1]-(half_width<<1)) ){
	new_idx -= ((image_dims.vec[1]-(half_width<<1))*(half_width<<1));
        co.vec[1] += (new_idx / image_dims.vec[0]);
	co.vec[0] = (new_idx % image_dims.vec[0]);
      }
      else{
	co.vec[0] = (num_skips%2)*(image_dims.vec[0]-half_width) + (new_idx%half_width);
      }
    }
    else{
      co.vec[0] = idx%image_dims.vec[0];
    }
    
    const int x = co.vec[0];
    const int y = co.vec[1];
    
    const int size_x = image_dims.vec[0];
    const int size_y = image_dims.vec[1];
    
    const int yminus = y-half_width;
    const int xminus = x-half_width;

    const REAL scale = get_one<REAL>()/((REAL)(kernel_width*kernel_width));
    
    typename complext<REAL>::Type result = get_zero<typename complext<REAL>::Type>();
 
#pragma unroll
    for (int ky = 0; ky < kernel_width; ky++) {
#pragma unroll
      for (int kx = 0; kx < kernel_width; kx++) {
	
	if( (yminus+ky >=0) ){
	  if( yminus+ky < size_y ){
	    if( xminus+kx >= 0 ){
	      if( xminus+kx < size_x ){
		
		int source_offset = 
		  batch*num_image_elements +
		  (yminus+ky)*size_x +
		  (xminus+kx);
		
		result += corrm[source_offset];
	      }
	    }
	  }
	}
      }
    }
    corrm_smooth[batch*num_image_elements+co_to_idx<2>(co,image_dims)] = scale*result;  
  }
}

template<class REAL, unsigned int D> void
smooth_correlation_matrices( cuNDArray<typename complext<REAL>::Type> *corrm, cuNDArray<typename complext<REAL>::Type> *corrm_smooth )
{
  typename intd<D>::Type image_dims;

  for( unsigned int i=0; i<D; i++ ){
    image_dims.vec[i] = corrm->get_size(i);
  }
  
  unsigned int number_of_batches = 1;
  
  for( unsigned int i=D; i<corrm->get_number_of_dimensions(); i++ ){
    number_of_batches *= corrm->get_size(i);
  }
  
  int device; hipGetDevice( &device );
  hipDeviceProp_t deviceProp; hipGetDeviceProperties( &deviceProp, device );

  dim3 blockDim(deviceProp.maxThreadsPerBlock);
  dim3 gridDim((unsigned int) ceil((double)prod(image_dims)/blockDim.x), number_of_batches);

  smooth_correlation_matrices_kernel<REAL><<<gridDim, blockDim>>>
    ( corrm->get_data_ptr(), corrm_smooth->get_data_ptr(), image_dims );
  
  CHECK_FOR_CUDA_ERROR();

  unsigned int number_of_border_threads = ((kernel_width>>1)<<1)*(sum(image_dims)-((kernel_width>>1)<<1));
  blockDim = dim3(128);
  gridDim = dim3((unsigned int) ceil((double)number_of_border_threads/blockDim.x), number_of_batches);
  
  smooth_correlation_matrices_border_kernel<REAL><<<gridDim, blockDim>>>
    ( corrm->get_data_ptr(), corrm_smooth->get_data_ptr(), image_dims, number_of_border_threads );

  CHECK_FOR_CUDA_ERROR();
}

extern __shared__ char shared_mem[];

// Extract CSM
template<class REAL> __global__ void
extract_csm_kernel( typename complext<REAL>::Type *corrm, typename complext<REAL>::Type *csm, unsigned int num_batches, unsigned int num_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int i = threadIdx.x;
  
  if( idx < num_elements ){    
    
    // Get the dominant eigenvector for each correlation matrix.
    // Copying Peter Kellman's approach we use the power method:
    //  b_k+1 = A*b_k / ||A*b_k||
    
    typename complext<REAL>::Type *data_out = (typename complext<REAL>::Type*) shared_mem;
    typename complext<REAL>::Type *tmp_v = &(((typename complext<REAL>::Type*) shared_mem)[num_batches*blockDim.x]);
    
    const unsigned int iterations = 2;
    
    for( unsigned int c=0; c<num_batches; c++){
      data_out[c*blockDim.x+i] = get_one<typename complext<REAL>::Type >();
    }
    
    for( unsigned int it=0; it<iterations; it++ ){
      
      for( unsigned int c=0; c<num_batches; c++){
	tmp_v[c*blockDim.x+i] = get_zero<typename complext<REAL>::Type >();
      }
      
      for( unsigned j=0; j<num_batches; j++){
	for( unsigned int k=0; k<num_batches; k++){
	  tmp_v[j*blockDim.x+i] += corrm[(k*num_batches+j)*num_elements+idx]*data_out[k*blockDim.x+i];
	}
      }
      
      REAL tmp = get_zero<REAL>();
      
      for (unsigned int c=0; c<num_batches; c++){
	tmp += norm_squared(tmp_v[c*blockDim.x+i]);
      }
      
      tmp = sqrt(tmp);
      tmp = reciprocal(tmp);
      
      for (unsigned int c=0; c<num_batches; c++){
	typename complext<REAL>::Type res = tmp*tmp_v[c*blockDim.x+i];
	data_out[c*blockDim.x+i] = res;
      }
    }
    
    for (unsigned int c=0; c<num_batches; c++){
      csm[c*num_elements+idx] = data_out[c*blockDim.x+i];
    }
  }
}

// Extract CSM
template<class REAL> __global__ void
extract_csm_kernel( typename complext<REAL>::Type *corrm, typename complext<REAL>::Type *csm, unsigned int num_batches, unsigned int num_elements, typename complext<REAL>::Type *tmp_v )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( idx < num_elements ){    
    
    // Get the dominant eigenvector for each correlation matrix.
    // Copying Peter Kellman's approach we use the power method:
    //  b_k+1 = A*b_k / ||A*b_k||
    
    const unsigned int iterations = 2;

    for( unsigned int c=0; c<num_batches; c++){
      csm[c*num_elements+idx] = get_one<typename complext<REAL>::Type >();
    }
    
    for( unsigned int it=0; it<iterations; it++ ){

      for( unsigned int c=0; c<num_batches; c++){
	tmp_v[c*num_elements+idx] = get_zero<typename complext<REAL>::Type >();
      }
      
      for( unsigned j=0; j<num_batches; j++){
	for( unsigned int k=0; k<num_batches; k++){
	  typedef typename complext<REAL>::Type T;
	  tmp_v[j*num_elements+idx] += mul<T>(corrm[(k*num_batches+j)*num_elements+idx],csm[k*num_elements+idx]);
	}
      }

      REAL tmp = get_zero<REAL>();
      
      for (unsigned int c=0; c<num_batches; c++){
	tmp += norm_squared<REAL>(tmp_v[c*num_elements+idx]);
      }
      
      tmp = sqrt(tmp);
      tmp = reciprocal(tmp);
      
      for (unsigned int c=0; c<num_batches; c++){
	typename complext<REAL>::Type res = tmp*tmp_v[c*num_elements+idx];
	csm[c*num_elements+idx] = res;
      }
    }
  }
}

// Extract CSM
template<class REAL> __host__ 
boost::shared_ptr<cuNDArray<typename complext<REAL>::Type> > extract_csm(cuNDArray<typename complext<REAL>::Type> *corrm_in, unsigned int number_of_batches, unsigned int number_of_elements )
{
  vector<unsigned int> image_dims;

  for( unsigned int i=0; i<corrm_in->get_number_of_dimensions()-1; i++ ){
    image_dims.push_back(corrm_in->get_size(i));
  }
  
  // Allocate output
  cuNDArray<typename complext<REAL>::Type> *out = new cuNDArray<typename complext<REAL>::Type>; out->create(&image_dims);

  dim3 blockDim(256);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  /*  
      if( out != 0x0 )
      extract_csm_kernel<REAL><<< gridDim, blockDim, number_of_batches*blockDim.x*2*sizeof(typename complext<REAL>::Type) >>>
      ( corrm_in->get_data_ptr(), out->get_data_ptr(), number_of_batches, number_of_elements );
  */

  // Temporary buffer. TODO: use shared memory
  cuNDArray<typename complext<REAL>::Type> *tmp_v = new cuNDArray<typename complext<REAL>::Type>; tmp_v->create(&image_dims);

  if( out != 0x0 && tmp_v != 0x0 )
    extract_csm_kernel<REAL><<< gridDim, blockDim >>>
      ( corrm_in->get_data_ptr(), out->get_data_ptr(), number_of_batches, number_of_elements, tmp_v->get_data_ptr() );

  CHECK_FOR_CUDA_ERROR();
  
  delete tmp_v;
  return boost::shared_ptr<cuNDArray<typename complext<REAL>::Type> >(out);
}

// Set refence phase
template<class REAL> __global__ void
set_phase_reference_kernel( typename complext<REAL>::Type *csm, unsigned int num_batches, unsigned int num_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( idx < num_elements ){
    REAL angle = arg<REAL>(csm[idx]); //Phase of the first coil
    REAL sin_a, cos_a; gad_sincos( angle, &sin_a, &cos_a );

    typename complext<REAL>::Type tmp;
    tmp.vec[0] = cos_a; tmp.vec[1] = sin_a;
    tmp = conj<typename complext<REAL>::Type>(tmp);

    for( unsigned int c=0; c<num_batches; c++ ){
      typename complext<REAL>::Type val = csm[c*num_elements+idx];
      typedef typename complext<REAL>::Type T;
      val = mul<T>( val, tmp );
      csm[c*num_elements+idx] = val;
    }
  }
}
  
// Set reference phase
template<class REAL> __host__ 
void set_phase_reference(cuNDArray<typename complext<REAL>::Type> *csm, unsigned int number_of_batches, unsigned int number_of_elements )
{
  dim3 blockDim(128);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));
  
  set_phase_reference_kernel<REAL><<< gridDim, blockDim >>>( csm->get_data_ptr(), number_of_batches, number_of_elements );
  
  CHECK_FOR_CUDA_ERROR();
}

//
// Template instantiation
//

//template EXPORTGPUPMRI boost::shared_ptr< cuNDArray<typename complext<float>::Type > > estimate_b1_map<float,1>(cuNDArray<typename complext<float>::Type >*);
template EXPORTGPUPMRI boost::shared_ptr< cuNDArray<typename complext<float>::Type > > estimate_b1_map<float,2>(cuNDArray<typename complext<float>::Type >*);
//template boost::shared_ptr< cuNDArray<typename complext<float>::Type > > estimate_b1_map<float,3>(cuNDArray<typename complext<float>::Type >*);
//template boost::shared_ptr< cuNDArray<typename complext<float>::Type > > estimate_b1_map<float,4>(cuNDArray<typename complext<float>::Type >*);

//template EXPORTGPUPMRI boost::shared_ptr< cuNDArray<typename complext<double>::Type > > estimate_b1_map<double,1>(cuNDArray<typename complext<double>::Type >*);
template EXPORTGPUPMRI boost::shared_ptr< cuNDArray<typename complext<double>::Type > > estimate_b1_map<double,2>(cuNDArray<typename complext<double>::Type >*);
//template EXPORTGPUPMRI boost::shared_ptr< cuNDArray<typename complext<double>::Type > > estimate_b1_map<double,3>(cuNDArray<typename complext<double>::Type >*);
//template EXPORTGPUPMRI boost::shared_ptr< cuNDArray<typename complext<double>::Type > > estimate_b1_map<double,4>(cuNDArray<typename complext<double>::Type >*);
