#include "hip/hip_runtime.h"
#include "cuCartesianSenseOperator.h"
#include "cuNDFFT.h"
#include "ndarray_vector_td_utilities.h"
#include <sstream>

template<class REAL> __global__ void 
sample_array_kernel( complext<REAL> *in, complext<REAL> *out,
		     unsigned int *idx, 
		     unsigned long image_elements,
		     unsigned long int samples,
		     unsigned int coils )
{
  unsigned long idx_in = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx_in < samples) {
    for (unsigned int i = 0; i < coils; i++) {
      out[idx_in + i*samples].vec[0] += in[idx[idx_in] + i*image_elements].vec[0];
      out[idx_in + i*samples].vec[1] += in[idx[idx_in] + i*image_elements].vec[1];
    }
  }
}

template<class REAL> __global__ void 
insert_samples_kernel( complext<REAL> *in, complext<REAL> *out,
				       unsigned int *idx, 
				       unsigned long image_elements,
				       unsigned long int samples,
				       unsigned int coils )
{
  unsigned long idx_in = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx_in < samples) {
    for (unsigned int i = 0; i < coils; i++) {
      out[idx[idx_in] + i*image_elements].vec[0] += in[idx_in + i*samples].vec[0];
      out[idx[idx_in] + i*image_elements].vec[1] += in[idx_in + i*samples].vec[1];
    }
  }
}

template<class REAL, unsigned int D> void
cuCartesianSenseOperator<REAL,D>::mult_M( cuNDArray<_complext> *in, cuNDArray<_complext> *out, bool accumulate )
{

  int ret = this->_set_device();
  if( ret<0 ){
    throw cuda_error("cuCartesianSenseOperator::mult_M: unable to set device");

  }
  
  if (!(in->dimensions_equal(this->get_domain_dimensions().get())) || !(out->dimensions_equal(this->get_codomain_dimensions().get())) ) {

    throw std::runtime_error("cuCartesianSenseOperator::mult_M dimensions mismatch");

  }


  std::vector<unsigned int> full_dimensions = *this->get_domain_dimensions();
  full_dimensions.push_back(this->ncoils_);
  cuNDArray<_complext> tmp(&full_dimensions);



  mult_csm(in,&tmp);

  cuNDFFT<_complext> ft;
  std::vector<unsigned int> ft_dims;
  for (unsigned int i = 0; i < this->get_domain_dimensions()->size(); i++) {
    ft_dims.push_back(i);
  }

  ft.fft(&tmp, &ft_dims);

  if (!accumulate) 
  	out->clear();


  dim3 blockDim(512,1,1);
  dim3 gridDim((unsigned int) ceil((double)idx_->get_number_of_elements()/blockDim.x), 1, 1 );
  sample_array_kernel<REAL><<< gridDim, blockDim >>>( tmp.get_data_ptr(), out->get_data_ptr(), idx_->get_data_ptr(),
						      in->get_number_of_elements(), idx_->get_number_of_elements(), this->ncoils_);
  hipError_t err = hipGetLastError();
  if( err != hipSuccess ){
     std::stringstream ss;
     ss <<"cuCartesianSenseOperator::mult_M : Unable to sample data: " <<
      hipGetErrorString(err);
     throw cuda_error(ss.str());

  }

  this->_restore_device();
  
}

template<class REAL, unsigned int D> void
cuCartesianSenseOperator<REAL,D>::mult_MH(cuNDArray<_complext> *in, cuNDArray<_complext> *out, bool accumulate)
{
  this->_set_device();


  if (!(out->dimensions_equal(this->get_domain_dimensions().get())) || 
      !(in->dimensions_equal(this->get_codomain_dimensions().get())) ) {
    throw std::runtime_error( "cuCartesianSenseOperator::mult_MH dimensions mismatch");

  }

  std::vector<unsigned int> tmp_dimensions = *this->get_domain_dimensions();
  tmp_dimensions.push_back(this->ncoils_);

  cuNDArray<_complext> tmp(&tmp_dimensions);

  tmp.clear();

  dim3 blockDim(512,1,1);
  dim3 gridDim((unsigned int) ceil((double)idx_->get_number_of_elements()/blockDim.x), 1, 1 );
  insert_samples_kernel<REAL><<< gridDim, blockDim >>>( in->get_data_ptr(), tmp.get_data_ptr(),
							idx_->get_data_ptr(),out->get_number_of_elements(),
							idx_->get_number_of_elements(), this->ncoils_);
  
  hipError_t err = hipGetLastError();
  if( err != hipSuccess ){
  	std::stringstream ss;
    ss << "cuCartesianSenseOperator::mult_EM : Unable to insert samples into array: " <<
      hipGetErrorString(err);
    throw cuda_error(ss.str());
  }

  cuNDFFT<_complext> ft;
  std::vector<unsigned int> ft_dims;
  for (unsigned int i = 0; i < this->get_domain_dimensions()->size(); i++) {
    ft_dims.push_back(i);
  }

  ft.ifft(&tmp, &ft_dims);

  if (!accumulate) 
    out->clear();
  
  mult_csm_conj_sum(&tmp,out);

  this->_restore_device();

}

//
// Instantiations
//

template class EXPORTGPUPMRI cuCartesianSenseOperator<float,2>;
template class EXPORTGPUPMRI cuCartesianSenseOperator<float,3>;
template class EXPORTGPUPMRI cuCartesianSenseOperator<float,4>;

template class EXPORTGPUPMRI cuCartesianSenseOperator<double,2>;
template class EXPORTGPUPMRI cuCartesianSenseOperator<double,3>;
template class EXPORTGPUPMRI cuCartesianSenseOperator<double,4>;

