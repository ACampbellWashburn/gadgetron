#include "hip/hip_runtime.h"
#include "cuTvOperator.h"
#include "cuNDArray_operators.h"
#include "cuNDArray_elemwise.h"
#include "vector_td_utilities.h"
#include "complext.h"
#include <iostream>
#include "check_CUDA.h"
#include "cudaDeviceManager.h"

using namespace Gadgetron;

template<class REAL, class T, unsigned int D> static inline  __device__ REAL gradient(const T* in, const vector_td<int,D>& dims, vector_td<int,D>& co)
{
  REAL grad = REAL(0);
  T xi = in[co_to_idx<D>((co+dims)%dims,dims)];
  for (int i = 0; i < D; i++){
    co[i]+=1;
    T dt = in[co_to_idx<D>((co+dims)%dims,dims)];
    grad += norm(xi-dt);
    co[i]-=1;
  }
  return sqrt(grad);
}


template<class REAL, class T, unsigned int D> static __global__ void tvGradient_kernel(const T* in, T* out, const vector_td<int,D> dims,REAL limit,REAL weight)
{
  const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
  if( idx < prod(dims) ){
    T xi = in[idx];
    T result=T(0);

    vector_td<int,D> co = idx_to_co<D>(idx, dims);

    REAL grad = gradient<REAL,T,D>(in,dims,co);

    if (grad > limit) {
      result += REAL(D)*xi/grad;
      for (int i = 0; i < D; i++){
	co[i]+=1;
	result -= in[co_to_idx<D>((co+dims)%dims,dims)]/grad;
	co[i]-=1;
      }
    }

    for (int i = 0; i < D; i++){
      co[i]-=1;
      grad = gradient<REAL,T,D>(in,dims,co);
      if (grad > limit) {
	result +=(xi-in[co_to_idx<D>((co+dims)%dims,dims)])/grad;
      }
      co[i]+=1;
    }
    out[idx] += result*weight;
  }
}





template<class T, unsigned int D> void cuTvOperator<T,D>::gradient (cuNDArray<T> * in,cuNDArray<T> * out, bool accumulate)
{
  if (!accumulate) 
    clear(out);

  const typename intd<D>::Type dims = to_intd( from_std_vector<size_t,D>(*(in->get_dimensions())));
  int elements = in->get_number_of_elements();

  int threadsPerBlock =std::min(prod(dims),cudaDeviceManager::Instance()->max_blockdim());
  dim3 dimBlock( threadsPerBlock);
  int totalBlocksPerGrid = std::max(1,prod(dims)/cudaDeviceManager::Instance()->max_blockdim());
  dim3 dimGrid(totalBlocksPerGrid);

  for (int i =0; i < (elements/prod(dims)); i++){
    tvGradient_kernel<<<dimGrid,dimBlock>>>(in->get_data_ptr()+i*prod(dims),out->get_data_ptr()+i*prod(dims),dims,limit_,this->weight_);
  }
  
  hipDeviceSynchronize();
  CHECK_FOR_CUDA_ERROR();
}

template<class REAL, class T, unsigned int D> static __global__ void tvMagnitude_kernel(const T* in,T* out,const vector_td<int,D> dims,REAL limit,REAL weight)
{
  const int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x + threadIdx.x;
  if( idx < prod(dims) ){
    vector_td<int,D> co = idx_to_co<D>(idx, dims);
    REAL grad = gradient<REAL,T,D>(in,dims,co);
    out[idx] = grad*weight;
  }
}


template<class T, unsigned int D> typename realType<T>::Type cuTvOperator<T,D>::magnitude (cuNDArray<T> * in)
{
  cuNDArray<T> out(in->get_dimensions());
  const typename intd<D>::Type dims = to_intd( from_std_vector<size_t,D>(*(in->get_dimensions())));
  int elements = in->get_number_of_elements();

  int threadsPerBlock =std::min(prod(dims),cudaDeviceManager::Instance()->max_blockdim());
  dim3 dimBlock( threadsPerBlock);
  int totalBlocksPerGrid = std::max(1,prod(dims)/cudaDeviceManager::Instance()->max_blockdim());
  dim3 dimGrid(totalBlocksPerGrid);

  for (int i =0; i < (elements/prod(dims)); i++){
    tvMagnitude_kernel<<<dimGrid,dimBlock>>>(in->get_data_ptr()+i*prod(dims),out.get_data_ptr()+i*prod(dims),dims,limit_,this->weight_);
  }

  hipDeviceSynchronize();
  CHECK_FOR_CUDA_ERROR();
  return asum(&out);
}

template class EXPORTGPUOPERATORS cuTvOperator<float,1>;
template class EXPORTGPUOPERATORS cuTvOperator<float,2>;
template class EXPORTGPUOPERATORS cuTvOperator<float,3>;
template class EXPORTGPUOPERATORS cuTvOperator<float,4>;

template class EXPORTGPUOPERATORS cuTvOperator<double,1>;
template class EXPORTGPUOPERATORS cuTvOperator<double,2>;
template class EXPORTGPUOPERATORS cuTvOperator<double,3>;
template class EXPORTGPUOPERATORS cuTvOperator<double,4>;

template class EXPORTGPUOPERATORS cuTvOperator<float_complext,1>;
template class EXPORTGPUOPERATORS cuTvOperator<float_complext,2>;
template class EXPORTGPUOPERATORS cuTvOperator<float_complext,3>;
template class EXPORTGPUOPERATORS cuTvOperator<float_complext,4>;

template class EXPORTGPUOPERATORS cuTvOperator<double_complext,1>;
template class EXPORTGPUOPERATORS cuTvOperator<double_complext,2>;
template class EXPORTGPUOPERATORS cuTvOperator<double_complext,3>;
template class EXPORTGPUOPERATORS cuTvOperator<double_complext,4>;
