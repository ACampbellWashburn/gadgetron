#include "hip/hip_runtime.h"
#include "cuNDArray.h"
#include "vector_td.h"

template <class T> 
__global__ void cuNDArray_permute_kernel(T* in, T* out, 
					 unsigned int ndim,
					 unsigned int* dims,
					 unsigned int* strides_out,
					 unsigned long int elements,
					 int shift_mode)
{
  unsigned long idx_in = blockIdx.x*blockDim.x+threadIdx.x;
  unsigned long idx_out = 0;

  unsigned long idx_in_tmp = idx_in;
  if (idx_in < elements) {

    unsigned int cur_index;
    for (unsigned int i = 0; i < ndim; i++) {
      unsigned long idx_in_remainder = idx_in_tmp / dims[i];
      cur_index = idx_in_tmp-(idx_in_remainder*dims[i]); //cur_index = idx_in_tmp%dims[i];
      if (shift_mode < 0) { //IFFTSHIFT
	idx_out += ((cur_index+(dims[i]>>1))%dims[i])*strides_out[i];
      } else if (shift_mode > 0) { //FFTSHIFT
	idx_out += ((cur_index+((dims[i]+1)>>1))%dims[i])*strides_out[i];
      } else {
	idx_out += cur_index*strides_out[i];
      }
      idx_in_tmp = idx_in_remainder;
    }

    out[idx_in] = in[idx_out];

  }

}


template <class T> int cuNDArray_permute(cuNDArray<T>* in,
 					 cuNDArray<T>* out,
					 std::vector<unsigned int> order,
					 int shift_mode)
{
  hipError_t err;

  T* in_ptr = in->data_;
  T* out_ptr = 0;

  if (out) {
    out_ptr = out->data_;
  } else {
    if (hipMalloc((void**) &out_ptr, in->elements_*sizeof(T)) != hipSuccess) {
      std::cerr << "cuNDArray_permute : Error allocating CUDA memory" << std::endl;
      out_ptr = 0;
      return -1;
    }
  }

  unsigned int* dims        = new unsigned int[in->dimensions_.size()];
  unsigned int* strides_out = new unsigned int[in->dimensions_.size()];
  if (!dims || !strides_out) {
    std::cerr << "cuNDArray_permute: failed to allocate temporary storage for arrays" << std::endl;
    return -1;
  }

  for (unsigned int i = 0; i < in->dimensions_.size(); i++) {
    dims[i] = in->dimensions_[order[i]];
    strides_out[i] = 1;
    
    for (unsigned int j = 0; j < order[i]; j++) {
      strides_out[i] *= in->dimensions_[j];
    }
  }

  unsigned int* dims_dev        = 0;
  unsigned int* strides_out_dev = 0;
  
  if (hipMalloc((void**) &dims_dev, in->dimensions_.size()*sizeof(unsigned int)) != hipSuccess) {
    std::cerr << "cuNDArray_permute : Error allocating CUDA dims memory" << std::endl;
    return -1;
  }
  
  if (hipMalloc((void**) &strides_out_dev, in->dimensions_.size()*sizeof(unsigned int)) != hipSuccess) {
    std::cerr << "cuNDArray_permute : Error allocating CUDA strides_out memory" << std::endl;
    return -1;
  }
  
  if (hipMemcpy(dims_dev, dims, in->dimensions_.size()*sizeof(unsigned int), hipMemcpyHostToDevice) !=
      hipSuccess) {

    err = hipGetLastError();
    std::cerr << "cuNDArray_permute : Error uploading dimensions to device, " 
	      << hipGetErrorString(err) << std::endl;
    return -1;
  }

  if (hipMemcpy(strides_out_dev, strides_out, in->dimensions_.size()*sizeof(unsigned int), hipMemcpyHostToDevice) !=
      hipSuccess) {
    std::cerr << "cuNDArray_permute : Error uploading strides to device" << std::endl;
    return -1;
  }

  dim3 blockDim(512,1,1);
  dim3 gridDim((unsigned int) ceil((double)in->elements_/blockDim.x), 1, 1 );

  cuNDArray_permute_kernel<<< gridDim, blockDim >>>( in_ptr, out_ptr, in->dimensions_.size(), 
						     dims_dev, strides_out_dev, in->elements_, shift_mode);

  err = hipGetLastError();
  if( err != hipSuccess ){
    std::cerr << "cuNDArray_permute : Error during kernel call: " << hipGetErrorString(err) << std::endl;
    return -1;
  }

  if (hipFree(dims_dev) != hipSuccess) {
    err = hipGetLastError();
    std::cerr << "cuNDArray_permute: failed to delete device memory (dims_dev) " 
	      << hipGetErrorString(err) << std::endl;
    return -1;
  }

  if (hipFree(strides_out_dev) != hipSuccess) {
    err = hipGetLastError();
    std::cerr << "cuNDArray_permute: failed to delete device memory (strides_out_dev) " 
	      << hipGetErrorString(err) << std::endl;
    return -1;
  }
  
  delete [] dims;
  delete [] strides_out;

  if (!out) {
    std::vector<unsigned int> new_dims;
    for (unsigned int i = 0; i < in->dimensions_.size(); i++) {
      new_dims.push_back(in->dimensions_[order[i]]);
    }
    in->dimensions_ = new_dims;
    if (hipFree(in->data_) != hipSuccess) {
	std::cerr << "cuNDArray_permute: failed to delete device memory" << std::endl;
	return -1;
    }
    in->data_ = out_ptr;
  }

  return 0;
}

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<int>* in,
				 cuNDArray<int>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<int2>* in,
				 cuNDArray<int2>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<int3>* in,
				 cuNDArray<int3>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<int4>* in,
				 cuNDArray<int4>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<unsigned int>* in,
				 cuNDArray<unsigned int>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<uint2>* in,
				 cuNDArray<uint2>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<uint3>* in,
				 cuNDArray<uint3>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<uint4>* in,
				 cuNDArray<uint4>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<float>* in,
				 cuNDArray<float>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<float2>* in,
				 cuNDArray<float2>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<float3>* in,
				 cuNDArray<float3>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<float4>* in,
				 cuNDArray<float4>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<double>* in,
				 cuNDArray<double>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<double2>* in,
				 cuNDArray<double2>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<double3>* in,
				 cuNDArray<double3>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<double4>* in,
				 cuNDArray<double4>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<intd<1>::Type>* in,
				 cuNDArray<intd<1>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<intd<2>::Type>* in,
				 cuNDArray<intd<2>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<intd<3>::Type>* in,
				 cuNDArray<intd<3>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<intd<4>::Type>* in,
				 cuNDArray<intd<4>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<uintd<1>::Type>* in,
				 cuNDArray<uintd<1>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<uintd<2>::Type>* in,
				 cuNDArray<uintd<2>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<uintd<3>::Type>* in,
				 cuNDArray<uintd<3>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<uintd<4>::Type>* in,
				 cuNDArray<uintd<4>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<floatd<1>::Type>* in,
				 cuNDArray<floatd<1>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<floatd<2>::Type>* in,
				 cuNDArray<floatd<2>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<floatd<3>::Type>* in,
				 cuNDArray<floatd<3>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<floatd<4>::Type>* in,
				 cuNDArray<floatd<4>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<doubled<1>::Type>* in,
				 cuNDArray<doubled<1>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<doubled<2>::Type>* in,
				 cuNDArray<doubled<2>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<doubled<3>::Type>* in,
				 cuNDArray<doubled<3>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<doubled<4>::Type>* in,
				 cuNDArray<doubled<4>::Type>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<intd1>* in,
				 cuNDArray<intd1>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<intd2>* in,
				 cuNDArray<intd2>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<intd3>* in,
				 cuNDArray<intd3>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<intd4>* in,
				 cuNDArray<intd4>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<uintd1>* in,
				 cuNDArray<uintd1>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<uintd2>* in,
				 cuNDArray<uintd2>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<uintd3>* in,
				 cuNDArray<uintd3>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<uintd4>* in,
				 cuNDArray<uintd4>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<floatd1>* in,
				 cuNDArray<floatd1>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<floatd2>* in,
				 cuNDArray<floatd2>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<floatd3>* in,
				 cuNDArray<floatd3>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<floatd4>* in,
				 cuNDArray<floatd4>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<doubled1>* in,
				 cuNDArray<doubled1>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<doubled2>* in,
				 cuNDArray<doubled2>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<doubled3>* in,
				 cuNDArray<doubled3>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<doubled4>* in,
				 cuNDArray<doubled4>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);
				   
template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<float_complext>* in,
				 cuNDArray<float_complext>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);

template EXPORTGPUCORE int cuNDArray_permute<>(cuNDArray<double_complext>* in,
				 cuNDArray<double_complext>* out,
				 std::vector<unsigned int> order,
				 int shift_mode);
