#include "hip/hip_runtime.h"
#include "cuNDArray.h"
#include "vector_td.h"
#include <sstream>

namespace Gadgetron{
template <class T> 
__global__ void cuNDArray_permute_kernel(T* in, T* out, 
					 unsigned int ndim,
					 unsigned int* dims,
					 unsigned int* strides_out,
					 unsigned long int elements,
					 int shift_mode)
{
  unsigned long idx_in = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
  unsigned long idx_out = 0;

  unsigned long idx_in_tmp = idx_in;
  if (idx_in < elements) {

    unsigned int cur_index;
    for (unsigned int i = 0; i < ndim; i++) {
      unsigned long idx_in_remainder = idx_in_tmp / dims[i];
      cur_index = idx_in_tmp-(idx_in_remainder*dims[i]); //cur_index = idx_in_tmp%dims[i];
      if (shift_mode < 0) { //IFFTSHIFT
	idx_out += ((cur_index+(dims[i]>>1))%dims[i])*strides_out[i];
      } else if (shift_mode > 0) { //FFTSHIFT
	idx_out += ((cur_index+((dims[i]+1)>>1))%dims[i])*strides_out[i];
      } else {
	idx_out += cur_index*strides_out[i];
      }
      idx_in_tmp = idx_in_remainder;
    }
    out[idx_in] = in[idx_out];
  }
}

template <class T> void cuNDArray_permute(cuNDArray<T>* in,
 					 cuNDArray<T>* out,
					 std::vector<unsigned int> *order,
					 int shift_mode)
{
  hipError_t err;

  T* in_ptr = in->data_;
  T* out_ptr = 0;

  if (out) {
    out_ptr = out->data_;
  } else {
    if (hipMalloc((void**) &out_ptr, in->elements_*sizeof(T)) != hipSuccess) {
      BOOST_THROW_EXCEPTION(cuda_error("cuNDArray_permute : Error allocating CUDA memory"));


    }
  }

  unsigned int* dims        = new unsigned int[in->get_number_of_dimensions()];
  unsigned int* strides_out = new unsigned int[in->get_number_of_dimensions()];
  if (!dims || !strides_out) {
    BOOST_THROW_EXCEPTION(cuda_error("cuNDArray_permute: failed to allocate temporary storage for arrays"));

  }

  for (unsigned int i = 0; i < in->dimensions_->size(); i++) {
    dims[i] = (*in->dimensions_)[(*order)[i]];
    strides_out[i] = 1;
    
    for (unsigned int j = 0; j < (*order)[i]; j++) {
      strides_out[i] *= (*in->dimensions_)[j];
    }
  }

  unsigned int* dims_dev        = 0;
  unsigned int* strides_out_dev = 0;
  
  if (hipMalloc((void**) &dims_dev, in->dimensions_->size()*sizeof(unsigned int)) != hipSuccess) {
    BOOST_THROW_EXCEPTION(cuda_error("cuNDArray_permute : Error allocating CUDA dims memory"));

  }
  
  if (hipMalloc((void**) &strides_out_dev, in->dimensions_->size()*sizeof(unsigned int)) != hipSuccess) {
    BOOST_THROW_EXCEPTION(cuda_error("cuNDArray_permute : Error allocating CUDA strides_out memory"));

  }
  
  if (hipMemcpy(dims_dev, dims, in->dimensions_->size()*sizeof(unsigned int), hipMemcpyHostToDevice) !=
      hipSuccess) {

    err = hipGetLastError();
    std::stringstream ss;
    ss << "cuNDArray_permute : Error uploading dimensions to device, " << hipGetErrorString(err);
    BOOST_THROW_EXCEPTION(cuda_error(ss.str()));
  }

  if (hipMemcpy(strides_out_dev, strides_out, in->dimensions_->size()*sizeof(unsigned int), hipMemcpyHostToDevice) !=
      hipSuccess) {
    BOOST_THROW_EXCEPTION(cuda_error("cuNDArray_permute : Error uploading strides to device"));

  }

  dim3 blockDim(512,1,1);
  dim3 gridDim;
  if( in->dimensions_->size() > 2 ){
    gridDim = dim3((unsigned int) ceil((double)in->get_size(0)*in->get_size(1)/blockDim.x), 1, 1 );
    for( unsigned int d=2; d<in->dimensions_->size(); d++ )
      gridDim.y *= in->get_size(d);
  }
  else
    gridDim = dim3((unsigned int) ceil((double)in->elements_/blockDim.x), 1, 1 );

  cuNDArray_permute_kernel<<< gridDim, blockDim >>>( in_ptr, out_ptr, in->dimensions_->size(), 
						     dims_dev, strides_out_dev, in->elements_, shift_mode);

  err = hipGetLastError();
  if( err != hipSuccess ){
	  std::stringstream ss;
	  ss <<"cuNDArray_permute : Error during kernel call: " << hipGetErrorString(err);
    BOOST_THROW_EXCEPTION(cuda_error(ss.str()));

  }

  if (hipFree(dims_dev) != hipSuccess) {
    err = hipGetLastError();
    std::stringstream ss;
    ss << "cuNDArray_permute: failed to delete device memory (dims_dev) " << hipGetErrorString(err);
    BOOST_THROW_EXCEPTION(cuda_error(ss.str()));

  }

  if (hipFree(strides_out_dev) != hipSuccess) {
    err = hipGetLastError();
    std::stringstream ss;
    ss << "cuNDArray_permute: failed to delete device memory (strides_out_dev) "<< hipGetErrorString(err);
    BOOST_THROW_EXCEPTION(cuda_error(ss.str()));
  }
  
  delete [] dims;
  delete [] strides_out;

  if (!out) {
    std::vector<unsigned int> new_dims;
    for (unsigned int i = 0; i < in->dimensions_->size(); i++) {
      new_dims.push_back((*in->dimensions_)[(*order)[i]]);
    }
    *in->dimensions_ = new_dims;
    if( in->delete_data_on_destruct() ){
      if (hipFree(in->data_) != hipSuccess) {
	BOOST_THROW_EXCEPTION(cuda_error("cuNDArray_permute: failed to delete device memory"));

      }
      in->data_ = out_ptr;
    }
    else{
      if( hipMemcpy( in->data_, out_ptr, in->elements_*sizeof(T),  hipMemcpyDeviceToDevice) != hipSuccess ) {
   	BOOST_THROW_EXCEPTION(cuda_error("cuNDArray_permute: failed to copy device memory"));

      }
    }
  }
  

}

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<int>* in,
				 cuNDArray<int>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<int2>* in,
				 cuNDArray<int2>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<int3>* in,
				 cuNDArray<int3>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<int4>* in,
				 cuNDArray<int4>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<unsigned int>* in,
				 cuNDArray<unsigned int>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<uint2>* in,
				 cuNDArray<uint2>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<uint3>* in,
				 cuNDArray<uint3>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<uint4>* in,
				 cuNDArray<uint4>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<float>* in,
				 cuNDArray<float>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<float2>* in,
				 cuNDArray<float2>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<float3>* in,
				 cuNDArray<float3>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<float4>* in,
				 cuNDArray<float4>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<double>* in,
				 cuNDArray<double>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<double2>* in,
				 cuNDArray<double2>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<double3>* in,
				 cuNDArray<double3>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<double4>* in,
				 cuNDArray<double4>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);



template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<intd1>* in,
				 cuNDArray<intd1>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<intd2>* in,
				 cuNDArray<intd2>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<intd3>* in,
				 cuNDArray<intd3>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<intd4>* in,
				 cuNDArray<intd4>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<uintd1>* in,
				 cuNDArray<uintd1>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<uintd2>* in,
				 cuNDArray<uintd2>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<uintd3>* in,
				 cuNDArray<uintd3>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<uintd4>* in,
				 cuNDArray<uintd4>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<floatd1>* in,
				 cuNDArray<floatd1>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<floatd2>* in,
				 cuNDArray<floatd2>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<floatd3>* in,
				 cuNDArray<floatd3>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<floatd4>* in,
				 cuNDArray<floatd4>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<doubled1>* in,
				 cuNDArray<doubled1>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<doubled2>* in,
				 cuNDArray<doubled2>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<doubled3>* in,
				 cuNDArray<doubled3>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<doubled4>* in,
				 cuNDArray<doubled4>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);
				   
template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<float_complext>* in,
				 cuNDArray<float_complext>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);

template EXPORTGPUCORE void cuNDArray_permute<>(cuNDArray<double_complext>* in,
				 cuNDArray<double_complext>* out,
				 std::vector<unsigned int> *order,
				 int shift_mode);
}
