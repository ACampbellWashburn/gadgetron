#include <hipblas.h>
#include "cuGTBLAS.h"
#include "GadgetronCuException.h"
#include "complext.h"
#include "cudaDeviceManager.h"

namespace Gadgetron{

#define CUBLAS_CALL(fun) {hipblasStatus_t err = fun; if (err != HIPBLAS_STATUS_SUCCESS) {BOOST_THROW_EXCEPTION(cuda_error(getCublasErrorString(err)));}}
//NRM2

template<> hipblasStatus_t cublas_nrm2<float>(hipblasHandle_t hndl, int n, const float*  x, int inc, float* res){
		return hipblasSnrm2(hndl,n,x,inc,res);
}
template<> hipblasStatus_t cublas_nrm2<double>(hipblasHandle_t hndl, int n, const double*  x, int inc, double* res){
		return hipblasDnrm2(hndl,n,x,inc,res);
}
template<> hipblasStatus_t cublas_nrm2<float_complext>(hipblasHandle_t hndl, int n, const float_complext*  x, int inc, float* res){
		return hipblasScnrm2(hndl,n,(const hipComplex*)x,inc,res);
}
template<> hipblasStatus_t cublas_nrm2<double_complext>(hipblasHandle_t hndl, int n, const double_complext*  x, int inc, double* res){
		return hipblasDznrm2(hndl,n,(const hipDoubleComplex*) x,inc,res);
}

//DOT
template<> hipblasStatus_t cublas_dot<float>(hipblasHandle_t hndl, int n , const float* x , int incx, const  float* y , int incy, float* res){
	return hipblasSdot( hndl, n, x, incx, y, incy, res);
}
template<> hipblasStatus_t cublas_dot<double>(hipblasHandle_t hndl, int n , const double* x , int incx, const  double* y , int incy, double* res){
	return hipblasDdot( hndl, n, x, incx, y, incy, res);
}
template<> hipblasStatus_t cublas_dot<float_complext>(hipblasHandle_t hndl, int n , const float_complext* x ,
		int incx, const  float_complext* y , int incy, float_complext* res){
	return hipblasCdotc( hndl, n, (const hipComplex*) x, incx, (const hipComplex*) y, incy, (hipComplex*) res);
}
template<> hipblasStatus_t cublas_dot<double_complext>(hipblasHandle_t hndl, int n , const double_complext* x ,
		int incx, const  double_complext* y , int incy, double_complext* res){
	return hipblasZdotc( hndl, n, (const hipDoubleComplex*) x, incx, (const hipDoubleComplex*) y, incy, (hipDoubleComplex*) res);
}

// AXPY
template<> hipblasStatus_t cublas_axpy<float>(hipblasHandle_t hndl , int n , const float* a , const float* x , int incx ,  float* y , int incy){
	return hipblasSaxpy(hndl,n,a,x,incx,y,incy);
}
template<> hipblasStatus_t cublas_axpy<double>(hipblasHandle_t hndl , int n , const double* a , const double* x , int incx ,  double* y , int incy){
	return hipblasDaxpy(hndl,n,a,x,incx,y,incy);
}
template<> hipblasStatus_t cublas_axpy<float_complext>(hipblasHandle_t hndl , int n , const float_complext* a , const float_complext* x , int incx ,  float_complext* y , int incy){
	return hipblasCaxpy(hndl,n,(const hipComplex*) a, (const hipComplex*) x,incx, (hipComplex*)y,incy);
}
template<> hipblasStatus_t cublas_axpy<double_complext>(hipblasHandle_t hndl , int n , const double_complext* a , const double_complext* x , int incx ,  double_complext* y , int incy){
	return hipblasZaxpy(hndl,n,(const hipDoubleComplex*) a, (const hipDoubleComplex*) x,incx, (hipDoubleComplex*)y,incy);
}

//SUM
template<> hipblasStatus_t cublas_asum<float>(hipblasHandle_t hndl, int n,const float *x, int incx, float *result){
	return hipblasSasum(hndl,n,x,incx,result);
}
template<> hipblasStatus_t cublas_asum<double>(hipblasHandle_t hndl, int n,const double *x, int incx, double *result){
	return hipblasDasum(hndl,n,x,incx,result);
}
template<> hipblasStatus_t cublas_asum<float_complext>(hipblasHandle_t hndl, int n,const float_complext *x, int incx, float *result){
	return hipblasScasum(hndl,n,(const hipComplex*) x,incx,result);
}
template<> hipblasStatus_t cublas_asum<double_complext>(hipblasHandle_t hndl, int n,const double_complext *x, int incx, double *result){
	return hipblasDzasum(hndl,n,(const hipDoubleComplex*) x,incx,result);
}

//AMIN
template<> hipblasStatus_t cublas_amin<float>(hipblasHandle_t hndl, int n,const float *x, int incx, int *result){
	return hipblasIsamin(hndl,n,x,incx,result);
}
template<> hipblasStatus_t cublas_amin<double>(hipblasHandle_t hndl, int n,const double *x, int incx, int *result){
	return hipblasIdamin(hndl,n,x,incx,result);
}
template<> hipblasStatus_t cublas_amin<float_complext>(hipblasHandle_t hndl, int n,const float_complext *x, int incx, int *result){
	return hipblasIcamin(hndl,n, (const hipComplex* ) x,incx,result);
}
template<> hipblasStatus_t cublas_amin<double_complext>(hipblasHandle_t hndl, int n,const double_complext *x, int incx, int *result){
	return hipblasIzamin(hndl,n, (const hipDoubleComplex* ) x,incx,result);
}

//AMAX
template<> hipblasStatus_t cublas_amax<float>(hipblasHandle_t hndl, int n,const float *x, int incx, int *result){
	return hipblasIsamax(hndl,n,x,incx,result);
}
template<> hipblasStatus_t cublas_amax<double>(hipblasHandle_t hndl, int n,const double *x, int incx, int *result){
	return hipblasIdamax(hndl,n,x,incx,result);
}
template<> hipblasStatus_t cublas_amax<float_complext>(hipblasHandle_t hndl, int n,const float_complext *x, int incx, int *result){
	return hipblasIcamax(hndl,n, (const hipComplex* ) x,incx,result);
}
template<> hipblasStatus_t cublas_amax<double_complext>(hipblasHandle_t hndl, int n,const double_complext *x, int incx, int *result){
	return hipblasIzamax(hndl,n, (const hipDoubleComplex* ) x,incx,result);
}


template<class T> typename realType<T>::type
nrm2( cuNDArray<T>* arr, int device )
{
	typedef typename realType<T>::type REAL;
	REAL ret;

  CUBLAS_CALL(cublas_nrm2<T>( cudaDeviceManager::Instance()->getHandle(device), arr->get_number_of_elements(),
		    arr->get_data_ptr(), 1,
		   &ret));
  hipDeviceSynchronize();
  return ret;
}

template<class T> T
dot( cuNDArray<T>* arr1, cuNDArray<T>* arr2, int device )
{

  T ret;

  CUBLAS_CALL(cublas_dot( cudaDeviceManager::Instance()->getHandle(device), arr1->get_number_of_elements(),
		   arr1->get_data_ptr(), 1,
		   arr2->get_data_ptr(), 1,
		   &ret));

  hipDeviceSynchronize();
  return ret;
}


template<class T> void
axpy(T a,  cuNDArray<T>* x, cuNDArray<T>* y, int device )
{
  CUBLAS_CALL(cublas_axpy(cudaDeviceManager::Instance()->getHandle(device), x->get_number_of_elements(),
		  &a, x->get_data_ptr(), 1,
		  y->get_data_ptr(), 1));
  hipDeviceSynchronize();
}


template<class T> typename realType<T>::type asum(cuNDArray<T>* x,int device){

	typename realType<T>::type result;
	CUBLAS_CALL(cublas_asum(cudaDeviceManager::Instance()->getHandle(device),x->get_number_of_elements(),x->get_data_ptr(),1,&result));

	return result;
}

template<class T> int amin(cuNDArray<T>* x,int device){

	int result;
	CUBLAS_CALL(cublas_amin(cudaDeviceManager::Instance()->getHandle(device),x->get_number_of_elements(),x->get_data_ptr(),1,&result));
	return result;
}

template<class T> int amax(cuNDArray<T>* x,int device){
	int result;
	CUBLAS_CALL(cublas_amax(cudaDeviceManager::Instance()->getHandle(device),x->get_number_of_elements(),x->get_data_ptr(),1,&result));
	return result;
}


std::string getCublasErrorString(hipblasStatus_t err){
	switch (err){
	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "NOT INITIALIZED";
	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "ALLOC FAILED";
	case HIPBLAS_STATUS_INVALID_VALUE:
		return "INVALID VALUE";
	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "ARCH MISMATCH";
	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "MAPPING ERROR";
	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "EXECUTION FAILED";
	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "INTERNAL ERROR";

	case HIPBLAS_STATUS_SUCCESS:
		return "SUCCES";
	default:
		return "UNKNOWN CUBLAS ERROR";
	}
}

template float dot(cuNDArray<float> *x,cuNDArray<float> *y,int device);
template float nrm2( cuNDArray<float>* arr, int device);
template void axpy(float a, cuNDArray<float>* x, cuNDArray<float>* y,int device);
template int amin(cuNDArray<float>* x,int device);
template int amax(cuNDArray<float>* x,int device);
template float asum(cuNDArray<float>* x,int device);

template double dot(cuNDArray<double> *x,cuNDArray<double> *y,int device);
template double nrm2( cuNDArray<double>* arr, int device);
template void axpy(double a, cuNDArray<double>* x, cuNDArray<double>* y,int device);
template int amin(cuNDArray<double>* x,int device);
template int amax(cuNDArray<double>* x,int device);
template double asum(cuNDArray<double>* x,int device);

template float_complext dot(cuNDArray<float_complext> *x,cuNDArray<float_complext> *y,int device);
template float nrm2( cuNDArray<float_complext>* arr, int device);
template void axpy(float_complext a, cuNDArray<float_complext>* x, cuNDArray<float_complext>* y,int device);
template void axpy(float a, cuNDArray<float_complext>* x, cuNDArray<float_complext>* y,int device);

template int amin(cuNDArray<float_complext>* x,int device);
template int amax(cuNDArray<float_complext>* x,int device);
template float asum(cuNDArray<float_complext>* x,int device);


template double_complext dot(cuNDArray<double_complext> *x,cuNDArray<double_complext> *y,int device);
template double nrm2( cuNDArray<double_complext>* arr, int device);
template void axpy(double_complext a, cuNDArray<double_complext>* x, cuNDArray<double_complext>* y,int device);
template void axpy(double a, cuNDArray<double_complext>* x, cuNDArray<double_complext>* y,int device);

template int amin(cuNDArray<double_complext>* x,int device);
template int amax(cuNDArray<double_complext>* x,int device);
template double asum(cuNDArray<double_complext>* x,int device);

}

