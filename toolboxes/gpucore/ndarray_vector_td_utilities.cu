#include "hip/hip_runtime.h"
#include "ndarray_vector_td_utilities.h"
#include "real_utilities.h"
#include "real_utilities_device.h"
#include "check_CUDA.h"

#include <hipblas.h>

#include <vector>
#include <cmath>
#include <sstream>
#include <boost/throw_exception.hpp>
#include "GadgetronCuException.h"
#include "cuGTBLAS.h"
#include "cudaDeviceManager.h"


// Default template arguments seems to require c++-0x, which we can't assume. 
// We use a dummy type instead...
typedef float dummy;

//
// Some internal utilities
//

// Prepare array for processing:
//
// Sets the device context to the denoted compute device and makes device copies if necessary.
// If compute mode is CUNDA_NDARRAY_DEVICE the mandatory array 'in1' determines the compute device
//

using namespace Gadgetron;
template< unsigned int D, typename I1, typename I2, typename I3 > 
static void prepare( int compute_device, int *cur_device, int *old_device,
		     cuNDArray<I1> *in1,       cuNDArray<I1> **in1_int,
		     cuNDArray<I2> *in2 = 0x0, cuNDArray<I2> **in2_int = 0x0,
		     cuNDArray<I3> *in3 = 0x0, cuNDArray<I3> **in3_int = 0x0 )
{
  // Test validity of D
  if( D==0 || D>3 ){
    BOOST_THROW_EXCEPTION(runtime_error( ">>>Internal error<<< :prepare: D out of range"));

  }

  if( !cur_device || !old_device ){
    BOOST_THROW_EXCEPTION(runtime_error( ">>>Internal error<<< :prepare: device ids 0x0"));

  }

  // Test validity of input pointer
  if( !in1 || !in1_int ){
    BOOST_THROW_EXCEPTION(runtime_error( "unable to process 0x0 input"));

  }
  if( D>1 && (!in2 || !in2_int) ){
    BOOST_THROW_EXCEPTION(runtime_error( "unable to process 0x0 input"));

  }
  if( D>2 && (!in3 || !in3_int) ){
    BOOST_THROW_EXCEPTION(runtime_error( "unable to process 0x0 input"));

  }
  
  // Get current Cuda device
  if( hipGetDevice(old_device) != hipSuccess ) {
    BOOST_THROW_EXCEPTION(runtime_error( "unable to get device no"));

  }

  // Set the cuda device to use for computation
  if( compute_device == CUNDA_CURRENT_DEVICE ){
    *cur_device = *old_device; 
  }
  else if( compute_device == CUNDA_NDARRAY_DEVICE ){
    // Let D indicate which ndarray that determines the device
    // D denotes the output array (the latter ndarray in the list), if any, otherwise a sole input ndarray
    if( D == 1)
      *cur_device = in1->get_device();
    else if( D == 2 )
      *cur_device = in2->get_device();
    else if( D == 3 )
      *cur_device = in3->get_device();
  }
  else{
    BOOST_THROW_EXCEPTION(runtime_error( ">>>Internal error<<< :prepare: unknown compute mode"));

  }

  if( *cur_device != *old_device && hipSetDevice(*cur_device) != hipSuccess) {
    BOOST_THROW_EXCEPTION(runtime_error( "unable to set device no"));

  }
  
  // Transfer arrays to compute device if necessary
  if( *cur_device != in1->get_device() )
    *in1_int = new cuNDArray<I1>(*in1); // device transfer
  else
    *in1_int = in1;
  
  if( D>1 ){
    if( *cur_device != in2->get_device() )
      *in2_int = new cuNDArray<I2>(*in2); // device transfer
    else
      *in2_int = in2;
  }
  
  if( D>2 ){
    if( *cur_device != in3->get_device() )
      *in3_int = new cuNDArray<I3>(*in3); // device transfer
    else
      *in3_int = in3;
  }
  

}  

// Restore active device and free internal memory 
//
template< unsigned int D, typename I1, typename O, typename I2, typename I3 > 
static void restore( int old_device,
		     cuNDArray<I1> *in1, cuNDArray<I1> *in1_int,
		     unsigned int out_idx = 0, cuNDA_device alloc_device = CUNDA_NDARRAY_DEVICE, cuNDArray<O>  *out = 0x0,
		     cuNDArray<I2> *in2 = 0x0, cuNDArray<I2> *in2_int = 0x0,
		     cuNDArray<I3> *in3 = 0x0, cuNDArray<I3> *in3_int = 0x0 )
{
  // Test validity of D
  if( D==0 || D>3 ){
    BOOST_THROW_EXCEPTION(runtime_error( ">>>Internal error<<< :prepare: D out of range"));

  }

  // Test validity of input pointer
  if( !in1 || !in1_int ){
    BOOST_THROW_EXCEPTION(runtime_error( "unable to process 0x0 input"));

  }
  if( D>1 && (!in2 || !in2_int) ){
    BOOST_THROW_EXCEPTION(runtime_error( "unable to process 0x0 input"));

  }
  if( D>2 && (!in3 || !in3_int) ){
    BOOST_THROW_EXCEPTION(runtime_error( "unable to process 0x0 input"));

  }

  // Check if output ndarray resides on the desired device
  //
  if( out ){
    if( alloc_device == CUNDA_CURRENT_DEVICE && out->get_device() != old_device ){
      out->set_device( old_device ); } // device copy
    else if( alloc_device == CUNDA_NDARRAY_DEVICE && out->get_device() != in1->get_device() ){
      out->set_device( in1->get_device() ); } // device copy
  }

  // Check if in_out ndarray resides on the desired device
  //
  if( out_idx > 0 && out_idx < 4 ){ 

   if( out_idx > D ){
      BOOST_THROW_EXCEPTION(runtime_error( ">>>Internal error<<< :restore: array index out of range"));

    }
   
   if( D == 1 ){
     if( in1->get_device() != in1_int->get_device() ){ 
       *in1 = *in1_int; } // device transfer by assignment
   }
   if( D == 2 ){
     if( out_idx == 1 && in1->get_device() != in1_int->get_device() ){ 
       *in1 = *in1_int; } // device transfer by assignment
     else if( out_idx == 2 && in2->get_device() != in2_int->get_device() ){ 
       *in2 = *in2_int; } // device transfer by assignment
   }
   if( D == 3 ){
     if( out_idx == 1 && in1->get_device() != in1_int->get_device() ){ 
       *in1 = *in1_int; } // device transfer by assignment
     else if( out_idx == 2 && in2->get_device() != in2_int->get_device() ){ 
       *in2 = *in2_int; } // device transfer by assignment
     else if( out_idx == 3 && in3->get_device() != in3_int->get_device() ){ 
       *in3 = *in3_int; } // device transfer by assignment
   }
  }
  else if( out_idx != 0 ){
    BOOST_THROW_EXCEPTION(runtime_error( ">>>Internal error<<< :restore: illegal device specified"));

  }

  // Check if internal array needs deletion (they do only if they were created in ::prepare()
  //
  if( in1->get_device() != in1_int->get_device() ){
    delete in1_int;
  }
  if( D>1 && in2->get_device() != in2_int->get_device() ){
    delete in2_int;
  }
  if( D>2 && in3->get_device() != in3_int->get_device() ){
    delete in3_int;
  }

  // Get current Cuda device
  int device;
  if( hipGetDevice(&device) != hipSuccess ) {
    BOOST_THROW_EXCEPTION(cuda_error( "unable to get device no"));

  }

  // Restore old device
  if( device != old_device && hipSetDevice(old_device) != hipSuccess) {
    BOOST_THROW_EXCEPTION(cuda_error( "unable to restore device no"));

  }
    

}

// Common block/grid configuration utility
//
static void setup_grid( unsigned int cur_device, unsigned int number_of_elements,
			dim3 *blockDim, dim3* gridDim, unsigned int num_batches=1 )
{

  // For small arrays we keep the block dimension fairly small
  *blockDim = dim3(256);
  *gridDim = dim3((number_of_elements+blockDim->x-1)/blockDim->x, num_batches);
  int maxGridDim = cudaDeviceManager::Instance()->max_griddim(cur_device);
  // Extend block/grid dimensions for large arrays
  if( gridDim->x > maxGridDim){
    blockDim->x = maxGridDim;
    gridDim->x = (number_of_elements+blockDim->x-1)/blockDim->x;
  }

  if( gridDim->x > maxGridDim ){
    gridDim->x = ((unsigned int)sqrt((float)number_of_elements)+blockDim->x-1)/blockDim->x;
    gridDim->y *= ((number_of_elements+blockDim->x*gridDim->x-1)/(blockDim->x*gridDim->x));
  }
   
  if( gridDim->x >maxGridDim || gridDim->y >maxGridDim){

    BOOST_THROW_EXCEPTION(cuda_error("Grid dimension larger than supported by device"));
  }


}

// Common stride setup utility
//
template<class T> static void find_stride( cuNDArray<T> *in, unsigned int dim,
					   unsigned int *stride, std::vector<unsigned int> *dims )
{
  *stride = 1;
  for( unsigned int i=0; i<in->get_number_of_dimensions(); i++ ){
    if( i != dim )
      dims->push_back(in->get_size(i));
    if( i < dim )
      *stride *= in->get_size(i);
  }
}

//
// Implementation of public utilities
//

// cAbs
//
template<class REAL, class T> __global__ void
cAbs_kernel( T *in, REAL *out, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

  if( idx<number_of_elements ){
    T val = in[idx];
    out[idx] = Gadgetron::abs(val);
  }
}

// Abs
//
template<class T>
boost::shared_ptr< cuNDArray<typename realType<T>::type > >
Gadgetron::abs( cuNDArray<T> *in,
	    cuNDA_device alloc_device, cuNDA_device compute_device )
{
	typedef typename realType<T>::type REAL;
  int cur_device, old_device;
  cuNDArray<T> *in_int;

  // Prepare 
  prepare<1,T,dummy,dummy>( compute_device, &cur_device, &old_device, in, &in_int );

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, in->get_number_of_elements(), &blockDim, &gridDim );

  // Invoke kernel
  boost::shared_ptr< cuNDArray<REAL> > out = cuNDArray<REAL>::allocate(in->get_dimensions().get());
  if( out.get() != 0x0 ) cAbs_kernel<REAL,T><<< gridDim, blockDim >>>( in_int->get_data_ptr(), out->get_data_ptr(), in->get_number_of_elements() );

  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<1,T,REAL,dummy,dummy>( old_device, in, in_int, 0, alloc_device, out.get() );

  return out;
}


template<typename T>
class minmax_clamp_functor : public thrust::unary_function<T,T>
{
public:
	minmax_clamp_functor(T _min,T _max):min(_min),max(_max) {};
	 __inline__ __host__ __device__ T operator()(const T &y) const {
		 if (y < min) return min;
		 else if (y > max) return max;
		 else return y;
	 }
private:
	const T min,max;
};


template<typename T>
class minmax_clamp_functor<complext<T> > : public thrust::unary_function<complext<T>, complext<T> >
{
public:
	minmax_clamp_functor(T _min,T _max):min(_min),max(_max) {};
	 __inline__ __host__ __device__ complext<T>  operator()(const complext<T>  &y) const {
		 if (real(y) < min) return  complext<T>(min);
		 else if (real(y) > max) return  complext<T>(max);
		 else return  complext<T>(real(y));
	 }
private:
	const T min,max;
};

// CLAMP functions
template<class  T> EXPORTGPUCORE
void Gadgetron::clamp(cuNDArray<T> *in_out, typename realType<T>::type min, typename realType<T>::type max){
	thrust::transform(in_out->begin(),in_out->end(),in_out->begin(),minmax_clamp_functor<T>(min,max));
}

template<typename T>
class max_clamp_functor : public thrust::unary_function<T,T>
{
public:

	max_clamp_functor(T _max):max(_max) {};
	 __inline__ __host__ __device__ T operator()(const T &y) const {
		 if (y > max) return max;
		 return y;
	 }
private:
	const T max;
};

template<typename T>
class max_clamp_functor<complext<T> > : public thrust::unary_function<complext<T>, complext<T> >
{
public:
		max_clamp_functor(T _max):max(_max) {};
	 __inline__ __host__ __device__ complext<T>  operator()(const complext<T>  &y) const {
		 if (real(y) > max) return  complext<T>(max);
		 else return  complext<T>(real(y));
	 }
private:
	const T max;
};
// CLAMP functions
template<class  T> EXPORTGPUCORE
void Gadgetron::clamp_max(cuNDArray<T> *in_out, typename realType<T>::type max){
	thrust::transform(in_out->begin(),in_out->end(),in_out->begin(),max_clamp_functor<T>(max));
}

template<typename T>
class min_clamp_functor : public thrust::unary_function<T,T>
{
public:

	min_clamp_functor(T _min):min(_min) {};
	 __inline__ __host__ __device__ T operator()(const T &y) const {
		 if (y < min) return min;
		 return y;
	 }
private:
	const T min;
};

template<typename T>
class min_clamp_functor<complext<T> > : public thrust::unary_function<complext<T>, complext<T> >
{
public:
	min_clamp_functor(T _min):min(_min) {};
	 __inline__ __host__ __device__ complext<T>  operator()(const complext<T>  &y) const {
		 if (real(y) < min) return  complext<T>(min);
		 else return  complext<T>(real(y));
	 }
private:
	const T min;
};
// CLAMP functions
template<class  T> EXPORTGPUCORE
void Gadgetron::clamp_min(cuNDArray<T> *in_out, typename realType<T>::type min){
	thrust::transform(in_out->begin(),in_out->end(),in_out->begin(),min_clamp_functor<T>(min));
}

// Sum
//
template<class T> __global__ void
sum_kernel( T *in, T *out,
		  unsigned int stride, 
		  unsigned int number_of_batches, 
		  unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

  if( idx < number_of_elements ){

    unsigned int in_idx = (idx/stride)*stride*number_of_batches+(idx%stride);
 
    T val = in[in_idx];
 
    for( unsigned int i=1; i<number_of_batches; i++ ) 
      val += in[i*stride+in_idx];

    out[idx] = val; 
  }
}

// Sum
//
template<class T>  
boost::shared_ptr< cuNDArray<T> >
Gadgetron::sum( cuNDArray<T> *in, unsigned int dim,
	   cuNDA_device alloc_device, cuNDA_device compute_device )
{
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<T> *in_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,T,dummy,dummy>( compute_device, &cur_device, &old_device, in, &in_int );
  
  // Some validity checks
  if( !(in->get_number_of_dimensions()>1) ){
    BOOST_THROW_EXCEPTION(runtime_error("sum: underdimensioned."));
  }
 
  if( dim > in->get_number_of_dimensions()-1 ){
    BOOST_THROW_EXCEPTION(runtime_error( "sum: dimension out of range."));
  }

  unsigned int number_of_batches = in->get_size(dim);
  unsigned int number_of_elements = in->get_number_of_elements()/number_of_batches;

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, number_of_elements, &blockDim, &gridDim );
 
  // Find element stride
  unsigned int stride; std::vector<unsigned int> dims;
  find_stride<T>( in, dim, &stride, &dims );

  // Invoke kernel
  boost::shared_ptr< cuNDArray<T> > out = cuNDArray<T>::allocate(&dims);
  if( out.get() != 0x0 ) sum_kernel<T><<< gridDim, blockDim >>>( in_int->get_data_ptr(), out->get_data_ptr(), stride, number_of_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<1,T,T,dummy,dummy>( old_device, in, in_int, 0, alloc_device, out.get() );

  return out;
}

// Expand
//
template<class T> __global__ void
expand_kernel( T *in, T *out,
		     unsigned int number_of_elements,
		     unsigned int new_dim_size )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

  if( idx < number_of_elements ){
    out[idx] = in[idx%(number_of_elements/new_dim_size)];
  }
}

// Expand
//
template<class T>  
boost::shared_ptr< cuNDArray<T> >
Gadgetron::expand( cuNDArray<T> *in, unsigned int new_dim_size,
	   cuNDA_device alloc_device, cuNDA_device compute_device )
{
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<T> *in_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,T,dummy,dummy>( compute_device, &cur_device, &old_device, in, &in_int );
   
  unsigned int number_of_elements = in->get_number_of_elements()*new_dim_size;

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, number_of_elements, &blockDim, &gridDim );
 
  // Find element stride
  std::vector<unsigned int> dims = *in->get_dimensions();
  dims.push_back(new_dim_size);

  // Invoke kernel
  boost::shared_ptr< cuNDArray<T> > out = cuNDArray<T>::allocate(&dims);
  if( out.get() != 0x0 ) expand_kernel<T><<< gridDim, blockDim >>>( in_int->get_data_ptr(), out->get_data_ptr(), number_of_elements, new_dim_size );
 
  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<1,T,T,dummy,dummy>( old_device, in, in_int, 0, alloc_device, out.get() );

  return out;
}

// SS
template<class REAL, class T> __inline__  __device__ REAL
_ss( unsigned int idx, T *in, unsigned int stride, unsigned int number_of_batches )
{
  unsigned int in_idx = (idx/stride)*stride*number_of_batches+(idx%stride);
  REAL ss = REAL(0);

  for( unsigned int i=0; i<number_of_batches; i++ )
    ss += norm(in[i*stride+in_idx]);

  return ss;
}

// SS
template<class REAL, class T> __global__ void
ss_kernel( T *in, REAL *out, unsigned int stride, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

  if( idx < number_of_elements ){
    out[idx] = _ss<REAL,T>(idx, in, stride, number_of_batches);
  }
}

// squaredNorm
template<class T>
boost::shared_ptr< cuNDArray<typename realType<T>::type> >
Gadgetron::squaredNorm( cuNDArray<T> *in, unsigned int dim,
	   cuNDA_device alloc_device, cuNDA_device compute_device )
{
	typedef typename realType<T>::type REAL;
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<T> *in_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,T,dummy,dummy>( compute_device, &cur_device, &old_device, in, &in_int );

  // Validity checks
  if( !(in->get_number_of_dimensions()>1) ){
    BOOST_THROW_EXCEPTION(runtime_error( "ss: underdimensioned."));

  }

  if( dim > in->get_number_of_dimensions()-1 ){
    BOOST_THROW_EXCEPTION(runtime_error( "ss: dimension out of range."));

  }

  unsigned int number_of_batches = in->get_size(dim);
  unsigned int number_of_elements = in->get_number_of_elements()/number_of_batches;

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, number_of_elements, &blockDim, &gridDim );

  // Find element stride
  unsigned int stride; std::vector<unsigned int> dims;
  find_stride<T>( in, dim, &stride, &dims );

  // Invoke kernel
  boost::shared_ptr< cuNDArray<REAL> > out = cuNDArray<REAL>::allocate(&dims);
  if ( out.get() != 0x0 ) ss_kernel<REAL,T><<< gridDim, blockDim >>>( in_int->get_data_ptr(), out->get_data_ptr(), stride, number_of_batches, number_of_elements );

  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<1,T,REAL,dummy,dummy>( old_device, in, in_int, 0, alloc_device, out.get() );

  return out;
}

// RSS
template<class REAL, class T> __inline__  __device__ REAL
_rss( unsigned int idx, T *in, unsigned int stride, unsigned int number_of_batches )
{
  unsigned int in_idx = (idx/stride)*stride*number_of_batches+(idx%stride);
  REAL rss = REAL(0);
  
  for( unsigned int i=0; i<number_of_batches; i++ ) 
    rss += norm(in[i*stride+in_idx]);
  
  rss = sqrt(rss); 

  return rss;
}

// RSS
template<class REAL, class T> __global__ void
rss_kernel( T *in, REAL *out, unsigned int stride, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

  if( idx < number_of_elements ){
    out[idx] = _rss<REAL,T>(idx, in, stride, number_of_batches); 
  }
}
/*
// RSS
template<class REAL, class T>  
boost::shared_ptr< cuNDArray<REAL> >
_rss( cuNDArray<T> *in, unsigned int dim,
	    cuNDA_device alloc_device, cuNDA_device compute_device )
{
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<T> *in_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,T,dummy,dummy>( compute_device, &cur_device, &old_device, in, &in_int );

  // Validity checks
  if( !(in->get_number_of_dimensions()>1) ){
    BOOST_THROW_EXCEPTION(runtime_error( "rss: underdimensioned."));

  }
 
  if( dim > in->get_number_of_dimensions()-1 ){
    BOOST_THROW_EXCEPTION(runtime_error( "rss: dimension out of range."));
  }

  unsigned int number_of_batches = in->get_size(dim);
  unsigned int number_of_elements = in->get_number_of_elements()/number_of_batches;

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, number_of_elements, &blockDim, &gridDim );

  // Find element stride
  unsigned int stride; std::vector<unsigned int> dims;
  find_stride<T>( in, dim, &stride, &dims );

  // Invoke kernel
  boost::shared_ptr< cuNDArray<REAL> > out = cuNDArray<REAL>::allocate(&dims); 
  if ( out.get() != 0x0 ) rss_kernel<REAL,T><<< gridDim, blockDim >>>( in_int->get_data_ptr(), out->get_data_ptr(), stride, number_of_batches, number_of_elements );
  
  CHECK_FOR_CUDA_ERROR();
 
  // Restore
  restore<1,T,REAL,dummy,dummy>( old_device, in, in_int, 0, alloc_device, out.get() );

  return out;
}*/

// cRSS
template<class REAL, class T> __global__ void
crss_kernel( T *in, T *out, unsigned int stride, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

  if( idx < number_of_elements ){

    REAL rss = _rss<REAL,T>(idx, in, stride, number_of_batches); 

    out[idx].vec[0] = rss;
    out[idx].vec[1] = REAL(0);
  }
}


// Build correlation matrix
template<class REAL, class T> __global__ void
correlation_kernel( T *in, T *corrm, unsigned int num_batches, unsigned int num_elements )
{
  const unsigned int p = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int i = threadIdx.y;

  if( p < num_elements ){
    for( unsigned int j=0; j<i; j++){
      T tmp = in[i*num_elements+p]*conj(in[j*num_elements+p]);
      corrm[(j*num_batches+i)*num_elements+p] = tmp;
      corrm[(i*num_batches+j)*num_elements+p] = conj(tmp);
    }
    T tmp = in[i*num_elements+p];
    corrm[(i*num_batches+i)*num_elements+p] = tmp*conj(tmp);
  }
}

// Build correlation matrix
template<class T>
boost::shared_ptr< cuNDArray<T> >
Gadgetron::correlation( cuNDArray<T> *in,
		    cuNDA_device alloc_device, cuNDA_device compute_device )
{
	typedef typename realType<T>::type REAL;
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<T> *in_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,T,dummy,dummy>( compute_device, &cur_device, &old_device, in, &in_int );

  // Validity checks
  if( !(in->get_number_of_dimensions()>1) ){
  	BOOST_THROW_EXCEPTION(runtime_error("correlation: underdimensioned."));
  }
 
  unsigned int number_of_batches = in->get_size(in->get_number_of_dimensions()-1);
  unsigned int number_of_elements = in->get_number_of_elements()/number_of_batches;

  int warp_size = cudaDeviceManager::Instance()->warp_size(old_device);
  int max_blockdim = cudaDeviceManager::Instance()->max_blockdim(old_device);
  dim3 blockDim(((max_blockdim/number_of_batches)/warp_size)*warp_size, number_of_batches);

  if( blockDim.x == 0 ){
  	BOOST_THROW_EXCEPTION(runtime_error("correlation: correlation dimension exceeds device capacity."));
  }
  
  dim3 gridDim((number_of_elements+blockDim.x-1)/blockDim.x);

  // Invoke kernel
  std::vector<unsigned int> dims = *in->get_dimensions(); dims.push_back(number_of_batches);
  boost::shared_ptr< cuNDArray<T> > out = cuNDArray<T>::allocate(&dims);
  if( out.get() != 0x0 ) correlation_kernel<REAL,T><<< gridDim, blockDim >>>( in_int->get_data_ptr(), out->get_data_ptr(), number_of_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<1,T,T,dummy,dummy>( old_device, in, in_int, 0, alloc_device, out.get() );

  return out;
}

// Real to complext
template<class REAL> __global__ void
real_to_complext_kernel( REAL *in, complext<REAL> *out, unsigned int num_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

  if( idx < num_elements ){
  	complext<REAL> z;
    z.vec[0] = in[idx];
    z.vec[1] = REAL(0);
    out[idx] = z;
  }
}

// Convert real to complext
template<class REAL>  
boost::shared_ptr< cuNDArray<complext<REAL> > >
Gadgetron::real_to_complext( cuNDArray<REAL> *in,
			cuNDA_device alloc_device, cuNDA_device compute_device )
{
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<REAL> *in_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,REAL,dummy,dummy>( compute_device, &cur_device, &old_device, in, &in_int );
 
  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, in->get_number_of_elements(), &blockDim, &gridDim );

  // Invoke kernel
  boost::shared_ptr< cuNDArray<complext<REAL> > > out = cuNDArray<complext<REAL> >::allocate(in->get_dimensions().get());
  if( out.get() != 0x0 ) real_to_complext_kernel<REAL><<< gridDim, blockDim >>>( in_int->get_data_ptr(), out->get_data_ptr(), in->get_number_of_elements());
  
  CHECK_FOR_CUDA_ERROR();
  
  // Restore
  restore<1,REAL,complext<REAL>,dummy,dummy>( old_device, in, in_int, 0, alloc_device, out.get() );

  return out;
}

// complext to real by cropping the imaginary component 
template<class REAL> __global__ void
complext_to_real_kernel( complext<REAL> *in, REAL *out, unsigned int num_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

  if( idx < num_elements ){
    out[idx] = in[idx].vec[0];
  }
}

// Convert complext to real by cropping the imaginary component 
template<class REAL>  
boost::shared_ptr< cuNDArray<REAL> >
Gadgetron::complext_to_real( cuNDArray<complext<REAL> > *in,
			cuNDA_device alloc_device, cuNDA_device compute_device )
{
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<complext<REAL> > *in_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,complext<REAL>,dummy,dummy>( compute_device, &cur_device, &old_device, in, &in_int ) ;
 
  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, in->get_number_of_elements(), &blockDim, &gridDim );

  // Invoke kernel
  boost::shared_ptr< cuNDArray<REAL> > out = cuNDArray<REAL>::allocate(in->get_dimensions().get());  
  if( out.get() != 0x0 ) complext_to_real_kernel<REAL><<< gridDim, blockDim >>>( in_int->get_data_ptr(), out->get_data_ptr(), in->get_number_of_elements());
  
  CHECK_FOR_CUDA_ERROR();
  
  // Restore
  restore<1,complext<REAL>,REAL,dummy,dummy>( old_device, in, in_int, 0, alloc_device, out.get() );

  return out;
}

// Downsample
template<class REAL, unsigned int D> __global__ void
downsample_kernel( REAL *in, REAL *out,
			 vector_td<unsigned int,D> matrix_size_in, vector_td<unsigned int,D> matrix_size_out,
			 unsigned int num_elements, unsigned int num_batches )
{
	typedef vector_td<unsigned int,D> uintd;
  // We have started a thread for each output element
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
  const unsigned int frame_offset = idx/num_elements;
  
  if( idx < num_elements*num_batches ){

    const uintd co_out = idx_to_co<D>( idx-frame_offset*num_elements, matrix_size_out );
    const uintd co_in = co_out << 1;

    const uintd twos = to_vector_td<unsigned int,D>(2);
    const unsigned int num_adds = 1 << D;
    unsigned int actual_adds = 0;

    REAL res = REAL(0);

    for( unsigned int i=0; i<num_adds; i++ ){
      const uintd local_co = idx_to_co<D>( i, twos );
      if( weak_greater_equal( local_co, matrix_size_out ) ) continue; // To allow array dimensions of 1
      const unsigned int in_idx = co_to_idx<D>(co_in+local_co, matrix_size_in)+frame_offset*prod(matrix_size_in);
      actual_adds++;
      res += in[in_idx];
    }
    
    out[idx] = res/REAL(actual_adds);
  }
}

// Downsample
template<class REAL, unsigned int D>
boost::shared_ptr< cuNDArray<REAL> >
Gadgetron::downsample( cuNDArray<REAL> *in,
		  cuNDA_device alloc_device, cuNDA_device compute_device )
{
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<REAL> *in_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,REAL,dummy,dummy>( compute_device, &cur_device, &old_device, in, &in_int );
     
  // A few sanity checks 
  if( in->get_number_of_dimensions() < D ){
    BOOST_THROW_EXCEPTION(runtime_error( "downsample: the number of array dimensions should be at least D"));

  }
  
  for( unsigned int d=0; d<D; d++ ){
    if( (in->get_size(d)%2) == 1 && in->get_size(d) != 1 ){
      BOOST_THROW_EXCEPTION(runtime_error( "downsample: uneven array dimensions larger than one not accepted"));
    }
  }
  
  typename uintd<D>::Type matrix_size_in = vector_to_uintd<D>( *in->get_dimensions() );
  typename uintd<D>::Type matrix_size_out = matrix_size_in >> 1;

  for( unsigned int d=0; d<D; d++ ){
    if( matrix_size_out[d] == 0 ) 
      matrix_size_out[d] = 1;
  }
  
  unsigned int number_of_elements = prod(matrix_size_out);
  unsigned int number_of_batches = 1;

  for( unsigned int d=D; d<in->get_number_of_dimensions(); d++ ){
    number_of_batches *= in->get_size(d);
  }
  
  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;

  setup_grid( cur_device, number_of_elements, &blockDim, &gridDim, number_of_batches ) ;
  
  // Invoke kernel
  std::vector<unsigned int> dims = uintd_to_vector<D>(matrix_size_out);
  for( unsigned int d=D; d<in->get_number_of_dimensions(); d++ ){
    dims.push_back(in->get_size(d));
  }
  
  boost::shared_ptr< cuNDArray<REAL> > out = cuNDArray<REAL>::allocate(&dims);
  if( out.get() != 0x0 ) 
    downsample_kernel<REAL,D><<< gridDim, blockDim >>>
      ( in_int->get_data_ptr(), out->get_data_ptr(), matrix_size_in, matrix_size_out, number_of_elements, number_of_batches );
  
  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<1,REAL,REAL,dummy,dummy>( old_device, in, in_int, 0, alloc_device, out.get() );

  return out;
}

// Nearest neighbor upsampling
template<class REAL, unsigned int D> __global__ void
upsample_nn_kernel( REAL *in, REAL *out,
		vector_td<unsigned int,D> matrix_size_in, vector_td<unsigned int,D> matrix_size_out,
		       unsigned int num_elements, unsigned int num_batches )
{
	typedef vector_td<unsigned int,D> uintd;
  // We have started a thread for each output element
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
  
  if( idx < num_elements*num_batches ){    
    const unsigned int frame_idx = idx/num_elements;
    const uintd co_out = idx_to_co<D>( idx-frame_idx*num_elements, matrix_size_out );
    const uintd co_in = co_out >> 1;
    out[idx] = in[co_to_idx<D>(co_in, matrix_size_in)+frame_idx*prod(matrix_size_in)];
  }
}

// Nearest neighbor upsampling
template<class REAL, unsigned int D>
boost::shared_ptr< cuNDArray<REAL> >
Gadgetron::upsample_nn( cuNDArray<REAL> *in,
		   cuNDA_device alloc_device, cuNDA_device compute_device )
{
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<REAL> *in_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,REAL,dummy,dummy>( compute_device, &cur_device, &old_device, in, &in_int );
     
  // A few sanity checks 
  if( in->get_number_of_dimensions() < D ){
    BOOST_THROW_EXCEPTION(runtime_error( "upsample: the number of array dimensions should be at least D" ));

  }
    
  typename uintd<D>::Type matrix_size_in = vector_to_uintd<D>( *in->get_dimensions() );
  typename uintd<D>::Type matrix_size_out = matrix_size_in << 1;

  unsigned int number_of_elements = prod(matrix_size_out);
  unsigned int number_of_batches = 1;

  for( unsigned int d=D; d<in->get_number_of_dimensions(); d++ ){
    number_of_batches *= in->get_size(d);
  }
  
  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;

  setup_grid( cur_device, number_of_elements, &blockDim, &gridDim, number_of_batches );
  
  // Invoke kernel
  std::vector<unsigned int> dims = uintd_to_vector<D>(matrix_size_out);
  for( unsigned int d=D; d<in->get_number_of_dimensions(); d++ ){
    dims.push_back(in->get_size(d));
  }
  boost::shared_ptr< cuNDArray<REAL> > out = cuNDArray<REAL>::allocate(&dims);
  if( out.get() != 0x0 ) 
    upsample_nn_kernel<REAL,D><<< gridDim, blockDim >>>
      ( in_int->get_data_ptr(), out->get_data_ptr(), matrix_size_in, matrix_size_out, number_of_elements, number_of_batches );
  
  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<1,REAL,REAL,dummy,dummy>( old_device, in, in_int, 0, alloc_device, out.get() );

  return out;
}

// Utility to check if all neighbors required for the linear interpolation exists
// ... do not include dimensions of size 1

template<class REAL, unsigned int D> __device__ 
bool is_border_pixel( vector_td<unsigned int,D> co, vector_td<unsigned int,D> dims )
{
  for( unsigned int dim=0; dim<D; dim++ ){
    if( dims[dim] > 1 && ( co[dim] == 0 || co[dim] == (dims[dim]-1) ) )
      return true;
  }
  return false;
}

// Linear upsampling
template<class REAL, unsigned int D> __global__ void
upsample_lin_kernel( REAL *in, REAL *out,
		vector_td<unsigned int,D> matrix_size_in, vector_td<unsigned int,D> matrix_size_out,
		       unsigned int num_elements, unsigned int num_batches )
{
	typedef vector_td<unsigned int,D> uintd;
  // We have started a thread for each output element
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
  
  if( idx < num_elements*num_batches ){

    REAL res = REAL(0);

    const unsigned int num_neighbors = 1 << D;
    const unsigned int frame_idx = idx/num_elements;
    const uintd co_out = idx_to_co<D>( idx-frame_idx*num_elements, matrix_size_out );

    // We will only proceed if all neighbours exist (this adds a zero-boundary to the upsampled image/vector field)
    //
    
    if( !is_border_pixel<REAL,D>(co_out, matrix_size_out) ){
      
      for( unsigned int i=0; i<num_neighbors; i++ ){
	
	// Determine coordinate of neighbor in input
	//

	const uintd twos = to_vector_td<unsigned int,D>(2);
	const uintd stride = idx_to_co<D>( i, twos );

	if( weak_greater_equal( stride, matrix_size_out ) ) continue; // To allow array dimensions of 1

	// Be careful about dimensions of size 1
	uintd ones = to_vector_td<unsigned int,D>(1);
	for( unsigned int d=0; d<D; d++ ){
	  if( matrix_size_out[d] == 1 )
	    ones[d] = 0;
	}
	uintd co_in = ((co_out-ones)>>1)+stride;
	
	// Read corresponding pixel value
	//
	
	const unsigned int in_idx = co_to_idx<D>(co_in, matrix_size_in)+frame_idx*prod(matrix_size_in);
	REAL value = in[in_idx];
	
	// Determine weight
	//
	
	REAL weight = REAL(1);
	
	for( unsigned int dim=0; dim<D; dim++ ){	  
	  if( matrix_size_in[dim] > 1 ){
	    if( stride.vec[dim] == (co_out.vec[dim]%2) ) {
	      weight *= REAL(0.25);
	    }
	    else{
	      weight *= REAL(0.75);
	    }
	  }
	}
	
	// Accumulate result
	//
	
	res += weight*value;
      }
    }
    out[idx] = res;
  }
}

// Linear interpolation upsampling
template<class REAL, unsigned int D>
boost::shared_ptr< cuNDArray<REAL> >
Gadgetron::upsample_lin( cuNDArray<REAL> *in,
		    cuNDA_device alloc_device, cuNDA_device compute_device )
{
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<REAL> *in_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,REAL,dummy,dummy>( compute_device, &cur_device, &old_device, in, &in_int );
     
  // A few sanity checks 
  if( in->get_number_of_dimensions() < D ){
    BOOST_THROW_EXCEPTION(runtime_error( "upsample: the number of array dimensions should be at least D"));
  }
    
  typename uintd<D>::Type matrix_size_in = vector_to_uintd<D>( *in->get_dimensions() );
  typename uintd<D>::Type matrix_size_out = matrix_size_in << 1;

  for( unsigned int d=0; d<D; d++ ){
    if( matrix_size_in[d] == 1 )
      matrix_size_out[d] = 1;
  }
  
  unsigned int number_of_elements = prod(matrix_size_out);
  unsigned int number_of_batches = 1;

  for( unsigned int d=D; d<in->get_number_of_dimensions(); d++ ){
    number_of_batches *= in->get_size(d);
  }
  
  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;

  setup_grid( cur_device, number_of_elements, &blockDim, &gridDim, number_of_batches );
  
  // Invoke kernel
  std::vector<unsigned int> dims = uintd_to_vector<D>(matrix_size_out);
  for( unsigned int d=D; d<in->get_number_of_dimensions(); d++ ){
    dims.push_back(in->get_size(d));
  }
  boost::shared_ptr< cuNDArray<REAL> > out = cuNDArray<REAL>::allocate(&dims);
  if( out.get() != 0x0 ) 
    upsample_lin_kernel<REAL,D><<< gridDim, blockDim >>>
      ( in_int->get_data_ptr(), out->get_data_ptr(), matrix_size_in, matrix_size_out, number_of_elements, number_of_batches );
  
  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<1,REAL,REAL,dummy,dummy>( old_device, in, in_int, 0, alloc_device, out.get() );

  return out;
}

template<typename T>
struct sign_functor
{
	  __host__ __device__
	  T operator()(const T & x) const{
		  return sgn(x);
	  }
};

template <class T> void Gadgetron::inplace_sgn(cuNDArray<T>* x) {
		  thrust::device_ptr<T> dev_ptr(x->get_data_ptr());
		  thrust::transform(dev_ptr, dev_ptr + x->get_number_of_elements(),dev_ptr,sign_functor<T>());
}


// Reciprocal square root
template<class T> __global__ 
void reciprocal_sqrt_kernel( T *in_out, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
 
  if( idx<number_of_elements ){
    in_out[idx] = rsqrt(in_out[idx]);
  }
}


// Normalized RSS
template<class REAL, class T> __global__ void
rss_normalize_kernel( T *in_out, unsigned int stride, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

  if( idx < number_of_elements ){

    REAL reciprocal_rss = 1/(_rss<REAL,T>(idx, in_out, stride, number_of_batches));
 
    unsigned int in_idx = (idx/stride)*stride*number_of_batches+(idx%stride);

    for( unsigned int i=0; i<number_of_batches; i++ ) {
      T out = in_out[i*stride+in_idx];
      out *= reciprocal_rss; // complex-scalar multiplication (element-wise operator)
      in_out[i*stride+in_idx] = out; 
    } 
  }
}

// Normalized RSS
template<class T>
void Gadgetron::rss_normalize( cuNDArray<T> *in_out, unsigned int dim,
			  cuNDA_device compute_device )
{
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<T> *in_out_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,T,dummy,dummy>( compute_device, &cur_device, &old_device, in_out, &in_out_int );

  // Validity checks
  if( !(in_out->get_number_of_dimensions()>1) ){
    BOOST_THROW_EXCEPTION(runtime_error( "rss_normalize: underdimensioned."));

  }
 
  if( dim > in_out->get_number_of_dimensions()-1 ){
  	BOOST_THROW_EXCEPTION(runtime_error("rss_normalize: dimension out of range."));

  }

  unsigned int number_of_batches = in_out->get_size(dim);
  unsigned int number_of_elements = in_out->get_number_of_elements()/number_of_batches;

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, number_of_elements, &blockDim, &gridDim );
  // Find element stride
  unsigned int stride; std::vector<unsigned int> dims;
  find_stride<T>( in_out, dim, &stride, &dims );

  // Invoke kernel
  rss_normalize_kernel<typename realType<T>::type,T><<< gridDim, blockDim >>>( in_out_int->get_data_ptr(), stride, number_of_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<1,T,dummy,dummy,dummy>( old_device, in_out, in_out_int, 1 );


}





// Normalize (float)
template<class T> EXPORTGPUCORE
T Gadgetron::normalize( cuNDArray<T> *data, T new_max, cuNDA_device compute_device )
{

  unsigned int number_of_elements = data->get_number_of_elements();

  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<T> *data_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,T,dummy,dummy>( compute_device, &cur_device, &old_device, data, &data_int );

  // Find the maximum value in the array
  int max_idx=amax(data_int);

  hipDeviceSynchronize();
  
  // Copy that value back to host memory
  T max_val;
  hipMemcpy(&max_val, (data_int->get_data_ptr()+max_idx-1), sizeof(T), hipMemcpyDeviceToHost);

  // Scale the array
  T scale = std::abs(new_max/max_val);
  *data_int *= scale;
  // Restore
  restore<1,T,dummy,dummy,dummy>( old_device, data, data_int, 1, compute_device );

  CHECK_FOR_CUDA_ERROR();
  return scale;
}

// Crop
template<class T, unsigned int D> __global__ void
crop_kernel( vector_td<unsigned int,D> offset, vector_td<unsigned int,D> matrix_size_in, vector_td<unsigned int,D> matrix_size_out,
		   T *in, T *out, unsigned int num_batches, unsigned int num_elements )
{
	typedef vector_td<unsigned int,D> uintd;
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
  const unsigned int frame_offset = idx/num_elements;

  if( idx < num_elements*num_batches ){
    const uintd co = idx_to_co<D>( idx-frame_offset*num_elements, matrix_size_out );
    const uintd co_os = offset + co;
    const unsigned int in_idx = co_to_idx<D>(co_os, matrix_size_in)+frame_offset*prod(matrix_size_in);
    out[idx] = in[in_idx];
  }
}

// Crop
template<class T, unsigned int D> EXPORTGPUCORE
void Gadgetron::crop( typename uintd<D>::Type offset,
	    cuNDArray<T> *in, cuNDArray<T> *out,
	    cuNDA_device compute_device )
{
  if( in == 0x0 || out == 0x0 ){

    BOOST_THROW_EXCEPTION(runtime_error("crop: 0x0 ndarray provided"));
  }

  if( in->get_number_of_dimensions() != out->get_number_of_dimensions() ){
  	BOOST_THROW_EXCEPTION(runtime_error("crop: image dimensions mismatch"));

  }

  if( in->get_number_of_dimensions() < D ){
    std::stringstream ss;
    ss << "crop: number of image dimensions should be at least " << D;

    BOOST_THROW_EXCEPTION(runtime_error(ss.str()));
  }

  typename uintd<D>::Type matrix_size_in = vector_to_uintd<D>( *in->get_dimensions() );
  typename uintd<D>::Type matrix_size_out = vector_to_uintd<D>( *out->get_dimensions() );
 
  unsigned int number_of_batches = 1;
  for( unsigned int d=D; d<in->get_number_of_dimensions(); d++ ){
    number_of_batches *= in->get_size(d);
  }

  if( weak_greater(offset+matrix_size_out, matrix_size_in) ){
    BOOST_THROW_EXCEPTION(runtime_error( "crop: cropping size mismatch"));

  }

  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<T> *in_int, *out_int;

  // Perform device copy if array is not residing on the current device
  prepare<2,T,T,dummy>( compute_device, &cur_device, &old_device, in, &in_int, out, &out_int );
  
  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, prod(matrix_size_out), &blockDim, &gridDim, number_of_batches );

  // Invoke kernel
  crop_kernel<T,D><<< gridDim, blockDim >>>
    ( offset, matrix_size_in, matrix_size_out, in_int->get_data_ptr(), out_int->get_data_ptr(), number_of_batches, prod(matrix_size_out) );
 
  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<2,T,dummy,T,dummy>( old_device, in, in_int, 2, compute_device, 0x0, out, out_int );

}

// Expand and zero fill
template<class T, unsigned int D> __global__ void
expand_with_zero_fill_kernel( vector_td<unsigned int,D> matrix_size_in, vector_td<unsigned int,D> matrix_size_out,
				    T *in, T *out, unsigned int number_of_batches, unsigned int num_elements )
{
	typedef vector_td<unsigned int,D> uintd;
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
  const unsigned int frame_offset = idx/num_elements;

  if( idx < num_elements*number_of_batches ){

    const uintd co_out = idx_to_co<D>( idx-frame_offset*num_elements, matrix_size_out );
    const uintd offset = (matrix_size_out-matrix_size_in)>>1;
    T _out;
    bool inside = (co_out>=offset) && (co_out<(matrix_size_in+offset));

    if( inside )
      _out = in[co_to_idx<D>(co_out-offset, matrix_size_in)+frame_offset*prod(matrix_size_in)];
    else{      
      _out = T(0);
    }

    out[idx] = _out;
  }
}

// Expand and zero fill
template<class T, unsigned int D> 
void Gadgetron::expand_with_zero_fill( cuNDArray<T> *in, cuNDArray<T> *out,
				  cuNDA_device compute_device )
{ 
  if( in == 0x0 || out == 0x0 ){
  	BOOST_THROW_EXCEPTION(runtime_error("zero_fill: 0x0 ndarray provided"));

  }

  if( in->get_number_of_dimensions() != out->get_number_of_dimensions() ){
  	BOOST_THROW_EXCEPTION(runtime_error("zero_fill: image dimensions mismatch"));

  }

  if( in->get_number_of_dimensions() < D ){
  	std::stringstream ss;
    ss << "zero_fill: number of image dimensions should be at least " << D;
    BOOST_THROW_EXCEPTION(runtime_error(ss.str()));

  }

  typename uintd<D>::Type matrix_size_in = vector_to_uintd<D>( *in->get_dimensions() );
  typename uintd<D>::Type matrix_size_out = vector_to_uintd<D>( *out->get_dimensions() );
  
  unsigned int number_of_batches = 1;
  for( unsigned int d=D; d<in->get_number_of_dimensions(); d++ ){
    number_of_batches *= in->get_size(d);
  }

  if( weak_greater(matrix_size_in,matrix_size_out) ){
    std::runtime_error("expand: size mismatch, cannot expand");

  }
 
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<T> *in_int, *out_int;

  // Perform device copy if array is not residing on the current device
  prepare<2,T,T,dummy>( compute_device, &cur_device, &old_device, in, &in_int, out, &out_int );

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, prod(matrix_size_out), &blockDim, &gridDim, number_of_batches );
 
  // Invoke kernel
  expand_with_zero_fill_kernel<T,D><<< gridDim, blockDim >>> ( matrix_size_in, matrix_size_out, in_int->get_data_ptr(), out_int->get_data_ptr(), number_of_batches, prod(matrix_size_out) );
 
  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<2,T,dummy,T,dummy>( old_device, in, in_int, 2, compute_device, 0x0, out, out_int );

}

// Zero fill border (rectangular)
template<class T, unsigned int D> __global__ void
zero_fill_border_kernel( vector_td<unsigned int,D> matrix_size_in, vector_td<unsigned int,D> matrix_size_out,
			       T *image, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

  if( idx < number_of_elements ){
    const vector_td<unsigned int,D> co_out = idx_to_co<D>( idx, matrix_size_out );
    const vector_td<unsigned int,D> offset = (matrix_size_out-matrix_size_in)>>1;
    if( weak_less( co_out, offset ) || weak_greater_equal( co_out, matrix_size_in+offset ) ){
      T zero = T(0);
      for( unsigned int batch=0; batch<number_of_batches; batch++ ){
	image[idx+batch*number_of_elements] = zero;
      }
    }
    else
      ; // do nothing
  }
}

// Zero fill border (rectangular)
template<class T, unsigned int D> 
void Gadgetron::zero_fill_border( typename uintd<D>::Type matrix_size_in, cuNDArray<T> *in_out,
			     cuNDA_device compute_device )
{ 
  typename uintd<D>::Type matrix_size_out = vector_to_uintd<D>( *in_out->get_dimensions() );
 
  if( weak_greater(matrix_size_in, matrix_size_out) ){
    BOOST_THROW_EXCEPTION(runtime_error("zero_fill: size mismatch, cannot zero fill"));

  }
 
  unsigned int number_of_batches = 1;
  for( unsigned int d=D; d<in_out->get_number_of_dimensions(); d++ ){
    number_of_batches *= in_out->get_size(d);
  }

 // Prepare internal array
  int cur_device, old_device;
  cuNDArray<T> *in_out_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,T,dummy,dummy>( compute_device, &cur_device, &old_device, in_out, &in_out_int ) ;

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, prod(matrix_size_out), &blockDim, &gridDim );
 
  // Invoke kernel
  zero_fill_border_kernel<T,D><<< gridDim, blockDim >>>
  		( matrix_size_in, matrix_size_out, in_out_int->get_data_ptr(), number_of_batches, prod(matrix_size_out) );
 
  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<1,T,dummy,dummy,dummy>( old_device, in_out, in_out_int, 1 );


}

// Zero fill border (circular)
template<class REAL, class T, unsigned int D> __global__ void
zero_fill_border_kernel( REAL radius, vector_td<int,D> dims, T *image,
			       unsigned int num_batches, unsigned int num_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
  const unsigned int frame_offset = idx/num_elements;
  
  if( idx < num_elements*num_batches ){
    const vector_td<int,D> co = idx_to_co<D>( idx-frame_offset*num_elements, dims ) - (dims>>1);
    if( REAL(norm_squared(co)) > radius*radius )
      image[idx] = T(0);
  }
}

// Zero fill border (circular, 2D)
template<class REAL, class T, unsigned int D> 
void Gadgetron::zero_fill_border( REAL radius, cuNDArray<T> *in_out,
			     cuNDA_device compute_device )
{

  
  unsigned int number_of_batches = 1;
  for( unsigned int d=2; d<in_out->get_number_of_dimensions(); d++ ){
    number_of_batches *= in_out->get_size(d);
  }

  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<T> *in_out_int;

  // Perform device copy if array is not residing on the current device
  prepare<1,T,dummy,dummy>( compute_device, &cur_device, &old_device, in_out, &in_out_int );

  typename intd<D>::Type dims = vector_to_intd<D>(*in_out->get_dimensions());

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, in_out->get_number_of_elements(), &blockDim, &gridDim, number_of_batches );
 
  // Invoke kernel
  zero_fill_border_kernel<REAL,T,D><<< gridDim, blockDim >>>
    ( radius, dims, in_out_int->get_data_ptr(), number_of_batches, prod(dims) );
 
  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<1,T,dummy,dummy,dummy>( old_device, in_out, in_out_int, 1 );


}

// Shrinkage
//

template<class REAL, class T> __global__ void 
shrink1_kernel( REAL gamma, T *in, T *out, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
 
  if( idx<number_of_elements ){
    T in_val = in[idx]; 
    REAL in_norm = abs(in_val);
    T _res;
    if( in_norm > REAL(0) )
      _res =  in_val/in_norm;
    else
      _res = T(0);
    REAL maximum = max( in_norm-gamma, REAL(0) );
    T res = maximum*_res;

    out[idx] = res;
  }
}

template<class REAL, class T> EXPORTGPUCORE
void Gadgetron::shrink1( REAL gamma, cuNDArray<T> *in, cuNDArray<T> *out )
{
  // TODO: multi-device handling

  if( !in || !out ){
    BOOST_THROW_EXCEPTION(runtime_error( "shrink1: 0x0 arrays not accepted" ));

  }

  if( in->get_number_of_elements() != out->get_number_of_elements() ){
    BOOST_THROW_EXCEPTION(runtime_error( "shrink1: i/o arrays must have an identical number of elements"));
  }
  
  // Get current Cuda device
  int cur_device;
  if( hipGetDevice(&cur_device) != hipSuccess ) {
    BOOST_THROW_EXCEPTION(runtime_error( "shrink1 : unable to get device no"));
  }

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, in->get_number_of_elements(), &blockDim, &gridDim );
  
  // Invoke kernel
  shrink1_kernel<REAL,T><<< gridDim, blockDim >>>( gamma, in->get_data_ptr(), out->get_data_ptr(), in->get_number_of_elements() );
  
  CHECK_FOR_CUDA_ERROR();
  

}

template<class REAL, class T> __global__ void 
shrinkd_kernel( REAL gamma, REAL *s_k, T *in, T *out, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
 
  if( idx<number_of_elements ){
    T in_val = in[idx]; 
    REAL s_k_val = s_k[idx];
    T _res;
    if( s_k_val > 0 )
      _res =  in_val/s_k_val;
    else
      _res = T(0);
    REAL maximum = max( s_k_val-gamma, REAL(0) );
    T res = maximum*_res;

    out[idx] = res;
  }
}

template<class REAL, class T> EXPORTGPUCORE
void Gadgetron::shrinkd( REAL gamma, cuNDArray<REAL> *s_k, cuNDArray<T> *in, cuNDArray<T> *out )
{
  // TODO: multi-device handling

  if( !in || !out || !s_k ){
    BOOST_THROW_EXCEPTION(runtime_error( "shrinkd: 0x0 arrays not accepted"));

  }

  if( in->get_number_of_elements() != out->get_number_of_elements() ){
    BOOST_THROW_EXCEPTION(runtime_error( "shrinkd: i/o arrays must have an identical number of elements"));

  }

  if( in->get_number_of_elements() != s_k->get_number_of_elements() ){
    BOOST_THROW_EXCEPTION(runtime_error( "shrinkd: i/o arrays must have an identical number of elements"));

  }
  
  // Get current Cuda device
  int cur_device;
  if( hipGetDevice(&cur_device) != hipSuccess ) {
    BOOST_THROW_EXCEPTION(runtime_error( "shrinkd : unable to get device no"));

  }

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, in->get_number_of_elements(), &blockDim, &gridDim );
  
  // Invoke kernel
  shrinkd_kernel<REAL,T><<< gridDim, blockDim >>>( gamma, s_k->get_data_ptr(), in->get_data_ptr(), out->get_data_ptr(), in->get_number_of_elements() );
  
  CHECK_FOR_CUDA_ERROR();

}

// Mirror, but keep the origin unchanged
template<class T, unsigned int D> __global__ void
origin_mirror_kernel( vector_td<unsigned int,D> matrix_size, vector_td<unsigned int,D> origin, T *in, T *out, bool zero_fill )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;

  if( idx < prod(matrix_size) ){

  	vector_td<unsigned int,D> in_co = idx_to_co<D>( idx, matrix_size );
  	vector_td<unsigned int,D> out_co = matrix_size-in_co;
    
    bool wrap = false;
    for( unsigned int d=0; d<D; d++ ){
      if( out_co.vec[d] == matrix_size.vec[d] ){
	out_co.vec[d] = 0;
	wrap = true;
      }
    }
    
    const unsigned int in_idx = co_to_idx<D>(in_co, matrix_size);
    const unsigned int out_idx = co_to_idx<D>(out_co, matrix_size);

    if( wrap && zero_fill )
      out[out_idx] = T(0);
    else
      out[out_idx] = in[in_idx];
  }
}

// Mirror around the origin -- !! leaving the origin unchanged !!
// This creates empty space "on the left" that can be filled by zero (default) or the left-over entry.
template<class T, unsigned int D> EXPORTGPUCORE
void Gadgetron::origin_mirror( cuNDArray<T> *in, cuNDArray<T> *out, bool zero_fill, cuNDA_device compute_device )
{
  if( in == 0x0 || out == 0x0 ){
  	BOOST_THROW_EXCEPTION(runtime_error( "origin_mirror: 0x0 ndarray provided"));

  }

  if( !in->dimensions_equal(out) ){
  	BOOST_THROW_EXCEPTION(runtime_error("origin_mirror: image dimensions mismatch"));

  }
  
  if( in->get_number_of_dimensions() != D ){
  	std::stringstream ss;
    ss << "origin_mirror: number of image dimensions is not " << D;
    BOOST_THROW_EXCEPTION(runtime_error(ss.str()));
  }

  typename uintd<D>::Type matrix_size = vector_to_uintd<D>( *in->get_dimensions() );
 
  // Prepare internal array
  int cur_device, old_device;
  cuNDArray<T> *in_int, *out_int;

  // Perform device copy if array is not residing on the current device
  prepare<2,T,T,dummy>( compute_device, &cur_device, &old_device, in, &in_int, out, &out_int );
  
  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, prod(matrix_size), &blockDim, &gridDim );

  // Invoke kernel
  origin_mirror_kernel<T,D><<< gridDim, blockDim >>> ( matrix_size, matrix_size>>1, in_int->get_data_ptr(), out_int->get_data_ptr(), zero_fill );
 
  CHECK_FOR_CUDA_ERROR();

  // Restore
  restore<2,T,dummy,T,dummy>( old_device, in, in_int, 2, compute_device, 0x0, out, out_int ) ;

}



// Minimum
//
template<class T> __global__ 
void minimum_kernel( T* in1,T* in2, T* out, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
 
  if( idx<number_of_elements ){
    out[idx]=min(in1[idx],in2[idx]);
  }
} 


// Minimum
//
template<class T>  
boost::shared_ptr< cuNDArray<T> >
Gadgetron::minimum( cuNDArray<T> *in1,cuNDArray<T> *in2,
	    cuNDA_device alloc_device, cuNDA_device compute_device )
{
  int cur_device, old_device;
  cuNDArray<T> *in1_int;
  cuNDArray<T> *in2_int;


  if ( in1->get_number_of_elements() !=  in2->get_number_of_elements()){
    BOOST_THROW_EXCEPTION(runtime_error( "minimum: input arrays have different number of elements"));

  }
  // Prepare 
  prepare<2,T,T,dummy>( compute_device, &cur_device, &old_device, in1, &in1_int, in2, &in2_int);

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, in1->get_number_of_elements(), &blockDim, &gridDim );

  // Invoke kernel
  boost::shared_ptr< cuNDArray<T> > out = cuNDArray<T>::allocate(in1->get_dimensions().get());
  if( out.get() != 0x0 ) minimum_kernel<T><<< gridDim, blockDim >>>( in1_int->get_data_ptr(), in2_int->get_data_ptr(),out->get_data_ptr(), in1->get_number_of_elements() );

  CHECK_FOR_CUDA_ERROR();

  // Restore 
  // Restore
  restore<2,T,T,T,dummy>( old_device, in1, in1_int, 0, compute_device, out.get(), in2, in2_int );

  return out;
}



// Maximum
//
template<class T> __global__ 
void maximum_kernel( T* in1,T* in2, T* out, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.y*gridDim.x*blockDim.x + blockIdx.x*blockDim.x+threadIdx.x;
 
  if( idx<number_of_elements ){
    out[idx]=max(in1[idx],in2[idx]);
  }
} 


// Minimum
//
template<class T>  
boost::shared_ptr< cuNDArray<T> >
Gadgetron::maximum( cuNDArray<T> *in1,cuNDArray<T> *in2,
	    cuNDA_device alloc_device, cuNDA_device compute_device )
{
  int cur_device, old_device;
  cuNDArray<T> *in1_int;
  cuNDArray<T> *in2_int;


  if ( in1->get_number_of_elements() !=  in2->get_number_of_elements()){
    BOOST_THROW_EXCEPTION(runtime_error( "maximum: input arrays have different number of elements"));
  }
  // Prepare 
  prepare<2,T,T,dummy>( compute_device, &cur_device, &old_device, in1, &in1_int, in2, &in2_int  ) ;

  // Setup block/grid dimensions
  dim3 blockDim; dim3 gridDim;
  setup_grid( cur_device, in1->get_number_of_elements(), &blockDim, &gridDim );

  // Invoke kernel
  boost::shared_ptr< cuNDArray<T> > out = cuNDArray<T>::allocate(in1->get_dimensions().get());
  if( out.get() != 0x0 ) maximum_kernel<T><<< gridDim, blockDim >>>( in1_int->get_data_ptr(), in2_int->get_data_ptr(),out->get_data_ptr(), in1->get_number_of_elements() );

  CHECK_FOR_CUDA_ERROR();

  // Restore 
  // Restore
  restore<2,T,T,T,dummy>( old_device, in1, in1_int, 0, compute_device, out.get(), in2, in2_int );

  return out;
}


//
// Instantiation
//

// A few functions have integer support

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<int> >
Gadgetron::sum<int>( cuNDArray<int>*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<intd<1>::Type> >
Gadgetron::sum<intd<1>::Type >( cuNDArray<intd<1>::Type >*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<intd<2>::Type> >
Gadgetron::sum<intd<2>::Type >( cuNDArray<intd<2>::Type >*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<intd<3>::Type> >
Gadgetron::sum<intd<3>::Type >( cuNDArray<intd<3>::Type >*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<intd<4>::Type> >
Gadgetron::sum<intd<4>::Type >( cuNDArray<intd<4>::Type >*, unsigned int, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<unsigned int> >
Gadgetron::sum<unsigned int>( cuNDArray<unsigned int>*, unsigned int, cuNDA_device, cuNDA_device);
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<uintd<1>::Type> >
Gadgetron::sum<uintd<1>::Type>( cuNDArray<uintd<1>::Type>*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<uintd<2>::Type> >
Gadgetron::sum<uintd<2>::Type>( cuNDArray<uintd<2>::Type>*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<uintd<3>::Type> >
Gadgetron::sum<uintd<3>::Type>( cuNDArray<uintd<3>::Type>*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<uintd<4>::Type> >
Gadgetron::sum<uintd<4>::Type>( cuNDArray<uintd<4>::Type>*, unsigned int, cuNDA_device, cuNDA_device );


template EXPORTGPUCORE void
Gadgetron::crop<int,1>( uintd1, cuNDArray<int>*, cuNDArray<int>*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,1>,1>( uintd1, cuNDArray<vector_td<int,1> >*, cuNDArray<vector_td<int,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,2>,1>( uintd1, cuNDArray<vector_td<int,2> >*, cuNDArray<vector_td<int,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,3>,1>( uintd1, cuNDArray<vector_td<int,3> >*, cuNDArray<vector_td<int,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,4>,1>( uintd1, cuNDArray<vector_td<int,4> >*, cuNDArray<vector_td<int,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,1>,2>( uintd2, cuNDArray<vector_td<int,1> >*, cuNDArray<vector_td<int,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,2>,2>( uintd2, cuNDArray<vector_td<int,2> >*, cuNDArray<vector_td<int,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,3>,2>( uintd2, cuNDArray<vector_td<int,3> >*, cuNDArray<vector_td<int,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,4>,2>( uintd2, cuNDArray<vector_td<int,4> >*, cuNDArray<vector_td<int,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,1>,3>( uintd3, cuNDArray<vector_td<int,1> >*, cuNDArray<vector_td<int,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,2>,3>( uintd3, cuNDArray<vector_td<int,2> >*, cuNDArray<vector_td<int,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,3>,3>( uintd3, cuNDArray<vector_td<int,3> >*, cuNDArray<vector_td<int,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,4>,3>( uintd3, cuNDArray<vector_td<int,4> >*, cuNDArray<vector_td<int,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,1>,4>( uintd4, cuNDArray<vector_td<int,1> >*, cuNDArray<vector_td<int,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,2>,4>( uintd4, cuNDArray<vector_td<int,2> >*, cuNDArray<vector_td<int,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,3>,4>( uintd4, cuNDArray<vector_td<int,3> >*, cuNDArray<vector_td<int,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<int,4>,4>( uintd4, cuNDArray<vector_td<int,4> >*, cuNDArray<vector_td<int,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<unsigned int,1>( uintd1, cuNDArray<unsigned int>*, cuNDArray<unsigned int>*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,1>,1>( uintd1, cuNDArray<vector_td<unsigned int,1> >*, cuNDArray<vector_td<unsigned int,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,2>,1>( uintd1, cuNDArray<vector_td<unsigned int,2> >*, cuNDArray<vector_td<unsigned int,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,3>,1>( uintd1, cuNDArray<vector_td<unsigned int,3> >*, cuNDArray<vector_td<unsigned int,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,4>,1>( uintd1, cuNDArray<vector_td<unsigned int,4> >*, cuNDArray<vector_td<unsigned int,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,1>,2>( uintd2, cuNDArray<vector_td<unsigned int,1> >*, cuNDArray<vector_td<unsigned int,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,2>,2>( uintd2, cuNDArray<vector_td<unsigned int,2> >*, cuNDArray<vector_td<unsigned int,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,3>,2>( uintd2, cuNDArray<vector_td<unsigned int,3> >*, cuNDArray<vector_td<unsigned int,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,4>,2>( uintd2, cuNDArray<vector_td<unsigned int,4> >*, cuNDArray<vector_td<unsigned int,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,1>,3>( uintd3, cuNDArray<vector_td<unsigned int,1> >*, cuNDArray<vector_td<unsigned int,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,2>,3>( uintd3, cuNDArray<vector_td<unsigned int,2> >*, cuNDArray<vector_td<unsigned int,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,3>,3>( uintd3, cuNDArray<vector_td<unsigned int,3> >*, cuNDArray<vector_td<unsigned int,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,4>,3>( uintd3, cuNDArray<vector_td<unsigned int,4> >*, cuNDArray<vector_td<unsigned int,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,1>,4>( uintd4, cuNDArray<vector_td<unsigned int,1> >*, cuNDArray<vector_td<unsigned int,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,2>,4>( uintd4, cuNDArray<vector_td<unsigned int,2> >*, cuNDArray<vector_td<unsigned int,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,3>,4>( uintd4, cuNDArray<vector_td<unsigned int,3> >*, cuNDArray<vector_td<unsigned int,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<unsigned int,4>,4>( uintd4, cuNDArray<vector_td<unsigned int,4> >*, cuNDArray<vector_td<unsigned int,4> >*, cuNDA_device );

// Instanciation -- single precision

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::sum<float>( cuNDArray<float>*, unsigned int, cuNDA_device, cuNDA_device);

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<complext<float> > >
Gadgetron::sum<complext<float> >( cuNDArray<complext<float> >*, unsigned int, cuNDA_device, cuNDA_device);

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<floatd<1>::Type> >
Gadgetron::sum<floatd<1>::Type>( cuNDArray<floatd<1>::Type>*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<floatd<2>::Type> >
Gadgetron::sum<floatd<2>::Type>( cuNDArray<floatd<2>::Type>*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<floatd<3>::Type> >
Gadgetron::sum<floatd<3>::Type>( cuNDArray<floatd<3>::Type>*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<floatd<4>::Type> >
Gadgetron::sum<floatd<4>::Type>( cuNDArray<floatd<4>::Type>*, unsigned int, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::expand<float>( cuNDArray<float>*, unsigned int, cuNDA_device, cuNDA_device);
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> >
Gadgetron::expand<float_complext>( cuNDArray<float_complext>*, unsigned int, cuNDA_device, cuNDA_device);


template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::abs<float_complext>( cuNDArray<float_complext>*, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::abs<float>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::minimum<float>( cuNDArray<float>*, cuNDArray<float>*, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::maximum<float>( cuNDArray<float>*, cuNDArray<float>*,cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::correlation<float>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> >
Gadgetron::correlation<float_complext>( cuNDArray<float_complext>*, cuNDA_device, cuNDA_device );



template EXPORTGPUCORE void
Gadgetron::crop<float,1>( uintd1, cuNDArray<float>*, cuNDArray<float>*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<float,2>( uintd2, cuNDArray<float>*, cuNDArray<float>*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<complext<float>,1>( uintd1, cuNDArray<complext<float> >*, cuNDArray< complext<float> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<complext<float>,2>( uintd2, cuNDArray<complext<float> >*, cuNDArray< complext<float> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<complext<float>,3>( uintd3, cuNDArray<complext<float> >*, cuNDArray< complext<float> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<complext<float>,4>( uintd4, cuNDArray<complext<float> >*, cuNDArray< complext<float> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,1>,1>( uintd1, cuNDArray<vector_td<float,1> >*, cuNDArray<vector_td<float,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,2>,1>( uintd1, cuNDArray<vector_td<float,2> >*, cuNDArray<vector_td<float,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,3>,1>( uintd1, cuNDArray<vector_td<float,3> >*, cuNDArray<vector_td<float,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,4>,1>( uintd1, cuNDArray<vector_td<float,4> >*, cuNDArray<vector_td<float,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,1>,2>( uintd2, cuNDArray<vector_td<float,1> >*, cuNDArray<vector_td<float,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,2>,2>( uintd2, cuNDArray<vector_td<float,2> >*, cuNDArray<vector_td<float,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,3>,2>( uintd2, cuNDArray<vector_td<float,3> >*, cuNDArray<vector_td<float,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,4>,2>( uintd2, cuNDArray<vector_td<float,4> >*, cuNDArray<vector_td<float,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,1>,3>( uintd3, cuNDArray<vector_td<float,1> >*, cuNDArray<vector_td<float,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,2>,3>( uintd3, cuNDArray<vector_td<float,2> >*, cuNDArray<vector_td<float,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,3>,3>( uintd3, cuNDArray<vector_td<float,3> >*, cuNDArray<vector_td<float,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,4>,3>( uintd3, cuNDArray<vector_td<float,4> >*, cuNDArray<vector_td<float,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,1>,4>( uintd4, cuNDArray<vector_td<float,1> >*, cuNDArray<vector_td<float,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,2>,4>( uintd4, cuNDArray<vector_td<float,2> >*, cuNDArray<vector_td<float,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,3>,4>( uintd4, cuNDArray<vector_td<float,3> >*, cuNDArray<vector_td<float,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<float,4>,4>( uintd4, cuNDArray<vector_td<float,4> >*, cuNDArray<vector_td<float,4> >*, cuNDA_device );


template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<float,1>( cuNDArray<float>*, cuNDArray<float>*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<float_complext,1>( cuNDArray<float_complext>*, cuNDArray<float_complext>*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<float,2>( cuNDArray<float>*, cuNDArray<float>*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<float_complext,2>( cuNDArray<float_complext>*, cuNDArray<float_complext>*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<float,3>( cuNDArray<float>*, cuNDArray<float>*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<float_complext,3>( cuNDArray<float_complext>*, cuNDArray<float_complext>*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<float,4>( cuNDArray<float>*, cuNDArray<float>*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<float_complext,4>( cuNDArray<float_complext>*, cuNDArray<float_complext>*, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float_complext> >
Gadgetron::real_to_complext<float>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::complext_to_real<float>( cuNDArray<float_complext>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::downsample<float,1>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::downsample<float,2>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::downsample<float,3>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::downsample<float,4>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::upsample_nn<float,1>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::upsample_nn<float,2>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::upsample_nn<float,3>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::upsample_nn<float,4>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::upsample_lin<float,1>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::upsample_lin<float,2>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::upsample_lin<float,3>( cuNDArray<float>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<float> >
Gadgetron::upsample_lin<float,4>( cuNDArray<float>*, cuNDA_device, cuNDA_device );


template EXPORTGPUCORE void Gadgetron::rss_normalize<float>( cuNDArray<float>*, unsigned int, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::rss_normalize<float_complext>( cuNDArray<float_complext>*, unsigned int, cuNDA_device );


template EXPORTGPUCORE void Gadgetron::zero_fill_border<float,1>(uintd1, cuNDArray<float>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<float_complext,1>(uintd1, cuNDArray<float_complext>*, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<float,2>(uintd2, cuNDArray<float>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<float_complext,2>(uintd2, cuNDArray<float_complext>*, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<float,3>(uintd3, cuNDArray<float>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<float_complext,3>(uintd3, cuNDArray<float_complext>*, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<float,4>(uintd4, cuNDArray<float>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<float_complext,4>(uintd4, cuNDArray<float_complext>*, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<float,float,2>(float, cuNDArray<float>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<float,float_complext,2>(float, cuNDArray<float_complext>*, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<float,float,3>(float, cuNDArray<float>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<float,float_complext,3>(float, cuNDArray<float_complext>*, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<float,float,4>(float, cuNDArray<float>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<float,float_complext,4>(float, cuNDArray<float_complext>*, cuNDA_device );



template EXPORTGPUCORE void Gadgetron::shrink1<float,float>( float, cuNDArray<float>*, cuNDArray<float>* );
template EXPORTGPUCORE void Gadgetron::shrink1<float,float_complext>( float, cuNDArray<float_complext>*, cuNDArray<float_complext>* );

template EXPORTGPUCORE void Gadgetron::shrinkd<float,float>( float, cuNDArray<float>*, cuNDArray<float>*, cuNDArray<float>* );
template EXPORTGPUCORE void Gadgetron::shrinkd<float,float_complext>( float, cuNDArray<float>*, cuNDArray<float_complext>*, cuNDArray<float_complext>* );

template EXPORTGPUCORE
void Gadgetron::origin_mirror<float,1>(cuNDArray<float>*, cuNDArray<float>*, bool, cuNDA_device);
template EXPORTGPUCORE
void Gadgetron::origin_mirror<float,2>(cuNDArray<float>*, cuNDArray<float>*, bool, cuNDA_device);
template EXPORTGPUCORE
void Gadgetron::origin_mirror<float,3>(cuNDArray<float>*, cuNDArray<float>*, bool, cuNDA_device);
template EXPORTGPUCORE
void Gadgetron::origin_mirror<float,4>(cuNDArray<float>*, cuNDArray<float>*, bool, cuNDA_device);

template EXPORTGPUCORE void
Gadgetron::origin_mirror<float_complext,1>(cuNDArray<float_complext>*, cuNDArray<float_complext>*, bool, cuNDA_device);
template EXPORTGPUCORE void
Gadgetron::origin_mirror<float_complext,2>(cuNDArray<float_complext>*, cuNDArray<float_complext>*, bool, cuNDA_device);
template EXPORTGPUCORE void
Gadgetron::origin_mirror<float_complext,3>(cuNDArray<float_complext>*, cuNDArray<float_complext>*, bool, cuNDA_device);
template EXPORTGPUCORE void
Gadgetron::origin_mirror<float_complext,4>(cuNDArray<float_complext>*, cuNDArray<float_complext>*, bool, cuNDA_device);


// Instanciation -- double precision

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::sum<double>( cuNDArray<double>*, unsigned int, cuNDA_device, cuNDA_device);

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext > >
Gadgetron::sum<complext<double> >( cuNDArray<double_complext >*, unsigned int, cuNDA_device, cuNDA_device);

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<doubled<1>::Type> >
Gadgetron::sum<doubled<1>::Type>( cuNDArray<doubled<1>::Type>*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<doubled<2>::Type> >
Gadgetron::sum<doubled<2>::Type>( cuNDArray<doubled<2>::Type>*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<doubled<3>::Type> >
Gadgetron::sum<doubled<3>::Type>( cuNDArray<doubled<3>::Type>*, unsigned int, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<doubled<4>::Type> >
Gadgetron::sum<doubled<4>::Type>( cuNDArray<doubled<4>::Type>*, unsigned int, cuNDA_device, cuNDA_device );



template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::expand<double>( cuNDArray<double>*, unsigned int, cuNDA_device, cuNDA_device);
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> >
Gadgetron::expand<double_complext>( cuNDArray<double_complext>*, unsigned int, cuNDA_device, cuNDA_device);


template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::abs<double_complext>( cuNDArray<double_complext>*, cuNDA_device, cuNDA_device );
template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::abs<double>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::correlation<double>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> >
Gadgetron::correlation<double_complext>( cuNDArray<double_complext>*, cuNDA_device, cuNDA_device );

//template EXPORTGPUCORE void axpy<double>( cuNDArray<double>*, cuNDArray<double_complext>*, cuNDArray<double_complext>*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<double,1>( uintd1, cuNDArray<double>*, cuNDArray<double>*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<complext<double> ,1>( uintd1, cuNDArray<complext<double> >*, cuNDArray< complext<double> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<complext<double> ,2>( uintd2, cuNDArray<complext<double> >*, cuNDArray< complext<double> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<complext<double> ,3>( uintd3, cuNDArray<complext<double> >*, cuNDArray< complext<double> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<complext<double> ,4>( uintd4, cuNDArray<complext<double> >*, cuNDArray< complext<double> >*, cuNDA_device );



template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,1>,1>( uintd1, cuNDArray<vector_td<double,1> >*, cuNDArray<vector_td<double,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,2>,1>( uintd1, cuNDArray<vector_td<double,2> >*, cuNDArray<vector_td<double,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,3>,1>( uintd1, cuNDArray<vector_td<double,3> >*, cuNDArray<vector_td<double,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,4>,1>( uintd1, cuNDArray<vector_td<double,4> >*, cuNDArray<vector_td<double,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,1>,2>( uintd2, cuNDArray<vector_td<double,1> >*, cuNDArray<vector_td<double,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,2>,2>( uintd2, cuNDArray<vector_td<double,2> >*, cuNDArray<vector_td<double,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,3>,2>( uintd2, cuNDArray<vector_td<double,3> >*, cuNDArray<vector_td<double,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,4>,2>( uintd2, cuNDArray<vector_td<double,4> >*, cuNDArray<vector_td<double,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,1>,3>( uintd3, cuNDArray<vector_td<double,1> >*, cuNDArray<vector_td<double,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,2>,3>( uintd3, cuNDArray<vector_td<double,2> >*, cuNDArray<vector_td<double,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,3>,3>( uintd3, cuNDArray<vector_td<double,3> >*, cuNDArray<vector_td<double,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,4>,3>( uintd3, cuNDArray<vector_td<double,4> >*, cuNDArray<vector_td<double,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,1>,4>( uintd4, cuNDArray<vector_td<double,1> >*, cuNDArray<vector_td<double,1> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,2>,4>( uintd4, cuNDArray<vector_td<double,2> >*, cuNDArray<vector_td<double,2> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,3>,4>( uintd4, cuNDArray<vector_td<double,3> >*, cuNDArray<vector_td<double,3> >*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::crop<vector_td<double,4>,4>( uintd4, cuNDArray<vector_td<double,4> >*, cuNDArray<vector_td<double,4> >*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<double,1>( cuNDArray<double>*, cuNDArray<double>*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<double_complext,1>( cuNDArray<double_complext>*, cuNDArray<double_complext>*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<double,2>( cuNDArray<double>*, cuNDArray<double>*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<double_complext,2>( cuNDArray<double_complext>*, cuNDArray<double_complext>*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<double,3>( cuNDArray<double>*, cuNDArray<double>*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<double_complext,3>( cuNDArray<double_complext>*, cuNDArray<double_complext>*, cuNDA_device );

template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<double,4>( cuNDArray<double>*, cuNDArray<double>*, cuNDA_device );
template EXPORTGPUCORE void
Gadgetron::expand_with_zero_fill<double_complext,4>( cuNDArray<double_complext>*, cuNDArray<double_complext>*, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double_complext> >
Gadgetron::real_to_complext<double>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::complext_to_real<double>( cuNDArray<double_complext>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::downsample<double,1>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::downsample<double,2>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::downsample<double,3>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::downsample<double,4>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::upsample_nn<double,1>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::upsample_nn<double,2>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::upsample_nn<double,3>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::upsample_nn<double,4>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::upsample_lin<double,1>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::upsample_lin<double,2>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::upsample_lin<double,3>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE boost::shared_ptr< cuNDArray<double> >
Gadgetron::upsample_lin<double,4>( cuNDArray<double>*, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::rss_normalize<double>( cuNDArray<double>*, unsigned int, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::rss_normalize<double_complext>( cuNDArray<double_complext>*, unsigned int, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<double,1>(uintd1, cuNDArray<double>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<double_complext,1>(uintd1, cuNDArray<double_complext>*, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<double,2>(uintd2, cuNDArray<double>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<double_complext,2>(uintd2, cuNDArray<double_complext>*, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<double,3>(uintd3, cuNDArray<double>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<double_complext,3>(uintd3, cuNDArray<double_complext>*, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<double,4>(uintd4, cuNDArray<double>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<double_complext,4>(uintd4, cuNDArray<double_complext>*, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<double,double,2>(double, cuNDArray<double>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<double,double_complext,2>(double, cuNDArray<double_complext>*, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<double,double,3>(double, cuNDArray<double>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<double,double_complext,3>(double, cuNDArray<double_complext>*, cuNDA_device );

template EXPORTGPUCORE void Gadgetron::zero_fill_border<double,double,4>(double, cuNDArray<double>*, cuNDA_device );
template EXPORTGPUCORE void Gadgetron::zero_fill_border<double,double_complext,4>(double, cuNDArray<double_complext>*, cuNDA_device );


template EXPORTGPUCORE void Gadgetron::shrink1<double,double>( double, cuNDArray<double>*, cuNDArray<double>* );
template EXPORTGPUCORE void Gadgetron::shrink1<double,double_complext>( double, cuNDArray<double_complext>*, cuNDArray<double_complext>* );

template EXPORTGPUCORE void Gadgetron::shrinkd<double,double>( double, cuNDArray<double>*, cuNDArray<double>*, cuNDArray<double>* );
template EXPORTGPUCORE void Gadgetron::shrinkd<double,double_complext>( double, cuNDArray<double>*, cuNDArray<double_complext>*, cuNDArray<double_complext>* );

template EXPORTGPUCORE
void Gadgetron::origin_mirror<double,1>(cuNDArray<double>*, cuNDArray<double>*, bool, cuNDA_device);
template EXPORTGPUCORE
void Gadgetron::origin_mirror<double,2>(cuNDArray<double>*, cuNDArray<double>*, bool, cuNDA_device);
template EXPORTGPUCORE
void Gadgetron::origin_mirror<double,3>(cuNDArray<double>*, cuNDArray<double>*, bool, cuNDA_device);
template EXPORTGPUCORE
void Gadgetron::origin_mirror<double,4>(cuNDArray<double>*, cuNDArray<double>*, bool, cuNDA_device);

template EXPORTGPUCORE void
Gadgetron::origin_mirror<double_complext,1>(cuNDArray<double_complext>*, cuNDArray<double_complext>*, bool, cuNDA_device);
template EXPORTGPUCORE void
Gadgetron::origin_mirror<double_complext,2>(cuNDArray<double_complext>*, cuNDArray<double_complext>*, bool, cuNDA_device);
template EXPORTGPUCORE void
Gadgetron::origin_mirror<double_complext,3>(cuNDArray<double_complext>*, cuNDArray<double_complext>*, bool, cuNDA_device);
template EXPORTGPUCORE void
Gadgetron::origin_mirror<double_complext,4>(cuNDArray<double_complext>*, cuNDArray<double_complext>*, bool, cuNDA_device);

template EXPORTGPUCORE
boost::shared_ptr< cuNDArray<float> >
Gadgetron::squaredNorm( cuNDArray<float> *, unsigned int, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE
boost::shared_ptr< cuNDArray<float> >
Gadgetron::squaredNorm( cuNDArray<float_complext> *, unsigned int ,cuNDA_device , cuNDA_device);

template EXPORTGPUCORE
boost::shared_ptr< cuNDArray<double> >
Gadgetron::squaredNorm( cuNDArray<double> *, unsigned int, cuNDA_device, cuNDA_device );

template EXPORTGPUCORE
boost::shared_ptr< cuNDArray<double> >
Gadgetron::squaredNorm( cuNDArray<double_complext> *, unsigned int ,cuNDA_device , cuNDA_device);

template EXPORTGPUCORE
void Gadgetron::inplace_sgn(cuNDArray<float>* x);
template EXPORTGPUCORE
void Gadgetron::inplace_sgn(cuNDArray<double>* x);


template void Gadgetron::clamp<double>(cuNDArray<double>*,double,double);
template void Gadgetron::clamp<float>(cuNDArray<float>*,float,float);

template void Gadgetron::clamp_min<double>(cuNDArray<double>*,double);
template void Gadgetron::clamp_min<float>(cuNDArray<float>*,float);

template void Gadgetron::clamp_max<double>(cuNDArray<double>*,double);
template void Gadgetron::clamp_max<float>(cuNDArray<float>*,float);

template void Gadgetron::clamp_min<double_complext>(cuNDArray<double_complext>*,double);
template void Gadgetron::clamp_min<float_complext>(cuNDArray<float_complext>*,float);



template float Gadgetron::normalize<float>( cuNDArray<float> *, float, cuNDA_device );
template double Gadgetron::normalize<double>( cuNDArray<double> *, double, cuNDA_device );
