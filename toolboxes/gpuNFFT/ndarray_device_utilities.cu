#include "hip/hip_runtime.h"
#include "ndarray_device_utilities.hcu"
#include "vectord_operators.hcu"
#include "vectord_utilities.hcu"
#include "check_CUDA.h"

#include <hipblas.h>
#include <vector>

using namespace std;

// Clear
template<class T> __global__ 
void cuNDA_clear_kernel( T *data, unsigned int number_of_elements )
{
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
 
  if( idx<number_of_elements ){
    T zero = get_zero<T>();
    data[idx] = zero;
  }
}

// Clear
template<class T> __host__
void cuNDA_clear( cuNDArray<T> *in )
{
  unsigned int number_of_elements = in->get_number_of_elements();

  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  // Make clear image
  cuNDA_clear_kernel<<< gridDim, blockDim >>>( in->get_data_ptr(), number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
}

// Abs
template<class T> __global__ 
void cuNDA_abs_kernel( T *in_out, unsigned int number_of_elements )
{
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
 
  if( idx<number_of_elements ){
    T val = in_out[idx]; 
    in_out[idx] = abs(val);
  }
}

// Abs
template<class T> __host__ 
void cuNDA_abs( cuNDArray<T> *in_out )
{
  unsigned int number_of_elements = in_out->get_number_of_elements();

  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));
 
  // Make modulus image
  cuNDA_abs_kernel<<< gridDim, blockDim >>>( in_out->get_data_ptr(), number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
}

// Reciprocal
template<class T> __global__ 
void cuNDA_reciprocal_kernel( T *data, unsigned int number_of_elements )
{
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
 
  if( idx<number_of_elements ){
    data[idx] = reciprocal<T>(data[idx]);
  }
}

// Reciprocal
template<class T> __host__
void cuNDA_reciprocal( cuNDArray<T> *in )
{
  unsigned int number_of_elements = in->get_number_of_elements();

  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  // Make reciprocal image
  cuNDA_reciprocal_kernel<<< gridDim, blockDim >>>( in->get_data_ptr(), number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
}

// Normalize (float)
__host__
void cuNDA_normalize( cuNDArray<float> *data, float new_max )
{
  unsigned int number_of_elements = data->get_number_of_elements();

  // Find the maximum value in the array
  int max_idx = hipblasIsamax (number_of_elements, data->get_data_ptr(), 1);

  // Copy that value back to host memory
  float max_val;
  hipMemcpy(&max_val, (data->get_data_ptr()+max_idx-1), sizeof(float), hipMemcpyDeviceToHost);

  // Scale the array
  hipblasSscal( number_of_elements, new_max/max_val, data->get_data_ptr(), 1 );

  CHECK_FOR_CUDA_ERROR();
}

// Normalize (double)
__host__
void cuNDA_normalize( cuNDArray<double> *data, double new_max )
{
  unsigned int number_of_elements = data->get_number_of_elements();

  // Find the maximum value in the array
  int max_idx = hipblasIdamax (number_of_elements, data->get_data_ptr(), 1);

  // Copy that value back to host memory
  double max_val;
  hipMemcpy(&max_val, (data->get_data_ptr()+max_idx-1), sizeof(double), hipMemcpyDeviceToHost);

  // Scale the array
  hipblasDscal( number_of_elements, new_max/max_val, data->get_data_ptr(), 1 );

  CHECK_FOR_CUDA_ERROR();
}

// Normalize
template<class REAL> __host__
void cuNDA_normalize( cuNDArray<REAL> *data, REAL new_max )
{
  cuNDA_normalize( data, new_max );
}

// Normalized RSS
template<class REAL, class T> __global__ void
cuNDA_rss_normalize_kernel( T *in_out, unsigned int stride, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( idx < number_of_elements ){

    unsigned int in_idx = (idx/stride)*stride*number_of_batches+(idx%stride);

    REAL rss = get_zero<REAL>();
 
    for( unsigned int i=0; i<number_of_batches; i++ )
      rss += norm_squared(in_out[i*stride+in_idx]);
 
    rss = sqrt(rss); // TODO: overload neccesary?
    rss += get_epsilon<REAL>(); // avoid potential division by zero
    rss = reciprocal(rss);
 
    for( unsigned int i=0; i<number_of_batches; i++ ) {
      T out = in_out[i*stride+in_idx];
      out *= rss; // this works since rss is scalar
      in_out[i*stride+in_idx] = out; 
    } 
  }
}

// Normalized RSS
template<class REAL, class T> __host__
bool cuNDA_rss_normalize( cuNDArray<T> *in_out, unsigned int dim )
{
  if( !(in_out->get_number_of_dimensions()>1) ){
    cout << endl << "cuNDA_rss_normalized:: underdimensioned." << endl; 
    return false;
  }
 
  if( dim > in_out->get_number_of_dimensions()-1 ){
    cout << endl << "cuNDA_rss_normalized:: dimension out of range." << endl; 
    return false;
  }

  unsigned int number_of_batches = in_out->get_size(dim);
  unsigned int number_of_elements = in_out->get_number_of_elements()/number_of_batches;

  unsigned int stride = 1;
  for( unsigned int i=0; i<dim; i++ )
    stride *= in_out->get_size(i);

  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  // Make reciprocal image
  cuNDA_rss_normalize_kernel<REAL,T><<< gridDim, blockDim >>>( in_out->get_data_ptr(), stride, number_of_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
  return true;
}

// Scale
template<class A, class X> __global__ 
void cuNDA_scale1_kernel( A a, X *x, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if( idx < number_of_elements ){
    X in = x[idx];
    in = mul<A,X>(a,in);
    x[idx] = in;
  }
}

// Scale 
template<class A, class X> __host__
void cuNDA_scale( A a, cuNDArray<X> *x )
{
  unsigned int number_of_elements = x->get_number_of_elements();

  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  // Invoke kernel
  cuNDA_scale1_kernel<A,X><<< gridDim, blockDim >>> ( a, x->get_data_ptr(), number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
}

// Scale
template<class A, class X> __global__ 
void cuNDA_scale2_kernel( A *a, X *x, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if( idx < number_of_elements ){
    A in_a = a[idx];
    for( unsigned int batch=0; batch<number_of_batches; batch++ ){
      X in_x = x[batch*number_of_elements+idx];
      x[batch*number_of_elements+idx] = mul<A,X>(in_a,in_x);
    }
  }
}

// Scale 
template<class A, class X> __host__
bool cuNDA_scale( cuNDArray<A> *a, cuNDArray<X> *x )
{
  if( x->get_number_of_elements() < a->get_number_of_elements() ||
      x->get_number_of_elements() % a->get_number_of_elements() ){
    cout << endl << "image dimensions mismatch, cannot scale" << endl;
    return false;
  }
 
  unsigned int number_of_elements = a->get_number_of_elements();
  unsigned int num_batches = x->get_number_of_elements() / a->get_number_of_elements();
 
  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));
 
  // Invoke kernel
  cuNDA_scale2_kernel<A,X><<< gridDim, blockDim >>> ( a->get_data_ptr(), x->get_data_ptr(), num_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
  return true;
}

// 'axpy'
template<class A, class XY> __global__ 
void cuNDA_axpy_kernel( A a, XY *x, XY *y, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if( idx < number_of_elements ){
    XY in_x = x[idx];
    XY in_y = y[idx];
    in_y += mul<A,XY>(a,in_x);
    y[idx] = in_y;
  }
}

// 'axpy' 
template<class A, class XY> __host__
bool cuNDA_axpy( A a, cuNDArray<XY> *x, cuNDArray<XY> *y )
{
  if( x->get_number_of_elements() != y->get_number_of_elements() ){
    cout << endl << "image dimensions mismatch in 'axpy'" << endl;
    return false;
  }

  unsigned int number_of_elements = y->get_number_of_elements();

  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  // Invoke kernel
  cuNDA_axpy_kernel<<< gridDim, blockDim >>> ( a, x->get_data_ptr(), y->get_data_ptr(), number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
  return true;
}

// 'axpby'
template<class A, class B, class XY> __global__ 
void cuNDA_axpby_kernel( A *a, XY *x, B *b, XY *y, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if( idx < number_of_elements ){
    A in_a = a[idx];
    B in_b = b[idx];
    for( unsigned int batch=0; batch<number_of_batches; batch++ ){
      unsigned int iidx = batch*number_of_elements + idx;
      XY in_x = x[iidx];
      XY in_y = y[iidx];
      in_y = mul<B,XY>(in_b,in_y);
      in_y += mul<A,XY>(in_a,in_x);
      y[iidx] = in_y;
    }
  }
}

// '.axpby' 
template<class A, class B, class XY> __host__
bool cuNDA_axpby( cuNDArray<A> *a, cuNDArray<XY> *x, cuNDArray<B> *b, cuNDArray<XY> *y )
{
  if( x->get_number_of_elements() != y->get_number_of_elements() ){
    cout << endl << "image dimensions mismatch in 'axpby'" << endl;
    return false;
  }

  if( a->get_number_of_elements() != b->get_number_of_elements() ){
    cout << endl << "image dimensions mismatch in 'axpby'" << endl;
    return false;
  }

  if( x->get_number_of_elements() < a->get_number_of_elements() ||
      x->get_number_of_elements() % a->get_number_of_elements() ){
    cout << endl << "image dimensions mismatch in 'axpby'" << endl;
    return false;
  }
 
  unsigned int number_of_batches = x->get_number_of_elements() / a->get_number_of_elements();
  unsigned int number_of_elements = a->get_number_of_elements();

  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  // Invoke kernel
  cuNDA_axpby_kernel<<< gridDim, blockDim >>> ( a->get_data_ptr(), x->get_data_ptr(), b->get_data_ptr(), y->get_data_ptr(), number_of_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
  return true;
}

// Norm
template<class REAL, class T> __global__ 
void cuNDA_norm_kernel( T *in, REAL *out, unsigned int number_of_elements )
{
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
 
  if( idx<number_of_elements ){
    T val = in[idx]; 
    out[idx] = norm(val);
  }
}

// Norm
template<class REAL, class T> __host__ 
auto_ptr< cuNDArray<REAL> > cuNDA_norm( cuNDArray<T> *in )
{
  unsigned int number_of_elements = in->get_number_of_elements();

  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  cuNDArray<REAL> *out = cuNDArray<REAL>::allocate(in->get_dimensions());
 
  // Make modulus image
  if( out != 0x0 )
    cuNDA_norm_kernel<REAL,T><<< gridDim, blockDim >>>( in->get_data_ptr(), out->get_data_ptr(), number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
 
  return auto_ptr< cuNDArray<REAL> >(out);
}


// Norm sqaured
template<class REAL, class T> __global__ 
void cuNDA_norm_squared_kernel( T *in, REAL *out, unsigned int number_of_elements )
{
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
 
  if( idx<number_of_elements ){
    T val = in[idx]; 
    out[idx] = norm_squared(val);
  }
}

// Norm squared
template<class REAL, class T> __host__
auto_ptr< cuNDArray<REAL> > cuNDA_norm_squared( cuNDArray<T> *in )
{
  unsigned int number_of_elements = in->get_number_of_elements();

  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  cuNDArray<REAL> *out = cuNDArray<REAL>::allocate(in->get_dimensions());
 
  // Make norm image
  if( out != 0x0 )
    cuNDA_norm_squared_kernel<<< gridDim, blockDim >>>( in->get_data_ptr(), out->get_data_ptr(), number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
 
  return auto_ptr< cuNDArray<REAL> >(out);
}

// RSS
template<class REAL, class T> __global__ void
cuNDA_rss_kernel( T *in, REAL *out, unsigned int stride, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( idx < number_of_elements ){

    unsigned int in_idx = (idx/stride)*stride*number_of_batches+(idx%stride);

    REAL rss = get_zero<REAL>();
 
    for( unsigned int i=0; i<number_of_batches; i++ ) 
      rss += norm_squared(in[i*stride+in_idx]);

    rss = sqrt(rss);

    out[idx] = rss; 
  }
}

// RSS
template<class REAL, class T> __host__ 
auto_ptr< cuNDArray<REAL> > cuNDA_rss( cuNDArray<T> *in, unsigned int dim )
{
  if( !(in->get_number_of_dimensions()>1) ){
    cout << endl << "cuNDA_rss:: underdimensioned." << endl; 
    return auto_ptr< cuNDArray<REAL> >(0x0);
  }
 
  if( dim > in->get_number_of_dimensions()-1 ){
    cout << endl << "cuNDA_rss:: dimension out of range." << endl; 
    return auto_ptr< cuNDArray<REAL> >(0x0);
  }

  unsigned int number_of_batches = in->get_size(dim);
  unsigned int number_of_elements = in->get_number_of_elements()/number_of_batches;
  unsigned int stride = 1;

  vector<unsigned int> dims;
  for( unsigned int i=0; i<in->get_number_of_dimensions(); i++ ){
    if( i != dim )
      dims.push_back(in->get_size(i));
    if( i < dim )
      stride *= in->get_size(i);
  }

  cuNDArray<REAL> *out = cuNDArray<REAL>::allocate(dims);
 
  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  if ( out != 0x0 )
    cuNDA_rss_kernel<<< gridDim, blockDim >>>( in->get_data_ptr(), out->get_data_ptr(), stride, number_of_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
 
  return auto_ptr< cuNDArray<REAL> >(out);
}

// Sum
template<class T> __global__ void
cuNDA_sum_kernel( T *in, T *out, unsigned int stride, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( idx < number_of_elements ){

    unsigned int in_idx = (idx/stride)*stride*number_of_batches+(idx%stride);
 
    T val = get_zero<T>();
 
    for( unsigned int i=0; i<number_of_batches; i++ ) 
      val += in[i*stride+in_idx];

    out[idx] = val; 
  }
}

// Sum
template<class T> __host__ 
auto_ptr< cuNDArray<T> > cuNDA_sum( cuNDArray<T> *in, unsigned int dim )
{
  if( !(in->get_number_of_dimensions()>1) ){
    cout << endl << "cuNDA_sum:: underdimensioned." << endl; 
    return auto_ptr< cuNDArray<T> >(0x0);
  }
 
  if( dim > in->get_number_of_dimensions()-1 ){
    cout << endl << "cuNDA_rss:: dimension out of range." << endl; 
    return auto_ptr< cuNDArray<T> >(0x0);
  }
 
  unsigned int number_of_batches = in->get_size(dim);
  unsigned int number_of_elements = in->get_number_of_elements()/number_of_batches;
  unsigned int stride = 1;

  vector<unsigned int> dims;
  for( unsigned int i=0; i<in->get_number_of_dimensions(); i++ ){
    if( i != dim )
      dims.push_back(in->get_size(i));
    if( i < dim )
      stride *= in->get_size(i);
  }

  cuNDArray<T> *out = cuNDArray<T>::allocate(dims);
 
  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  if( out != 0x0 )
    cuNDA_sum_kernel<<< gridDim, blockDim >>>( in->get_data_ptr(), out->get_data_ptr(), stride, number_of_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();

  return auto_ptr< cuNDArray<T> >(out);
}

// Build correlation matrix
template<class T> __global__ void
cuNDA_correlation_kernel( T *in, T *corrm, unsigned int num_batches, unsigned int num_elements )
{
  const unsigned int p = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int i = threadIdx.y;

  if( p < num_elements ){
    for( unsigned int j=0; j<i; j++){
      corrm[(j*num_batches+i)*num_elements+p] = mul<T,T>(in[i*num_elements+p], conj(in[j*num_elements+p]));
      corrm[(i*num_batches+j)*num_elements+p] = conj(corrm[(j*num_batches+i)*num_elements+p]);
    }
    corrm[(i*num_batches+i)*num_elements+p] = mul<T,T>(in[i*num_elements+p],conj(in[i*num_elements+p]));
  }
}

// Build correlation matrix
template<class T> __host__ 
auto_ptr< cuNDArray<T> > cuNDA_correlation( cuNDArray<T> *in )
{
  if( !(in->get_number_of_dimensions()>1) ){
    cout << endl << "cuNDA_correlation:: underdimensioned." << endl; 
    return auto_ptr< cuNDArray<T> >(0x0);
  }

  unsigned int number_of_batches = in->get_size(in->get_number_of_dimensions()-1);
  unsigned int number_of_elements = in->get_number_of_elements()/number_of_batches;

  vector<unsigned int> dims = in->get_dimensions();
  dims.push_back(number_of_batches);

  cuNDArray<T> *out = cuNDArray<T>::allocate(dims);
 
  int device; hipGetDevice( &device );
  hipDeviceProp_t deviceProp; hipGetDeviceProperties( &deviceProp, device );
  unsigned int warp_size = deviceProp.warpSize;

  dim3 blockDim(((512/number_of_batches)/warp_size)*warp_size, number_of_batches);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  if( blockDim.x == 0 ){
    cout << endl << "cuNDA_correlation:: correlation dimension exceeds capacity." << endl; 
    return auto_ptr< cuNDArray<T> >(0x0);
  }

  if( out != 0x0 )
    cuNDA_correlation_kernel<<< gridDim, blockDim >>>( in->get_data_ptr(), out->get_data_ptr(), number_of_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
 
  return auto_ptr< cuNDArray<T> >(out);
}

// Crop
template<class T, unsigned int D> __global__ void
cuNDA_crop_kernel( vectord<unsigned int,D> offset, vectord<unsigned int,D> matrix_size_in, vectord<unsigned int,D> matrix_size_out, 
		   T *in, T *out, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( idx < number_of_elements ){
    const vectord<unsigned int,D> co = idx_to_co( idx, matrix_size_out );
    const vectord<unsigned int,D> co_os = offset + co;
    const unsigned int source_idx = co_to_idx(co_os, matrix_size_in);
    const unsigned int source_elements = prod(matrix_size_in);
    for( unsigned int image=0; image<number_of_batches; image++ )
      out[image*number_of_elements+idx] = in[image*source_elements+source_idx];
  }
}

// Crop
template<class T, unsigned int D> __host__
bool cuNDA_crop( vectord<unsigned int,D> offset, cuNDArray<T> *in, cuNDArray<T> *out )
{ 
  if( in->get_number_of_dimensions() != out->get_number_of_dimensions() ){
    cout << endl << "image dimensions mismatch, cannot crop" << endl;
    return false;
  }

  if( !(in->get_number_of_dimensions() == D || in->get_number_of_dimensions() == D+1) ){
    cout << endl << "image dimensions mismatch, cannot crop" << endl;
    return false;
  }

  unsigned int number_of_batches = 
    (out->get_number_of_dimensions() == D ) ? 1 : out->get_size(out->get_number_of_dimensions()-1);

  vectord<unsigned int, D> matrix_size_in; cuNDA_fromVec<D>( in->get_dimensions(), matrix_size_in );
  vectord<unsigned int, D> matrix_size_out; cuNDA_fromVec<D>( out->get_dimensions(), matrix_size_out );
 
  if( weak_greater(offset+matrix_size_out, matrix_size_in) ){
    cout << endl << "cropping size mismatch, cannot crop" << endl;
    return false;
  }
 
  unsigned int number_of_elements = prod(matrix_size_out);

  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  // Invoke kernel
  cuNDA_crop_kernel<T,D><<< gridDim, blockDim >>> ( offset, matrix_size_in, matrix_size_out, in->get_data_ptr(), out->get_data_ptr(), number_of_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();

  return true;
}

// Expand and zero fill
template<class T, unsigned int D> __global__ void
cuNDA_expand_with_zero_fill_kernel( vectord<unsigned int,D> matrix_size_in, vectord<unsigned int, D> matrix_size_out, 
				    T *in, T *out, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( idx < number_of_elements ){
    const vectord<unsigned int,D> co_out = idx_to_co( idx, matrix_size_out );
    const vectord<unsigned int,D> offset = (matrix_size_out-matrix_size_in)>>1;
    T _out;
    bool inside = (co_out>=offset) && (co_out<(matrix_size_in+offset));
    for( unsigned int batch=0; batch<number_of_batches; batch++ ){
      if( inside )
	_out = in[co_to_idx(co_out-offset, matrix_size_in)+batch*prod(matrix_size_in)];
      else{
	T zero = get_zero<T>();
	_out = zero;
      } 
      out[idx+batch*number_of_elements] = _out;
    }
  }
}

// Expand and zero fill
template<class T, unsigned int D> __host__
bool cuNDA_expand_with_zero_fill( cuNDArray<T> *in, cuNDArray<T> *out )
{ 
  if( in->get_number_of_dimensions() != out->get_number_of_dimensions() ){
    cout << endl << "Image dimensions mismatch, cannot expand" << endl;
    return false;
  }
 
  vectord<unsigned int, D> matrix_size_in; cuNDA_fromVec( in->get_dimensions(), matrix_size_in );
  vectord<unsigned int, D> matrix_size_out; cuNDA_fromVec( out->get_dimensions(), matrix_size_out );
 
  if( weak_greater(matrix_size_in,matrix_size_out) ){
    cout << endl << "Size mismatch, cannot expand" << endl;
    return false;
  }
 
  unsigned int number_of_batches = 
    (out->get_number_of_dimensions() == D ) ? 1 : out->get_size(out->get_number_of_dimensions()-1);

  unsigned int number_of_elements = prod(matrix_size_out);
 
  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));
 
  // Invoke kernel
  cuNDA_expand_with_zero_fill_kernel<<< gridDim, blockDim >>> ( matrix_size_in, matrix_size_out, in->get_data_ptr(), out->get_data_ptr(), number_of_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();

  return true;
}

// Zero fill border (rectangular)
template<class T, unsigned int D> __global__ void
cuNDA_zero_fill_border_kernel( vectord<unsigned int, D> matrix_size_in, vectord<unsigned int, D> matrix_size_out, 
			       T *image, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( idx < number_of_elements ){
    const vectord<unsigned int,D> co_out = idx_to_co( idx, matrix_size_out );
    const vectord<unsigned int,D> offset = (matrix_size_out-matrix_size_in)>>1;
    if( weak_less( co_out, offset ) || weak_greater_equal( co_out, matrix_size_in+offset ) ){
      T zero = get_zero<T>();
      for( unsigned int batch=0; batch<number_of_batches; batch++ ){
	image[idx+batch*number_of_elements] = zero;
      }
    }
    else
      ; // do nothing
  }
}

// Zero fill border (rectangular)
template<class T, unsigned int D> __host__
bool cuNDA_zero_fill_border( vectord<unsigned int, D> matrix_size_in, cuNDArray<T> *out )
{ 
  vectord<unsigned int, D> matrix_size_out; cuNDA_fromVec( out->get_dimensions(), matrix_size_out );
 
  if( weak_greater(matrix_size_in, matrix_size_out) ){
    cout << endl << "Size mismatch, cannot zero fill" << endl;
    return false;
  }
 
  unsigned int number_of_batches = 
    (out->get_number_of_dimensions() == D ) ? 1 : out->get_size(out->get_number_of_dimensions()-1);

  unsigned int number_of_elements = prod(matrix_size_out);
 
  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));
 
  // Invoke kernel
  cuNDA_zero_fill_border_kernel<<< gridDim, blockDim >>> ( matrix_size_in, matrix_size_out, out->get_data_ptr(), number_of_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();

  return true;
}

// Zero fill border (circular)
template<class REAL, class T, unsigned int D> __global__ void
cuNDA_zero_fill_border_kernel( vectord<REAL,D> radius, vectord<unsigned int, D> matrix_size_out, 
			       T *image, unsigned int number_of_batches, unsigned int number_of_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
 
  if( idx < number_of_elements ){
    
    vectord<REAL,D> half_matrix_size_out_real; to_reald( half_matrix_size_out_real, matrix_size_out>>1 );

    const vectord<unsigned int,D> co_out = idx_to_co( idx, matrix_size_out );
    vectord<REAL,D> co_out_real; to_reald( co_out_real, co_out );
    
    vectord<REAL,D> co_f = abs( co_out_real - half_matrix_size_out_real );
    
    if( co_f<radius )
      ; // do nothing
    else{
      T zero = get_zero<T>();
      for( unsigned int batch=0; batch<number_of_batches; batch++ ){
	image[idx+batch*number_of_elements] = zero;
      }
    } 
  }
}

// Zero fill border (circular)
template<class REAL, class T, unsigned int D> __host__
bool cuNDA_zero_fill_border( vectord<REAL,D> radius, cuNDArray<T> *out )
{
  if( out->get_number_of_dimensions() != D ){
    cout << endl << "Image dimensions mismatch, cannot zero fill" << endl;
    return false;
  }
 
  vectord<unsigned int, D> matrix_size_out; cuNDA_fromVec( out->get_dimensions(), matrix_size_out );
  vectord<REAL,D> matrix_size_out_real; to_reald( matrix_size_out_real, matrix_size_out );

  if( weak_greater(radius, matrix_size_out_real) ){
    cout << endl << "Size mismatch, cannot zero fill" << endl;
    return false;
  }
 
  unsigned int number_of_batches = 
    (out->get_number_of_dimensions() == D ) ? 1 : out->get_size(out->get_number_of_dimensions()-1);

  unsigned int number_of_elements = prod(matrix_size_out);
 
  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));
 
  // Invoke kernel
  cuNDA_zero_fill_border_kernel<<< gridDim, blockDim >>> ( radius, matrix_size_out, out->get_data_ptr(), number_of_batches, number_of_elements );
 
  CHECK_FOR_CUDA_ERROR();
 
  return true;
}

// cuNDArray to std::vector
template<unsigned int D> __host__
vector<unsigned int> cuNDA_toVec( vectord<unsigned int,D> dims )
{
  vector<unsigned int> out(D);
  for( unsigned int i=0; i<D; i++ )
    out[i] = dims.vec[i];
  return out;
}

// std::vector to cuNDArray
template<unsigned int D> __host__
bool cuNDA_fromVec( vector<unsigned int> from, vectord<unsigned int,D> &to )
{
  if( from.size() < D ){
    cout << "Cannot convert vector to UINTd" << endl;
    return false;
  }
 
  vector<unsigned int>::iterator it = from.begin();
  for( unsigned int i=0; i<D; i++ )
    to.vec[i] = *it;
 
  return true;
}

//
// Instantiation
//

template std::vector<unsigned int> cuNDA_toVec<2>(vectord<unsigned int,2>);
template std::vector<unsigned int> cuNDA_toVec<3>(vectord<unsigned int,3>);
template std::vector<unsigned int> cuNDA_toVec<4>(vectord<unsigned int,4>);

template bool cuNDA_fromVec<2>( std::vector<unsigned int>, vectord<unsigned int,2>&);
template bool cuNDA_fromVec<3>( std::vector<unsigned int>, vectord<unsigned int,3>&);
template bool cuNDA_fromVec<4>( std::vector<unsigned int>, vectord<unsigned int,4>&);

// Instanciation -- single precision

template void cuNDA_clear<float>(cuNDArray<float>*);
template void cuNDA_clear<vectord<float,2> >(cuNDArray<vectord<float,2> >*);
template void cuNDA_clear<real_complex<float> >(cuNDArray<real_complex<float> >*);
template void cuNDA_clear<float_complex>(cuNDArray<float_complex>*);

template void cuNDA_abs<float>(cuNDArray<float>*);

template void cuNDA_reciprocal<float>(cuNDArray<float>*);
template void cuNDA_reciprocal<vectord<float,2> >(cuNDArray<vectord<float,2> >*);
template void cuNDA_reciprocal<real_complex<float> >(cuNDArray<real_complex<float> >*);
template void cuNDA_reciprocal<float_complex>(cuNDArray<float_complex>*);

template bool cuNDA_rss_normalize<float, float>(cuNDArray<float>*, unsigned int);
template bool cuNDA_rss_normalize<float, vectord<float,2> >(cuNDArray<vectord<float,2> >*, unsigned int);
template bool cuNDA_rss_normalize<float, real_complex<float> >(cuNDArray<real_complex<float> >*, unsigned int);
template bool cuNDA_rss_normalize<float, float_complex>(cuNDArray<float_complex>*, unsigned int);

template void cuNDA_scale<float, float>(float, cuNDArray<float>*);
template void cuNDA_scale<float, vectord<float,2> >(float, cuNDArray<vectord<float,2> >*);
template void cuNDA_scale<vectord<float,2>, vectord<float,2> >(vectord<float,2>, cuNDArray<vectord<float,2> >*);
template void cuNDA_scale<float, real_complex<float> >(float, cuNDArray<real_complex<float> >*);
template void cuNDA_scale<real_complex<float>, real_complex<float> >(real_complex<float>, cuNDArray<real_complex<float> >*);
template void cuNDA_scale<float, float_complex>(float, cuNDArray<float_complex>*);
template void cuNDA_scale<float_complex, float_complex>(float_complex, cuNDArray<float_complex>*);

template bool cuNDA_scale<float, float>(cuNDArray<float>*, cuNDArray<float>*);
template bool cuNDA_scale<float, vectord<float,2> >(cuNDArray<float>*, cuNDArray<vectord<float,2> >*);
template bool cuNDA_scale<vectord<float,2>, vectord<float,2> >(cuNDArray<vectord<float,2> >*, cuNDArray<vectord<float,2> >*);
template bool cuNDA_scale<float, real_complex<float> >(cuNDArray<float>*, cuNDArray<real_complex<float> >*);
template bool cuNDA_scale<real_complex<float>, real_complex<float> >(cuNDArray<real_complex<float> >*, cuNDArray<real_complex<float> >*);
template bool cuNDA_scale<float, float_complex>(cuNDArray<float>*, cuNDArray<float_complex>*);
template bool cuNDA_scale<float_complex, float_complex>(cuNDArray<float_complex>*, cuNDArray<float_complex>*);

template bool cuNDA_axpy<float, float>( float, cuNDArray<float>*, cuNDArray<float>*);
template bool cuNDA_axpy<float, vectord<float,2> >( float, cuNDArray<vectord<float,2> >*, cuNDArray<vectord<float,2> >*);
template bool cuNDA_axpy<float, real_complex<float> >( float, cuNDArray<real_complex<float> >*, cuNDArray<real_complex<float> >*);
template bool cuNDA_axpy<float, float_complex>( float, cuNDArray<float_complex>*, cuNDArray<float_complex>*);

template bool cuNDA_axpby<float, float, float>( cuNDArray<float>*, cuNDArray<float>*, cuNDArray<float>*, cuNDArray<float>*);
template bool cuNDA_axpby<float, float, vectord<float,2> >( cuNDArray<float>*, cuNDArray<vectord<float,2> >*, cuNDArray<float>*, cuNDArray<vectord<float,2> >*);
template bool cuNDA_axpby<float, float, real_complex<float> >( cuNDArray<float>*, cuNDArray<real_complex<float> >*, cuNDArray<float>*, cuNDArray<real_complex<float> >*);
template bool cuNDA_axpby<float, float, float_complex>( cuNDArray<float>*, cuNDArray<float_complex>*, cuNDArray<float>*, cuNDArray<float_complex>*);

template auto_ptr< cuNDArray<float> > cuNDA_norm<float,float>( cuNDArray<float>*);
template auto_ptr< cuNDArray<float> > cuNDA_norm<float,real_complex<float> >( cuNDArray<real_complex<float> >*);
template auto_ptr< cuNDArray<float> > cuNDA_norm<float,float_complex>( cuNDArray<float_complex>*);
template auto_ptr< cuNDArray<float> > cuNDA_norm<float,vectord<float,2> >( cuNDArray<vectord<float,2> >*);

template auto_ptr< cuNDArray<float> > cuNDA_norm_squared<float, float>( cuNDArray<float>*);
template auto_ptr< cuNDArray<float> > cuNDA_norm_squared<float, vectord<float,2> >( cuNDArray<vectord<float,2> >*);
template auto_ptr< cuNDArray<float> > cuNDA_norm_squared<float, real_complex<float> >( cuNDArray<real_complex<float> >*);
template auto_ptr< cuNDArray<float> > cuNDA_norm_squared<float, float_complex>( cuNDArray<float_complex>*);

template auto_ptr< cuNDArray<float> > cuNDA_rss<float, float>( cuNDArray<float>*, unsigned int);
template auto_ptr< cuNDArray<float> > cuNDA_rss<float, vectord<float,2> >( cuNDArray<vectord<float,2> >*, unsigned int);
template auto_ptr< cuNDArray<float> > cuNDA_rss<float, real_complex<float> >( cuNDArray<real_complex<float> >*, unsigned int);
template auto_ptr< cuNDArray<float> > cuNDA_rss<float, float_complex>( cuNDArray<float_complex>*, unsigned int);

template auto_ptr< cuNDArray<float> > cuNDA_sum<float>( cuNDArray<float>*, unsigned int);
template auto_ptr< cuNDArray<vectord<float,2> > > cuNDA_sum<vectord<float,2> >( cuNDArray<vectord<float,2> >*, unsigned int);
template auto_ptr< cuNDArray<real_complex<float> > > cuNDA_sum<real_complex<float> >( cuNDArray<real_complex<float> >*, unsigned int);
template auto_ptr< cuNDArray<float_complex> > cuNDA_sum<float_complex>( cuNDArray<float_complex>*, unsigned int);

template auto_ptr< cuNDArray<float> > cuNDA_correlation<float>( cuNDArray<float>*);
template auto_ptr< cuNDArray<vectord<float,2> > > cuNDA_correlation<vectord<float,2> >( cuNDArray<vectord<float,2> >*);
template auto_ptr< cuNDArray<real_complex<float> > > cuNDA_correlation<real_complex<float> >( cuNDArray<real_complex<float> >*);
template auto_ptr< cuNDArray<float_complex> > cuNDA_correlation<float_complex>( cuNDArray<float_complex>*);

template bool cuNDA_crop<float,2>( vectord<unsigned int,2>, cuNDArray<float>*, cuNDArray<float>*);
template bool cuNDA_crop<vectord<float,2>,2>( vectord<unsigned int,2>, cuNDArray<vectord<float,2> >*, cuNDArray<vectord<float,2> >*);
template bool cuNDA_crop<real_complex<float>,2>( vectord<unsigned int,2>, cuNDArray<real_complex<float> >*, cuNDArray<real_complex<float> >*);
template bool cuNDA_crop<float_complex,2>( vectord<unsigned int,2>, cuNDArray<float_complex>*, cuNDArray<float_complex>*);

template bool cuNDA_crop<float,3>( vectord<unsigned int,3>, cuNDArray<float>*, cuNDArray<float>*);
template bool cuNDA_crop<vectord<float,2>,3>( vectord<unsigned int,3>, cuNDArray<vectord<float,2> >*, cuNDArray<vectord<float,2> >*);
template bool cuNDA_crop<real_complex<float>,3>( vectord<unsigned int,3>, cuNDArray<real_complex<float> >*, cuNDArray<real_complex<float> >*);
template bool cuNDA_crop<float_complex,3>( vectord<unsigned int,3>, cuNDArray<float_complex>*, cuNDArray<float_complex>*);

template bool cuNDA_crop<float,4>( vectord<unsigned int,4>, cuNDArray<float>*, cuNDArray<float>*);
template bool cuNDA_crop<vectord<float,2>,4>( vectord<unsigned int,4>, cuNDArray<vectord<float,2> >*, cuNDArray<vectord<float,2> >*);
template bool cuNDA_crop<real_complex<float>,4>( vectord<unsigned int,4>, cuNDArray<real_complex<float> >*, cuNDArray<real_complex<float> >*);
template bool cuNDA_crop<float_complex,4>( vectord<unsigned int,4>, cuNDArray<float_complex>*, cuNDArray<float_complex>*);

template bool cuNDA_expand_with_zero_fill<float,2>( cuNDArray<float>*, cuNDArray<float>*);
template bool cuNDA_expand_with_zero_fill<vectord<float,2>,2>( cuNDArray<vectord<float,2> >*, cuNDArray<vectord<float,2> >*);
template bool cuNDA_expand_with_zero_fill<real_complex<float>,2>( cuNDArray<real_complex<float> >*, cuNDArray<real_complex<float> >*);
template bool cuNDA_expand_with_zero_fill<float_complex,2>( cuNDArray<float_complex>*, cuNDArray<float_complex>*);

template bool cuNDA_expand_with_zero_fill<float,3>( cuNDArray<float>*, cuNDArray<float>*);
template bool cuNDA_expand_with_zero_fill<vectord<float,2>,3>( cuNDArray<vectord<float,2> >*, cuNDArray<vectord<float,2> >*);
template bool cuNDA_expand_with_zero_fill<real_complex<float>,3>( cuNDArray<real_complex<float> >*, cuNDArray<real_complex<float> >*);
template bool cuNDA_expand_with_zero_fill<float_complex,3>( cuNDArray<float_complex>*, cuNDArray<float_complex>*);

template bool cuNDA_expand_with_zero_fill<float,4>( cuNDArray<float>*, cuNDArray<float>*);
template bool cuNDA_expand_with_zero_fill<vectord<float,2>,4>( cuNDArray<vectord<float,2> >*, cuNDArray<vectord<float,2> >*);
template bool cuNDA_expand_with_zero_fill<real_complex<float>,4>( cuNDArray<real_complex<float> >*, cuNDArray<real_complex<float> >*);
template bool cuNDA_expand_with_zero_fill<float_complex,4>( cuNDArray<float_complex>*, cuNDArray<float_complex>*);

template bool cuNDA_zero_fill_border<float,2>(vectord<unsigned int, 2>, cuNDArray<float>*);
template bool cuNDA_zero_fill_border<vectord<float,2>,2>(vectord<unsigned int, 2>, cuNDArray<vectord<float,2> >*);
template bool cuNDA_zero_fill_border<real_complex<float>,2>(vectord<unsigned int, 2>, cuNDArray<real_complex<float> >*);
template bool cuNDA_zero_fill_border<float_complex,2>(vectord<unsigned int, 2>, cuNDArray<float_complex>*);

template bool cuNDA_zero_fill_border<float,3>(vectord<unsigned int, 3>, cuNDArray<float>*);
template bool cuNDA_zero_fill_border<vectord<float,2>,3>(vectord<unsigned int, 3>, cuNDArray<vectord<float,2> >*);
template bool cuNDA_zero_fill_border<real_complex<float>,3>(vectord<unsigned int, 3>, cuNDArray<real_complex<float> >*);
template bool cuNDA_zero_fill_border<float_complex,3>(vectord<unsigned int, 3>, cuNDArray<float_complex>*);

template bool cuNDA_zero_fill_border<float,4>(vectord<unsigned int, 4>, cuNDArray<float>*);
template bool cuNDA_zero_fill_border<vectord<float,2>,4>(vectord<unsigned int, 4>, cuNDArray<vectord<float,2> >*);
template bool cuNDA_zero_fill_border<real_complex<float>,4>(vectord<unsigned int, 4>, cuNDArray<real_complex<float> >*);
template bool cuNDA_zero_fill_border<float_complex,4>(vectord<unsigned int, 4>, cuNDArray<float_complex>*);

template bool cuNDA_zero_fill_border<float,float,2>(vectord<float,2>, cuNDArray<float>*);
template bool cuNDA_zero_fill_border<float,vectord<float,2>,2>(vectord<float,2>, cuNDArray<vectord<float,2> >*);
template bool cuNDA_zero_fill_border<float,real_complex<float>,2>(vectord<float,2>, cuNDArray<real_complex<float> >*);
template bool cuNDA_zero_fill_border<float,float_complex,2>(vectord<float,2>, cuNDArray<float_complex>*);

//
// Double precision.
// Copy & Paste from above. Replace float with double
//

template void cuNDA_clear<double>(cuNDArray<double>*);
template void cuNDA_clear<vectord<double,2> >(cuNDArray<vectord<double,2> >*);
template void cuNDA_clear<real_complex<double> >(cuNDArray<real_complex<double> >*);
template void cuNDA_clear<double_complex>(cuNDArray<double_complex>*);

template void cuNDA_abs<double>(cuNDArray<double>*);

template void cuNDA_reciprocal<double>(cuNDArray<double>*);
template void cuNDA_reciprocal<vectord<double,2> >(cuNDArray<vectord<double,2> >*);
template void cuNDA_reciprocal<real_complex<double> >(cuNDArray<real_complex<double> >*);
template void cuNDA_reciprocal<double_complex>(cuNDArray<double_complex>*);

template bool cuNDA_rss_normalize<double, double>(cuNDArray<double>*, unsigned int);
template bool cuNDA_rss_normalize<double, vectord<double,2> >(cuNDArray<vectord<double,2> >*, unsigned int);
template bool cuNDA_rss_normalize<double, real_complex<double> >(cuNDArray<real_complex<double> >*, unsigned int);
template bool cuNDA_rss_normalize<double, double_complex>(cuNDArray<double_complex>*, unsigned int);

template void cuNDA_scale<double, double>(double, cuNDArray<double>*);
template void cuNDA_scale<double, vectord<double,2> >(double, cuNDArray<vectord<double,2> >*);
template void cuNDA_scale<vectord<double,2>, vectord<double,2> >(vectord<double,2>, cuNDArray<vectord<double,2> >*);
template void cuNDA_scale<double, real_complex<double> >(double, cuNDArray<real_complex<double> >*);
template void cuNDA_scale<real_complex<double>, real_complex<double> >(real_complex<double>, cuNDArray<real_complex<double> >*);
template void cuNDA_scale<double, double_complex>(double, cuNDArray<double_complex>*);
template void cuNDA_scale<double_complex, double_complex>(double_complex, cuNDArray<double_complex>*);

template bool cuNDA_scale<double, double>(cuNDArray<double>*, cuNDArray<double>*);
template bool cuNDA_scale<double, vectord<double,2> >(cuNDArray<double>*, cuNDArray<vectord<double,2> >*);
template bool cuNDA_scale<vectord<double,2>, vectord<double,2> >(cuNDArray<vectord<double,2> >*, cuNDArray<vectord<double,2> >*);
template bool cuNDA_scale<double, real_complex<double> >(cuNDArray<double>*, cuNDArray<real_complex<double> >*);
template bool cuNDA_scale<real_complex<double>, real_complex<double> >(cuNDArray<real_complex<double> >*, cuNDArray<real_complex<double> >*);
template bool cuNDA_scale<double, double_complex>(cuNDArray<double>*, cuNDArray<double_complex>*);
template bool cuNDA_scale<double_complex, double_complex>(cuNDArray<double_complex>*, cuNDArray<double_complex>*);

template bool cuNDA_axpy<double, double>( double, cuNDArray<double>*, cuNDArray<double>*);
template bool cuNDA_axpy<double, vectord<double,2> >( double, cuNDArray<vectord<double,2> >*, cuNDArray<vectord<double,2> >*);
template bool cuNDA_axpy<double, real_complex<double> >( double, cuNDArray<real_complex<double> >*, cuNDArray<real_complex<double> >*);
template bool cuNDA_axpy<double, double_complex>( double, cuNDArray<double_complex>*, cuNDArray<double_complex>*);

template bool cuNDA_axpby<double, double, double>( cuNDArray<double>*, cuNDArray<double>*, cuNDArray<double>*, cuNDArray<double>*);
template bool cuNDA_axpby<double, double, vectord<double,2> >( cuNDArray<double>*, cuNDArray<vectord<double,2> >*, cuNDArray<double>*, cuNDArray<vectord<double,2> >*);
template bool cuNDA_axpby<double, double, real_complex<double> >( cuNDArray<double>*, cuNDArray<real_complex<double> >*, cuNDArray<double>*, cuNDArray<real_complex<double> >*);
template bool cuNDA_axpby<double, double, double_complex>( cuNDArray<double>*, cuNDArray<double_complex>*, cuNDArray<double>*, cuNDArray<double_complex>*);

template auto_ptr< cuNDArray<double> > cuNDA_norm<double,double>( cuNDArray<double>*);
template auto_ptr< cuNDArray<double> > cuNDA_norm<double,real_complex<double> >( cuNDArray<real_complex<double> >*);
template auto_ptr< cuNDArray<double> > cuNDA_norm<double,double_complex>( cuNDArray<double_complex>*);
template auto_ptr< cuNDArray<double> > cuNDA_norm<double,vectord<double,2> >( cuNDArray<vectord<double,2> >*);

template auto_ptr< cuNDArray<double> > cuNDA_norm_squared<double, double>( cuNDArray<double>*);
template auto_ptr< cuNDArray<double> > cuNDA_norm_squared<double, vectord<double,2> >( cuNDArray<vectord<double,2> >*);
template auto_ptr< cuNDArray<double> > cuNDA_norm_squared<double, real_complex<double> >( cuNDArray<real_complex<double> >*);
template auto_ptr< cuNDArray<double> > cuNDA_norm_squared<double, double_complex>( cuNDArray<double_complex>*);

template auto_ptr< cuNDArray<double> > cuNDA_rss<double, double>( cuNDArray<double>*, unsigned int);
template auto_ptr< cuNDArray<double> > cuNDA_rss<double, vectord<double,2> >( cuNDArray<vectord<double,2> >*, unsigned int);
template auto_ptr< cuNDArray<double> > cuNDA_rss<double, real_complex<double> >( cuNDArray<real_complex<double> >*, unsigned int);
template auto_ptr< cuNDArray<double> > cuNDA_rss<double, double_complex>( cuNDArray<double_complex>*, unsigned int);

template auto_ptr< cuNDArray<double> > cuNDA_sum<double>( cuNDArray<double>*, unsigned int);
template auto_ptr< cuNDArray<vectord<double,2> > > cuNDA_sum<vectord<double,2> >( cuNDArray<vectord<double,2> >*, unsigned int);
template auto_ptr< cuNDArray<real_complex<double> > > cuNDA_sum<real_complex<double> >( cuNDArray<real_complex<double> >*, unsigned int);
template auto_ptr< cuNDArray<double_complex> > cuNDA_sum<double_complex>( cuNDArray<double_complex>*, unsigned int);

template auto_ptr< cuNDArray<double> > cuNDA_correlation<double>( cuNDArray<double>*);
template auto_ptr< cuNDArray<vectord<double,2> > > cuNDA_correlation<vectord<double,2> >( cuNDArray<vectord<double,2> >*);
template auto_ptr< cuNDArray<real_complex<double> > > cuNDA_correlation<real_complex<double> >( cuNDArray<real_complex<double> >*);
template auto_ptr< cuNDArray<double_complex> > cuNDA_correlation<double_complex>( cuNDArray<double_complex>*);

template bool cuNDA_crop<double,2>( vectord<unsigned int,2>, cuNDArray<double>*, cuNDArray<double>*);
template bool cuNDA_crop<vectord<double,2>,2>( vectord<unsigned int,2>, cuNDArray<vectord<double,2> >*, cuNDArray<vectord<double,2> >*);
template bool cuNDA_crop<real_complex<double>,2>( vectord<unsigned int,2>, cuNDArray<real_complex<double> >*, cuNDArray<real_complex<double> >*);
template bool cuNDA_crop<double_complex,2>( vectord<unsigned int,2>, cuNDArray<double_complex>*, cuNDArray<double_complex>*);

template bool cuNDA_crop<double,3>( vectord<unsigned int,3>, cuNDArray<double>*, cuNDArray<double>*);
template bool cuNDA_crop<vectord<double,2>,3>( vectord<unsigned int,3>, cuNDArray<vectord<double,2> >*, cuNDArray<vectord<double,2> >*);
template bool cuNDA_crop<real_complex<double>,3>( vectord<unsigned int,3>, cuNDArray<real_complex<double> >*, cuNDArray<real_complex<double> >*);
template bool cuNDA_crop<double_complex,3>( vectord<unsigned int,3>, cuNDArray<double_complex>*, cuNDArray<double_complex>*);

template bool cuNDA_crop<double,4>( vectord<unsigned int,4>, cuNDArray<double>*, cuNDArray<double>*);
template bool cuNDA_crop<vectord<double,2>,4>( vectord<unsigned int,4>, cuNDArray<vectord<double,2> >*, cuNDArray<vectord<double,2> >*);
template bool cuNDA_crop<real_complex<double>,4>( vectord<unsigned int,4>, cuNDArray<real_complex<double> >*, cuNDArray<real_complex<double> >*);
template bool cuNDA_crop<double_complex,4>( vectord<unsigned int,4>, cuNDArray<double_complex>*, cuNDArray<double_complex>*);

template bool cuNDA_expand_with_zero_fill<double,2>( cuNDArray<double>*, cuNDArray<double>*);
template bool cuNDA_expand_with_zero_fill<vectord<double,2>,2>( cuNDArray<vectord<double,2> >*, cuNDArray<vectord<double,2> >*);
template bool cuNDA_expand_with_zero_fill<real_complex<double>,2>( cuNDArray<real_complex<double> >*, cuNDArray<real_complex<double> >*);
template bool cuNDA_expand_with_zero_fill<double_complex,2>( cuNDArray<double_complex>*, cuNDArray<double_complex>*);

template bool cuNDA_expand_with_zero_fill<double,3>( cuNDArray<double>*, cuNDArray<double>*);
template bool cuNDA_expand_with_zero_fill<vectord<double,2>,3>( cuNDArray<vectord<double,2> >*, cuNDArray<vectord<double,2> >*);
template bool cuNDA_expand_with_zero_fill<real_complex<double>,3>( cuNDArray<real_complex<double> >*, cuNDArray<real_complex<double> >*);
template bool cuNDA_expand_with_zero_fill<double_complex,3>( cuNDArray<double_complex>*, cuNDArray<double_complex>*);

template bool cuNDA_expand_with_zero_fill<double,4>( cuNDArray<double>*, cuNDArray<double>*);
template bool cuNDA_expand_with_zero_fill<vectord<double,2>,4>( cuNDArray<vectord<double,2> >*, cuNDArray<vectord<double,2> >*);
template bool cuNDA_expand_with_zero_fill<real_complex<double>,4>( cuNDArray<real_complex<double> >*, cuNDArray<real_complex<double> >*);
template bool cuNDA_expand_with_zero_fill<double_complex,4>( cuNDArray<double_complex>*, cuNDArray<double_complex>*);

template bool cuNDA_zero_fill_border<double,2>(vectord<unsigned int, 2>, cuNDArray<double>*);
template bool cuNDA_zero_fill_border<vectord<double,2>,2>(vectord<unsigned int, 2>, cuNDArray<vectord<double,2> >*);
template bool cuNDA_zero_fill_border<real_complex<double>,2>(vectord<unsigned int, 2>, cuNDArray<real_complex<double> >*);
template bool cuNDA_zero_fill_border<double_complex,2>(vectord<unsigned int, 2>, cuNDArray<double_complex>*);

template bool cuNDA_zero_fill_border<double,3>(vectord<unsigned int, 3>, cuNDArray<double>*);
template bool cuNDA_zero_fill_border<vectord<double,2>,3>(vectord<unsigned int, 3>, cuNDArray<vectord<double,2> >*);
template bool cuNDA_zero_fill_border<real_complex<double>,3>(vectord<unsigned int, 3>, cuNDArray<real_complex<double> >*);
template bool cuNDA_zero_fill_border<double_complex,3>(vectord<unsigned int, 3>, cuNDArray<double_complex>*);

template bool cuNDA_zero_fill_border<double,4>(vectord<unsigned int, 4>, cuNDArray<double>*);
template bool cuNDA_zero_fill_border<vectord<double,2>,4>(vectord<unsigned int, 4>, cuNDArray<vectord<double,2> >*);
template bool cuNDA_zero_fill_border<real_complex<double>,4>(vectord<unsigned int, 4>, cuNDArray<real_complex<double> >*);
template bool cuNDA_zero_fill_border<double_complex,4>(vectord<unsigned int, 4>, cuNDArray<double_complex>*);

template bool cuNDA_zero_fill_border<double,double,2>(vectord<double,2>, cuNDArray<double>*);
template bool cuNDA_zero_fill_border<double,vectord<double,2>,2>(vectord<double,2>, cuNDArray<vectord<double,2> >*);
template bool cuNDA_zero_fill_border<double,real_complex<double>,2>(vectord<double,2>, cuNDArray<real_complex<double> >*);
template bool cuNDA_zero_fill_border<double,double_complex,2>(vectord<double,2>, cuNDArray<double_complex>*);
