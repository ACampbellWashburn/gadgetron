#include "hip/hip_runtime.h"
#include "b1_map.hcu"
#include "uintd_operators.hcu"
#include "uintd_utilities.hcu"
#include "ndarray_device_utilities.hcu"
#include "check_CUDA.h"
#include "cuNDFFT.h"

#include <math_functions.h>
#include <iostream>

using namespace std;

template< class UINTd, class T> __host__ 
auto_ptr< cuNDArray<T> > set_box_convkernel( UINTd dims, UINTd box );

template<class REAL, class T> __host__ 
auto_ptr< cuNDArray<T> > extract_csm( cuNDArray<T> *corrm_in, unsigned int number_of_batches, unsigned int number_of_elements );

template<class REAL, class T> __host__ 
void set_phase_reference( cuNDArray<T> *csm, unsigned int number_of_batches, unsigned int number_of_elements );

//
// Main method:
//

template<class UINTd, class REAL, class T> auto_ptr< cuNDArray<T> >
estimate_b1_map( cuNDArray<T>* data_in )
{
  if( data_in->get_number_of_dimensions() < 2 ){
    cout << endl << "estimate_b1_map:: dimensionality mismatch." << endl; 
    return auto_ptr< cuNDArray<T> >(0x0);
  }

  unsigned int d = data_in->get_number_of_dimensions()-1;

  if( d != sizeof(UINTd)/sizeof(unsigned int) ){
    cout << endl << "estimate_b1_map:: dimensionality mismatch." << endl; 
    return auto_ptr< cuNDArray<T> >(0x0);
  }

  vector<unsigned int> image_dims, dims_to_xform;
  unsigned int pixels_per_coil = 1;
  
  for( unsigned int i=0; i<d; i++ ){
    image_dims.push_back(data_in->get_size(i));
    dims_to_xform.push_back(i);
    pixels_per_coil *= data_in->get_size(i);
  }
  
  unsigned int ncoils = data_in->get_size(d);

  // Make a copy of input data
  cuNDArray<T> _data_out = *data_in;
  auto_ptr< cuNDArray<T> > data_out(&_data_out);

  // Normalize by the RSS of the coils
  if( !cuNDA_rss_normalize<REAL, T>( data_out.get(), d ) ){
    cout << endl << "estimate_b1_map:: error in rss_normalize" << endl;
    return auto_ptr< cuNDArray<T> >(0x0);
  }
  
  // Now calculate the correlation matrices
  auto_ptr< cuNDArray<T> > corrm = cuNDA_correlation( data_out.get() );
  //data_out.reset(); // TODO: why does this statement generate a runtime error?
  
// Compute smoothing kernel for convolution
  UINTd dims; cuNDA_fromVec( image_dims, dims );

  UINTd box = uint_to_uintd<UINTd>(7,7);
  auto_ptr< cuNDArray<T> > conv_kernel = set_box_convkernel<UINTd,T>( dims, box );

  // Perform convolution by multiplication in image space
  cuNDFFT().fft( conv_kernel.get() );
  cuNDFFT().fft( corrm.get(), dims_to_xform );
  cuNDA_scale( conv_kernel.get(), corrm.get() );
  cuNDFFT().ifft( corrm.get(), dims_to_xform );
  //conv_kernel.reset();

  // Get the dominant eigenvector for each correlation matrix.
  auto_ptr< cuNDArray<T> > csm = extract_csm<REAL,T>( corrm.get(), ncoils, pixels_per_coil );
  //corrm.reset();
  
  // Set phase according to reference (coil 0)
  set_phase_reference<REAL, T>( csm.get(), ncoils, pixels_per_coil );
  
  return csm;
}

template< class UINTd, class T> __global__ void
set_box_convkernel_kernel( T *out, UINTd dims, UINTd box )
{
  unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( idx < prod(dims) ){
    
    UINTd co = idx_to_co(idx,dims);
    UINTd offset_dim = (dims>>1);
    UINTd offset_box = (box>>1);
    
    if( weak_less(co, offset_dim-offset_box ) || weak_greater_equal(co, offset_dim+offset_box ))
      out[idx] = get_zero<T>();
    else{
      T _out = get_one<T>();
      out[idx] = _out*reciprocal(uintd_to_reald(prod(box)));
    }
  }
}

template< class UINTd, class T> auto_ptr< cuNDArray<T> >
set_box_convkernel( UINTd dims, UINTd box )
{
  cuNDArray<T> *out = cuNDArray<T>::allocate(cuNDA_toVec(dims));
  
  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)prod(dims)/blockDim.x));
  
  if( out != 0x0 )
    set_box_convkernel_kernel<UINTd,T><<< gridDim, blockDim >>>( out->get_data_ptr(), dims, box );
  
  CHECK_FOR_CUDA_ERROR();
  
  return auto_ptr< cuNDArray<T> >(out);
}

extern __shared__ char shared_mem[];

// Extract CSM
template<class REAL, class T> __global__ void
extract_csm_kernel( T *corrm, T *csm, unsigned int num_batches, unsigned int num_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int i = threadIdx.x;

  if( idx < num_elements ){    
    
    // Get the dominant eigenvector for each correlation matrix.
    // Copying Peter Kellman's approach we use the power method:
    //  b_k+1 = A*b_k / ||A*b_k||
    
    T *data_out = (T*) shared_mem;
    T *tmp_v = &(((T*) shared_mem)[num_batches*blockDim.x]);

    const unsigned int iterations = 2;

    for( unsigned int c=0; c<num_batches; c++){
      data_out[c*blockDim.x+i] = get_one<T>();
    }
    
    for( unsigned int it=0; it<iterations; it++ ){

      for( unsigned int c=0; c<num_batches; c++){
	tmp_v[c*blockDim.x+i] = get_zero<T>();
      }
      
      for( unsigned j=0; j<num_batches; j++){
	for( unsigned int k=0; k<num_batches; k++){
	  tmp_v[j*blockDim.x+i] += corrm[(k*num_batches+j)*num_elements+idx]*data_out[k*blockDim.x+i];
	}
      }

      REAL tmp = get_zero<REAL>();
      
      for (unsigned int c=0; c<num_batches; c++){
	tmp += norm_sq(tmp_v[c*blockDim.x+i]);
      }
      
      tmp = sqrt(tmp);
      tmp = reciprocal(tmp);
      
      for (unsigned int c=0; c<num_batches; c++){
	data_out[c*blockDim.x+i] = tmp*tmp_v[c*blockDim.x+i];
      }
    }

    for (unsigned int c=0; c<num_batches; c++){
      csm[c*num_elements+idx] = data_out[c*blockDim.x+i];
    }
  }
}

// Extract CSM
template<class REAL, class T> __host__ 
auto_ptr< cuNDArray<T> > extract_csm( cuNDArray<T> *corrm_in, unsigned int number_of_batches, unsigned int number_of_elements )
{
  vector<unsigned int> image_dims;

  for( unsigned int i=0; i<corrm_in->get_number_of_dimensions()-1; i++ ){
    image_dims.push_back(corrm_in->get_size(i));
  }
  
  cuNDArray<T> *out = cuNDArray<T>::allocate(image_dims);
  
  dim3 blockDim(128);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));
  
  if( out != 0x0 )
    extract_csm_kernel<REAL, T><<< gridDim, blockDim, number_of_batches*blockDim.x*2*sizeof(T) >>>( corrm_in->get_data_ptr(), out->get_data_ptr(), number_of_batches, number_of_elements );
  
  CHECK_FOR_CUDA_ERROR();
  
  return auto_ptr< cuNDArray<T> >(out);
}

// Set refence phase
template<class REAL, class T> __global__ void
set_phase_reference_kernel( T *csm, unsigned int num_batches, unsigned int num_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( idx < num_elements ){    
    REAL angle = arg(csm[idx]); //Phase of the first coil
    REAL sin_a, cos_a; sin_cos( angle, &sin_a, &cos_a );

    T tmp = conj(make_realComplex(cos_a,sin_a));

    for( unsigned int c=0; c<num_batches; c++ ){
      T val =  csm[c*num_elements+idx];
      val *= tmp;
      csm[c*num_elements+idx] = val;
    }
  }
}
  
// Extract CSM
template<class REAL, class T> __host__ 
void set_phase_reference( cuNDArray<T> *csm, unsigned int number_of_batches, unsigned int number_of_elements )
{
  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));
  
  set_phase_reference_kernel<REAL, T><<< gridDim, blockDim >>>( csm->get_data_ptr(), number_of_batches, number_of_elements );
  
  CHECK_FOR_CUDA_ERROR();
}

//
// Template instantiation
//

template auto_ptr< cuNDArray<hipFloatComplex> > estimate_b1_map<uint2, float, hipFloatComplex>(cuNDArray<hipFloatComplex>*);
template auto_ptr< cuNDArray<hipFloatComplex> > estimate_b1_map<uint3, float, hipFloatComplex>(cuNDArray<hipFloatComplex>*);
template auto_ptr< cuNDArray<hipFloatComplex> > estimate_b1_map<uint4, float, hipFloatComplex>(cuNDArray<hipFloatComplex>*);
