#include "hip/hip_runtime.h"
#include "b1_map.h"
#include "vector_td_operators.hcu"
#include "vector_td_utilities.hcu"
#include "ndarray_vector_td_utilities.hcu"
#include "check_CUDA.h"
#include "cuNDFFT.h"

#include <math_functions.h>
#include <iostream>

using namespace std;

template<class REAL, unsigned int D> void
smooth_correlation_matrices( cuNDArray<typename complext<REAL>::Type> *corrm );

template<class REAL> __host__ 
auto_ptr< cuNDArray<typename complext<REAL>::Type> > extract_csm( cuNDArray<typename complext<REAL>::Type> *corrm_in, unsigned int number_of_batches, unsigned int number_of_elements );

template<class REAL> __host__ 
void set_phase_reference( cuNDArray<typename complext<REAL>::Type> *csm, unsigned int number_of_batches, unsigned int number_of_elements );

//
// Main method
//

template<class REAL, unsigned int D> auto_ptr< cuNDArray<typename complext<REAL>::Type> >
estimate_b1_map( cuNDArray<typename complext<REAL>::Type> *data_in )
{
  if( data_in->get_number_of_dimensions() < 2 ){
    cout << endl << "estimate_b1_map:: dimensionality mismatch." << endl; 
    return auto_ptr< cuNDArray<typename complext<REAL>::Type > >(0x0);
  }

  if( data_in->get_number_of_dimensions()-1 != D ){
    cout << endl << "estimate_b1_map:: dimensionality mismatch." << endl; 
    return auto_ptr< cuNDArray<typename complext<REAL>::Type > >(0x0);
  }

  vector<unsigned int> image_dims, dims_to_xform;
  unsigned int pixels_per_coil = 1;
  
  for( unsigned int i=0; i<D; i++ ){
    image_dims.push_back(data_in->get_size(i));
    dims_to_xform.push_back(i);
    pixels_per_coil *= data_in->get_size(i);
  }
  
  unsigned int ncoils = data_in->get_size(D);

  // Make a copy of input data
  cuNDArray<typename complext<REAL>::Type > *_data_out = new cuNDArray<typename complext<REAL>::Type>(*data_in);
  auto_ptr< cuNDArray<typename complext<REAL>::Type> > data_out(_data_out);
  
  // Normalize by the RSS of the coils
  if( !cuNDA_rss_normalize<REAL, typename complext<REAL>::Type>( data_out.get(), D ) ){
    cout << endl << "estimate_b1_map:: error in rss_normalize" << endl;
    return auto_ptr< cuNDArray<typename complext<REAL>::Type> >(0x0);
  }
  
  // Now calculate the correlation matrices
  auto_ptr<cuNDArray<typename complext<REAL>::Type> > corrm = cuNDA_correlation<REAL>( data_out.get() );
  data_out.reset();
  
  // Smooth
  smooth_correlation_matrices<REAL,D>( corrm.get() );

  // Get the dominant eigenvector for each correlation matrix.
  auto_ptr<cuNDArray<typename complext<REAL>::Type> > csm = extract_csm<REAL>( corrm.get(), ncoils, pixels_per_coil );
  corrm.reset();
  
  // Set phase according to reference (coil 0)
  set_phase_reference<REAL>( csm.get(), ncoils, pixels_per_coil );
  
  return csm;
}

// Smooth correlation matrices by box filter (1D)
template<class REAL> __global__ void
smooth_correlation_matrices_kernel( typename complext<REAL>::Type *corrm, intd<1>::Type image_dims )
{
  const int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const int batch = blockIdx.y;

  const int num_image_elements = prod(image_dims);

  if( idx < num_image_elements ){
    
    const int co = idx;    
    const int x = co;
    
    const int size_x = image_dims.vec[0];
    
    const int kernel_width = 7;
    const REAL scale = get_one<REAL>()/((REAL)kernel_width);
    
    typename complext<REAL>::Type result = get_zero<typename complext<REAL>::Type>();
    
    for (int kx = 0; kx < kernel_width; kx++) {
      
      if ((x-(kernel_width>>1)+kx) >= 0 &&
	  (x-(kernel_width>>1)+kx) < size_x)
	{	    
	  int source_offset = 
	    batch*num_image_elements +
	    (x-(kernel_width>>1)+kx);
	  
	  result += corrm[source_offset];
	}
    }
    corrm[batch*num_image_elements+idx] = scale*result;
  }
}

// Smooth correlation matrices by box filter (2D)
template<class REAL> __global__ void
smooth_correlation_matrices_kernel( typename complext<REAL>::Type *corrm, intd<2>::Type image_dims )
{
  const int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const int batch = blockIdx.y;

  const int num_image_elements = prod(image_dims);

  if( idx < num_image_elements ){
    
    const intd2::Type co = idx_to_co<2>(idx, image_dims);
    
    const int x = co.vec[0];
    const int y = co.vec[1];
    
    const int size_x = image_dims.vec[0];
    const int size_y = image_dims.vec[1];
    
    const int kernel_width = 7;
    const REAL scale = get_one<REAL>()/((REAL)(kernel_width*kernel_width));
    
    typename complext<REAL>::Type result = get_zero<typename complext<REAL>::Type>();
    
    for (int ky = 0; ky < kernel_width; ky++) {
      for (int kx = 0; kx < kernel_width; kx++) {
	
	if ((y-(kernel_width>>1)+ky) >= 0 &&
	    (y-(kernel_width>>1)+ky) < size_y &&
	    (x-(kernel_width>>1)+kx) >= 0 &&
	    (x-(kernel_width>>1)+kx) < size_x) 
	  {	    
	    int source_offset = 
	      batch*num_image_elements +
	      (y-(kernel_width>>1)+ky)*size_x +
	      (x-(kernel_width>>1)+kx);
	    
	    result += corrm[source_offset];
	  }
      }
    }
    corrm[batch*num_image_elements+idx] = scale*result;
  }
}

// Smooth correlation matrices by box filter (3D)
template<class REAL> __global__ void
smooth_correlation_matrices_kernel( typename complext<REAL>::Type *corrm, intd<3>::Type image_dims )
{
  const int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const int batch = blockIdx.y;

  const int num_image_elements = prod(image_dims);

  if( idx < num_image_elements ){
    
    const intd3::Type co = idx_to_co<3>(idx, image_dims);
    
    const int x = co.vec[0];
    const int y = co.vec[1];
    const int z = co.vec[2];
    
    const int size_x = image_dims.vec[0];
    const int size_y = image_dims.vec[1];
    const int size_z = image_dims.vec[2];
    
    const int kernel_width = 7;
    const REAL scale = get_one<REAL>()/((REAL)(kernel_width*kernel_width*kernel_width));
    
    typename complext<REAL>::Type result = get_zero<typename complext<REAL>::Type>();
    
    for (int kz = 0; kz < kernel_width; kz++) {
      for (int ky = 0; ky < kernel_width; ky++) {
	for (int kx = 0; kx < kernel_width; kx++) {
	
	  if ((z-(kernel_width>>1)+kz) >= 0 &&
	      (z-(kernel_width>>1)+kz) < size_z &&
	      (y-(kernel_width>>1)+ky) >= 0 &&
	      (y-(kernel_width>>1)+ky) < size_y &&
	      (x-(kernel_width>>1)+kx) >= 0 &&
	      (x-(kernel_width>>1)+kx) < size_x) 
	    {	    
	      int source_offset = 
		batch*num_image_elements +
		(z-(kernel_width>>1)+kz)*size_x*size_y +
		(y-(kernel_width>>1)+ky)*size_x +
		(x-(kernel_width>>1)+kx);
	    
	      result += corrm[source_offset];
	    }
	}
      }
    }
    corrm[batch*num_image_elements+idx] = scale*result;
  }
}

// Smooth correlation matrices by box filter (3D)
template<class REAL> __global__ void
smooth_correlation_matrices_kernel( typename complext<REAL>::Type *corrm, intd<4>::Type image_dims )
{
  const int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const int batch = blockIdx.y;

  const int num_image_elements = prod(image_dims);

  if( idx < num_image_elements ){
    
    const intd4::Type co = idx_to_co<4>(idx, image_dims);
    
    const int x = co.vec[0];
    const int y = co.vec[1];
    const int z = co.vec[2];
    const int w = co.vec[3];
    
    const int size_x = image_dims.vec[0];
    const int size_y = image_dims.vec[1];
    const int size_z = image_dims.vec[2];    
    const int size_w = image_dims.vec[3];
    
    const int kernel_width = 7;
    const REAL scale = get_one<REAL>()/((REAL)(kernel_width*kernel_width*kernel_width*kernel_width));
    
    typename complext<REAL>::Type result = get_zero<typename complext<REAL>::Type>();
    
    for (int kw = 0; kw < kernel_width; kw++) {
      for (int kz = 0; kz < kernel_width; kz++) {
	for (int ky = 0; ky < kernel_width; ky++) {
	  for (int kx = 0; kx < kernel_width; kx++) {
	
	    if ((w-(kernel_width>>1)+kw) >= 0 &&
		(w-(kernel_width>>1)+kw) < size_w &&
		(z-(kernel_width>>1)+kz) >= 0 &&
		(z-(kernel_width>>1)+kz) < size_z &&
		(y-(kernel_width>>1)+ky) >= 0 &&
		(y-(kernel_width>>1)+ky) < size_y &&
		(x-(kernel_width>>1)+kx) >= 0 &&
		(x-(kernel_width>>1)+kx) < size_x) 
	      {	    
		int source_offset = 
		  batch*num_image_elements +
		  (w-(kernel_width>>1)+kw)*size_x*size_y*size_z +
		  (z-(kernel_width>>1)+kz)*size_x*size_y +
		  (y-(kernel_width>>1)+ky)*size_x +
		  (x-(kernel_width>>1)+kx);
	    
		result += corrm[source_offset];
	      }
	  }
	}
      }
    }
    corrm[batch*num_image_elements+idx] = scale*result;
  }
}

template<class REAL, unsigned int D> void
smooth_correlation_matrices( cuNDArray<typename complext<REAL>::Type> *corrm )
{
  typename intd<D>::Type image_dims;

  for( unsigned int i=0; i<D; i++ ){
    image_dims.vec[i] = corrm->get_size(i);
  }
  
  unsigned int number_of_batches = 1;
  
  for( unsigned int i=D; i<corrm->get_number_of_dimensions(); i++ ){
    number_of_batches *= corrm->get_size(i);
  }
  
  dim3 blockDim(512);
  dim3 gridDim((unsigned int) ceil((double)prod(image_dims)/blockDim.x), number_of_batches);

  smooth_correlation_matrices_kernel<REAL><<<gridDim, blockDim>>>
    ( corrm->get_data_ptr(), image_dims );
  
  CHECK_FOR_CUDA_ERROR();
}


/*
  extern __shared__ char shared_mem[];

  // Extract CSM
  template<class REAL> __global__ void
  extract_csm_kernel( typename complext<REAL>::Type *corrm, typename complext<REAL>::Type *csm, unsigned int num_batches, unsigned int num_elements )
  {
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int i = threadIdx.x;

  if( idx < num_elements ){    
    
  // Get the dominant eigenvector for each correlation matrix.
  // Copying Peter Kellman's approach we use the power method:
  //  b_k+1 = A*b_k / ||A*b_k||
    
  typename complext<REAL>::Type *data_out = (typename complext<REAL>::Type*) shared_mem;
  typename complext<REAL>::Type *tmp_v = &(((typename complext<REAL>::Type*) shared_mem)[num_batches*blockDim.x]);

  const unsigned int iterations = 2;

  for( unsigned int c=0; c<num_batches; c++){
  data_out[c*blockDim.x+i] = get_one<typename complext<REAL>::Type >();
  }
    
  for( unsigned int it=0; it<iterations; it++ ){

  for( unsigned int c=0; c<num_batches; c++){
  tmp_v[c*blockDim.x+i] = get_zero<typename complext<REAL>::Type >();
  }
      
  for( unsigned j=0; j<num_batches; j++){
  for( unsigned int k=0; k<num_batches; k++){
  tmp_v[j*blockDim.x+i] += corrm[(k*num_batches+j)*num_elements+idx]*data_out[k*blockDim.x+i];
  }
  }

  REAL tmp = get_zero<REAL>();
      
  for (unsigned int c=0; c<num_batches; c++){
  tmp += norm_squared(tmp_v[c*blockDim.x+i]);
  }
      
  tmp = sqrt(tmp);
  tmp = reciprocal(tmp);
      
  for (unsigned int c=0; c<num_batches; c++){
  vectord<REAL,2> _res = tmp*tmp_v[c*blockDim.x+i];
  typename complext<REAL>::Type res; res.vec[0]=_res.vec[0]; res.vec[1]=_res.vec[1]; // TODO: do this assignment elegantly
  data_out[c*blockDim.x+i] = res;
  }
  }

  for (unsigned int c=0; c<num_batches; c++){
  csm[c*num_elements+idx] = data_out[c*blockDim.x+i];
  }
  }
  }

*/
// Extract CSM
template<class REAL> __global__ void
extract_csm_kernel( typename complext<REAL>::Type *corrm, typename complext<REAL>::Type *csm, unsigned int num_batches, unsigned int num_elements, typename complext<REAL>::Type *tmp_v )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( idx < num_elements ){    
    
    // Get the dominant eigenvector for each correlation matrix.
    // Copying Peter Kellman's approach we use the power method:
    //  b_k+1 = A*b_k / ||A*b_k||
    
    const unsigned int iterations = 2;

    for( unsigned int c=0; c<num_batches; c++){
      csm[c*num_elements+idx] = get_one<typename complext<REAL>::Type >();
    }
    
    for( unsigned int it=0; it<iterations; it++ ){

      for( unsigned int c=0; c<num_batches; c++){
	tmp_v[c*num_elements+idx] = get_zero<typename complext<REAL>::Type >();
      }
      
      for( unsigned j=0; j<num_batches; j++){
	for( unsigned int k=0; k<num_batches; k++){
	  typedef typename complext<REAL>::Type T;
	  tmp_v[j*num_elements+idx] += mul<T,T>(corrm[(k*num_batches+j)*num_elements+idx],csm[k*num_elements+idx]);
	}
      }

      REAL tmp = get_zero<REAL>();
      
      for (unsigned int c=0; c<num_batches; c++){
	tmp += norm_squared(tmp_v[c*num_elements+idx]);
      }
      
      tmp = sqrt(tmp);
      tmp = reciprocal(tmp);
      
      for (unsigned int c=0; c<num_batches; c++){
	typename complext<REAL>::Type res = tmp*tmp_v[c*num_elements+idx];
	csm[c*num_elements+idx] = res;
      }
    }
  }
}

// Extract CSM
template<class REAL> __host__ 
auto_ptr<cuNDArray<typename complext<REAL>::Type> > extract_csm(cuNDArray<typename complext<REAL>::Type> *corrm_in, unsigned int number_of_batches, unsigned int number_of_elements )
{
  vector<unsigned int> image_dims;

  for( unsigned int i=0; i<corrm_in->get_number_of_dimensions()-1; i++ ){
    image_dims.push_back(corrm_in->get_size(i));
  }
  
  // Allocate output
  cuNDArray<typename complext<REAL>::Type> *out = cuNDArray<typename complext<REAL>::Type>::allocate(image_dims);

  dim3 blockDim(128);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));

  /*  
      if( out != 0x0 )
      extract_csm_kernel<REAL><<< gridDim, blockDim, number_of_batches*blockDim.x*2*sizeof(typename complext<REAL>::Type) >>>
      ( corrm_in->get_data_ptr(), out->get_data_ptr(), number_of_batches, number_of_elements );
  */

  // Temporary buffer. TODO: use shared memory
  cuNDArray<typename complext<REAL>::Type> *tmp_v = cuNDArray<typename complext<REAL>::Type>::allocate(image_dims);

  if( out != 0x0 && tmp_v != 0x0 )
    extract_csm_kernel<REAL><<< gridDim, blockDim >>>
      ( corrm_in->get_data_ptr(), out->get_data_ptr(), number_of_batches, number_of_elements, tmp_v->get_data_ptr() );

  CHECK_FOR_CUDA_ERROR();
  
  delete tmp_v;
  return auto_ptr<cuNDArray<typename complext<REAL>::Type> >(out);
}

// Set refence phase
template<class REAL> __global__ void
set_phase_reference_kernel( typename complext<REAL>::Type *csm, unsigned int num_batches, unsigned int num_elements )
{
  const unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( idx < num_elements ){
    REAL angle = arg<REAL>(csm[idx]); //Phase of the first coil
    REAL sin_a, cos_a; sin_cos( angle, &sin_a, &cos_a );

    typename complext<REAL>::Type tmp;
    tmp.vec[0] = cos_a; tmp.vec[1] = sin_a;
    tmp = conj<REAL>(tmp);

    for( unsigned int c=0; c<num_batches; c++ ){
      typename complext<REAL>::Type val = csm[c*num_elements+idx];
      typedef typename complext<REAL>::Type T;
      val = mul<T,T>( val, tmp );
      csm[c*num_elements+idx] = val;
    }
  }
}
  
// Set reference phase
template<class REAL> __host__ 
void set_phase_reference(cuNDArray<typename complext<REAL>::Type> *csm, unsigned int number_of_batches, unsigned int number_of_elements )
{
  dim3 blockDim(128);
  dim3 gridDim((unsigned int) ceil((double)number_of_elements/blockDim.x));
  
  set_phase_reference_kernel<REAL><<< gridDim, blockDim >>>( csm->get_data_ptr(), number_of_batches, number_of_elements );
  
  CHECK_FOR_CUDA_ERROR();
}

//
// Template instantiation
//

template auto_ptr< cuNDArray<typename complext<float>::Type > > estimate_b1_map<float,1>(cuNDArray<typename complext<float>::Type >*);
template auto_ptr< cuNDArray<typename complext<float>::Type > > estimate_b1_map<float,2>(cuNDArray<typename complext<float>::Type >*);
template auto_ptr< cuNDArray<typename complext<float>::Type > > estimate_b1_map<float,3>(cuNDArray<typename complext<float>::Type >*);
template auto_ptr< cuNDArray<typename complext<float>::Type > > estimate_b1_map<float,4>(cuNDArray<typename complext<float>::Type >*);

template auto_ptr< cuNDArray<typename complext<double>::Type > > estimate_b1_map<double,1>(cuNDArray<typename complext<double>::Type >*);
template auto_ptr< cuNDArray<typename complext<double>::Type > > estimate_b1_map<double,2>(cuNDArray<typename complext<double>::Type >*);
template auto_ptr< cuNDArray<typename complext<double>::Type > > estimate_b1_map<double,3>(cuNDArray<typename complext<double>::Type >*);
template auto_ptr< cuNDArray<typename complext<double>::Type > > estimate_b1_map<double,4>(cuNDArray<typename complext<double>::Type >*);
