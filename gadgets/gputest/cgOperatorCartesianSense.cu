#include "hip/hip_runtime.h"
#include "cgOperatorCartesianSense.h"

#include "cuNDFFT.h"

__global__ void clear_array(float2* in, unsigned long int elements)
{
  unsigned long idx_in = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx_in < elements) {
    in[idx_in].x = 0.0;
    in[idx_in].y = 0.0;
  }
}


__global__ void sample_array_kernel(float2* in, float2* out, 
				    unsigned int* idx, 
				    unsigned long image_elements,
				    unsigned long int samples,
				    unsigned int coils)
{
  unsigned long idx_in = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx_in < samples) {
    for (unsigned int i = 0; i < coils; i++) {
      out[idx_in + i*samples].x += in[idx[idx_in] + i*image_elements].x;
      out[idx_in + i*samples].y += in[idx[idx_in] + i*image_elements].y;
    }
  }
}

__global__ void insert_samples_kernel(float2* in, float2* out, 
				      unsigned int* idx, 
				      unsigned long image_elements,
				      unsigned long int samples,
				      unsigned int coils)
{
  unsigned long idx_in = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx_in < samples) {
    for (unsigned int i = 0; i < coils; i++) {
      out[idx[idx_in] + i*image_elements].x += in[idx_in + i*samples].x;
      out[idx[idx_in] + i*image_elements].y += in[idx_in + i*samples].y;
    }
  }
}

__global__ void coil_sensitivity_multiplication(float2* in, float2* out, float2* csm, 
						unsigned long image_elements,
						unsigned int coils)
{
  unsigned long idx_in = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx_in < image_elements) {
    for (unsigned int i = 0; i < coils; i++) {
      out[idx_in + i*image_elements] = hipCmulf(in[idx_in],csm[idx_in + i*image_elements]);
    }
  }
}

__global__ void coil_sensitivity_conj_mult_sum(float2* in, float2* out, float2* csm, 
					       unsigned long image_elements,
					       unsigned int coils)
{
  unsigned long idx_in = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx_in < image_elements) {
    for (unsigned int i = 0; i < coils; i++) {
      float2 tmp = hipCmulf(in[idx_in + i*image_elements],hipConjf(csm[idx_in + i*image_elements]));
      out[idx_in].x += tmp.x;
      out[idx_in].y += tmp.y; 
    }
  }
}


int cgOperatorCartesianSense::mult_M(cuNDArray<float2>* in, 
				     cuNDArray<float2>* out, 
				     bool accumulate)
{
  if (!(in->dimensions_equal(dimensions_)) ||
      !(out->dimensions_equal(dimensions_out_)) ) {

    std::cerr << "cgOperatorCartesianSense::mult_M dimensions mismatch" << std::endl;

    return -1;
  }

  cuNDArray<float2> tmp;
  std::vector<unsigned int> full_dimensions = dimensions_;
  full_dimensions.push_back(coils_);

  if (!tmp.create(full_dimensions)) {
    std::cerr << "cgOperatorCartesianSense::mult_M unable to allocate temp array" << std::endl;
    return -1;    
  }

  dim3 blockDim(512,1,1);
  dim3 gridDim((unsigned int) ceil((double)in->get_number_of_elements()/blockDim.x), 1, 1 );
  coil_sensitivity_multiplication<<< gridDim, blockDim >>>( in->get_data_ptr(), tmp.get_data_ptr(),
							    csm_->get_data_ptr(), in->get_number_of_elements(), coils_);


  hipError_t err = hipGetLastError();
  if( err != hipSuccess ){
    std::cerr << "cgOperatorCartesianSense::mult_M : Unable to multiply with coil sensitivities: " << 
      hipGetErrorString(err) << std::endl;
    return -1;
  }

  cuNDFFT ft;
  std::vector<unsigned int> ft_dims;
  for (unsigned int i = 0; i < dimensions_.size(); i++) {
    ft_dims.push_back(i);
  }

  ft.fft(&tmp, ft_dims);

  if (!accumulate) clear(out);


  gridDim = dim3((unsigned int) ceil((double)idx_->get_number_of_elements()/blockDim.x), 1, 1 );
  sample_array_kernel<<< gridDim, blockDim >>>( tmp.get_data_ptr(), out->get_data_ptr(), idx_->get_data_ptr(),
						in->get_number_of_elements(), idx_->get_number_of_elements(), coils_);
  err = hipGetLastError();
  if( err != hipSuccess ){
    std::cerr << "cgOperatorCartesianSense::mult_M : Unable to sample data: " << 
      hipGetErrorString(err) << std::endl;
    return -1;
  }

  return 0;
}

int cgOperatorCartesianSense::mult_MH(cuNDArray<float2>* in, cuNDArray<float2>* out, bool accumulate)
{

  if (!(out->dimensions_equal(dimensions_)) ||
      !(in->dimensions_equal(dimensions_out_)) ) {
    std::cerr << "cgOperatorCartesianSense::mult_MH dimensions mismatch" << std::endl;
    return -1;
  }

  std::vector<unsigned int> tmp_dimensions = dimensions_;
  tmp_dimensions.push_back(coils_);

  cuNDArray<float2> tmp;
  if (!tmp.create(tmp_dimensions)) {
    std::cerr << "cgOperatorCartesianSense::mult_MH: Unable to create temp storage" << std::endl;
    return -1;
  }

  clear(&tmp);

  dim3 blockDim(512,1,1);
  dim3 gridDim((unsigned int) ceil((double)idx_->get_number_of_elements()/blockDim.x), 1, 1 );
  insert_samples_kernel<<< gridDim, blockDim >>>( in->get_data_ptr(), tmp.get_data_ptr(),
						  idx_->get_data_ptr(),out->get_number_of_elements(),
						  idx_->get_number_of_elements(), coils_);
  
  hipError_t err = hipGetLastError();
  if( err != hipSuccess ){
    std::cerr << "cgOperatorCartesianSense::mult_EM : Unable to insert samples into array: " << 
      hipGetErrorString(err) << std::endl;
    return -1;
  }

  cuNDFFT ft;
  std::vector<unsigned int> ft_dims;
  for (unsigned int i = 0; i < dimensions_.size(); i++) {
    ft_dims.push_back(i);
  }

  ft.ifft(&tmp, ft_dims);

  if (!accumulate) clear(out);

  gridDim = dim3((unsigned int) ceil((double)out->get_number_of_elements()/blockDim.x), 1, 1 );
  coil_sensitivity_conj_mult_sum<<< gridDim, blockDim >>>( tmp.get_data_ptr(), out->get_data_ptr(),
							   csm_->get_data_ptr(),out->get_number_of_elements(),
							   coils_);
  
  err = hipGetLastError();
  if( err != hipSuccess ){
    std::cerr << "cgOperatorCartesianSense::mult_EM : Unable to combine coils " << 
      hipGetErrorString(err) << std::endl;
    return -1;
  }

  return 0;
}

int cgOperatorCartesianSense::mult_MH_M(cuNDArray<float2>* in, cuNDArray<float2>* out, bool accumulate)
{
  cuNDArray<float2> tmp;
  if (!tmp.create(dimensions_out_)) {
    std::cerr << "cgOperatorCartesianSense::mult_MH_M: Unable to create temporary storage" << std::endl;
    return -1;
  }

  if (mult_M(in, &tmp, false) < 0) {
    std::cerr << "cgOperatorCartesianSense::mult_MH_M: Unable to perform mult_M" << std::endl;
    return -2;
  }

  if (mult_MH(&tmp, out, accumulate) < 0) {
    std::cerr << "cgOperatorCartesianSense::mult_MH_M: Unable to perform mult_M" << std::endl;
    return -2;
  }

  return 0;
}

int cgOperatorCartesianSense::clear(cuNDArray<float2>* in)
{
  dim3 blockDim(512,1,1);
  dim3 gridDim((unsigned int) ceil((double)in->get_number_of_elements()/blockDim.x), 1, 1 );

  clear_array<<< gridDim, blockDim >>>( in->get_data_ptr(), in->get_number_of_elements());

  hipError_t err = hipGetLastError();
  if( err != hipSuccess ){
    std::cerr << "cgOperatorCartesianSense::clear : Error during kernel call: " << hipGetErrorString(err) << std::endl;
    return -1;
  }

  return 0;
}

