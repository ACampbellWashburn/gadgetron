
#include <hip/hip_runtime.h>
//
// Kaiser-Bessel convolution kernels
//

// 'bessi0' is taken from numerical recipes in C

__inline__ __device__ double 
bessi0(double x)
{
  double ax,ans,y;
  if ((ax=fabs(x))<3.75) 
    {
      y=x/3.75;
      y*=y;
      ans=1.0+y*(3.5156229+y*(3.0899424+y*(1.2067492+y*(0.2659732+y*(0.0360768+y*0.0045813)))));
    } 
  else 
    {
      y=3.75/ax;
      ans=(-0.02057706+y*(0.02635537+y*(-0.01647633+(y*0.00392377))));
      ans=(exp(ax)/sqrt(ax))*(0.39894228+y*(0.01328592+y*(0.00225319+y*(-0.00157565+y*(0.00916281+y*ans)))));
    }
  return ans;
}

__inline__ __device__ float 
bessi0(float x)
{
  float ax,ans,y;
  if ((ax=fabsf(x)) <3.75f) 
    {
      y=x/3.75f;
      y*=y;
      ans=1.0f+y*(3.5156229f+y*(3.0899424f+y*(1.2067492f+y*(0.2659732f+y*(0.0360768f+y*0.0045813f)))));
    } 
  else 
    {
      y=3.75f/ax;
      ans=(-0.02057706f+y*(0.02635537f+y*(-0.01647633f+(y*0.00392377f))));
      ans=(expf(ax)/sqrtf(ax))*(0.39894228f+y*(0.01328592f+y*(0.00225319f+y*(-0.00157565f+y*(0.00916281f+y*ans)))));
    }
  return ans;
}

// Kaiser Bessel according to Beatty et. al. IEEE TMI 2005;24(6):799-808.
// There is a slight difference wrt Jackson's formulation, IEEE TMI 1991;10(3):473-478.

__inline__ __device__ double
KaiserBessel( double u, double matrix_size_os, double one_over_W, double beta )
{
  double _tmp = 2.0*u*one_over_W;
  double tmp = _tmp*_tmp;
  double arg = beta*sqrt(1.0-tmp);
  double bessi = bessi0(arg);
  double ret = matrix_size_os*bessi*one_over_W;
  return ret;
}

__inline__ __device__ float
KaiserBessel( float u, float matrix_size_os, float one_over_W, float beta )
{
  float _tmp = 2.0f*u*one_over_W;
  float tmp = _tmp*_tmp;
  float arg = beta*sqrtf(1.0f-tmp);
  float bessi = bessi0(arg);
  float ret = matrix_size_os*bessi*one_over_W;
  return ret;
}

//
// Below the intended interface
//

template<class REALd, class REAL> __inline__ __device__ REAL
KaiserBessel( REALd u, REALd matrix_size_os, REAL one_over_W, REAL beta, uint2 fixedDims )
{
  REAL one; get_one(one);
  REAL phi_x = (fixedDims.x) ? one : KaiserBessel( u.x, matrix_size_os.x, one_over_W, beta );
  REAL phi_y = (fixedDims.y) ? one : KaiserBessel( u.y, matrix_size_os.y, one_over_W, beta );

  return phi_x*phi_y;
}

template<class REALd, class REAL> __inline__ __device__ REAL
KaiserBessel( REALd u, REALd matrix_size_os, REAL one_over_W, REAL beta, uint3 fixedDims )
{
  REAL one; get_one(one);
  REAL phi_x = (fixedDims.x) ? one : KaiserBessel( u.x, matrix_size_os.x, one_over_W, beta );
  REAL phi_y = (fixedDims.y) ? one : KaiserBessel( u.y, matrix_size_os.y, one_over_W, beta );
  REAL phi_z = (fixedDims.z) ? one : KaiserBessel( u.z, matrix_size_os.z, one_over_W, beta );

  return phi_x*phi_y*phi_z;
}

template<class REALd, class REAL> __inline__ __device__ REAL
KaiserBessel( REALd u, REALd matrix_size_os, REAL one_over_W, REAL beta, uint4 fixedDims )
{
  REAL one; get_one(one);
  REAL phi_x = (fixedDims.x) ? one : KaiserBessel( u.x, matrix_size_os.x, one_over_W, beta );
  REAL phi_y = (fixedDims.y) ? one : KaiserBessel( u.y, matrix_size_os.y, one_over_W, beta );
  REAL phi_z = (fixedDims.z) ? one : KaiserBessel( u.z, matrix_size_os.z, one_over_W, beta );
  REAL phi_w = (fixedDims.w) ? one : KaiserBessel( u.w, matrix_size_os.w, one_over_W, beta );

  return phi_x*phi_y*phi_z*phi_w;
}
